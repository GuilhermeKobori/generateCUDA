#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#define pow powf
#define cell 1.000000
#define ksynp53mRNA 0.001000
#define kdegp53mRNA 0.000100
#define ksynMdm2mRNA 0.000500
#define kdegMdm2mRNA 0.000500
#define ksynMdm2mRNAGSK3bp53 0.000700
#define ksynp53 0.007000
#define kdegp53 0.005000
#define kbinMdm2p53 0.001155
#define krelMdm2p53 0.000012
#define kbinGSK3bp53 0.000002
#define krelGSK3bp53 0.002000
#define ksynMdm2 0.000495
#define kdegMdm2 0.010000
#define kbinE1Ub 0.000200
#define kbinE2Ub 0.001000
#define kp53Ub 0.000050
#define kp53PolyUb 0.010000
#define kbinProt 0.000002
#define kactDUBp53 0.000000
#define kactDUBProtp53 0.000100
#define kactDUBMdm2 0.000000
#define kMdm2Ub 0.000005
#define kMdm2PUb 0.000007
#define kMdm2PolyUb 0.004560
#define kdam 0.080000
#define krepair 0.000020
#define kactATM 0.000100
#define kinactATM 0.000500
#define kphosp53 0.000200
#define kdephosp53 0.500000
#define kphosMdm2 2.000000
#define kdephosMdm2 0.500000
#define kphosMdm2GSK3b 0.005000
#define kphosMdm2GSK3bp53 0.500000
#define kphospTauGSK3bp53 0.100000
#define kphospTauGSK3b 0.000200
#define kdephospTau 0.010000
#define kbinMTTau 0.100000
#define krelMTTau 0.000100
#define ksynTau 0.000080
#define kbinTauProt 0.000000
#define kdegTau20SProt 0.010000
#define kaggTau 0.000000
#define kaggTauP1 0.000000
#define kaggTauP2 0.000000
#define ktangfor 0.001000
#define kinhibprot 0.000000
#define ksynp53mRNAAbeta 0.000010
#define kdamROS 0.000010
#define kgenROSAbeta 0.000020
#define kgenROSPlaque 0.000010
#define kgenROSGlia 0.000010
#define kproteff 1.000000
#define kremROS 0.000070
#define kprodAbeta 0.000019
#define kprodAbeta2 0.000019
#define kdegAbeta 0.000015
#define kaggAbeta 0.000003
#define kdisaggAbeta 0.000001
#define kdisaggAbeta1 0.000200
#define kdisaggAbeta2 0.000001
#define kdegAbetaGlia 0.005000
#define kpf 0.200000
#define kpg 0.150000
#define kpghalf 10.000000
#define kactglia1 0.000001
#define kactglia2 0.000001
#define kinactglia1 0.000005
#define kinactglia2 0.000005
#define kbinAbetaGlia 0.000010
#define krelAbetaGlia 0.000050
#define kdegAntiAb 0.000003
#define kbinAbantiAb 0.000001
#define simulateStepReaction(i) \
switch (i) { \
case 0: \
reactionRate = ksynp53mRNA * Source; \
atomicAdd(p53_mRNA_aux, step * 1.000000 * reactionRate); \
break; \
case 1: \
reactionRate = kdegp53mRNA * p53_mRNA; \
atomicAdd(p53_mRNA_aux, - step * 1.000000 * reactionRate); \
break; \
case 2: \
reactionRate = ksynMdm2 * Mdm2_mRNA; \
atomicAdd(Mdm2_mRNA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_mRNA_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_aux, step * 1.000000 * reactionRate); \
break; \
case 3: \
reactionRate = ksynMdm2mRNA * p53; \
atomicAdd(p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_mRNA_aux, step * 1.000000 * reactionRate); \
break; \
case 4: \
reactionRate = ksynMdm2mRNA * p53_P; \
atomicAdd(p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_mRNA_aux, step * 1.000000 * reactionRate); \
break; \
case 5: \
reactionRate = ksynMdm2mRNAGSK3bp53 * GSK3b_p53; \
atomicAdd(GSK3b_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_mRNA_aux, step * 1.000000 * reactionRate); \
break; \
case 6: \
reactionRate = ksynMdm2mRNAGSK3bp53 * GSK3b_p53_P; \
atomicAdd(GSK3b_p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_mRNA_aux, step * 1.000000 * reactionRate); \
break; \
case 7: \
reactionRate = kdegMdm2mRNA * Mdm2_mRNA; \
atomicAdd(Mdm2_mRNA_aux, - step * 1.000000 * reactionRate); \
break; \
case 8: \
reactionRate = kbinMdm2p53 * p53 * Mdm2; \
atomicAdd(p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_aux, step * 1.000000 * reactionRate); \
break; \
case 9: \
reactionRate = krelMdm2p53 * Mdm2_p53; \
atomicAdd(Mdm2_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_aux, step * 1.000000 * reactionRate); \
break; \
case 10: \
reactionRate = kbinGSK3bp53 * GSK3b * p53; \
atomicAdd(GSK3b_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, step * 1.000000 * reactionRate); \
break; \
case 11: \
reactionRate = krelGSK3bp53 * GSK3b_p53; \
atomicAdd(GSK3b_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53_aux, step * 1.000000 * reactionRate); \
break; \
case 12: \
reactionRate = kbinGSK3bp53 * GSK3b * p53_P; \
atomicAdd(GSK3b_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, step * 1.000000 * reactionRate); \
break; \
case 13: \
reactionRate = krelGSK3bp53 * GSK3b_p53_P; \
atomicAdd(GSK3b_p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53_P_aux, step * 1.000000 * reactionRate); \
break; \
case 14: \
reactionRate = kbinE1Ub * E1 * Ub * ATP / (5000 + ATP); \
atomicAdd(E1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E1_Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 15: \
reactionRate = kbinE2Ub * E2 * E1_Ub; \
atomicAdd(E2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E1_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(E1_aux, step * 1.000000 * reactionRate); \
break; \
case 16: \
reactionRate = kMdm2Ub * Mdm2 * E2_Ub; \
atomicAdd(Mdm2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 17: \
reactionRate = kMdm2PolyUb * Mdm2_Ub * E2_Ub; \
atomicAdd(Mdm2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub2_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 18: \
reactionRate = kMdm2PolyUb * Mdm2_Ub2 * E2_Ub; \
atomicAdd(Mdm2_Ub2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub3_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 19: \
reactionRate = kMdm2PolyUb * Mdm2_Ub3 * E2_Ub; \
atomicAdd(Mdm2_Ub3_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub4_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 20: \
reactionRate = kactDUBMdm2 * Mdm2_Ub4 * Mdm2DUB; \
atomicAdd(Mdm2_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub3_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 21: \
reactionRate = kactDUBMdm2 * Mdm2_Ub3 * Mdm2DUB; \
atomicAdd(Mdm2_Ub3_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub2_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 22: \
reactionRate = kactDUBMdm2 * Mdm2_Ub2 * Mdm2DUB; \
atomicAdd(Mdm2_Ub2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 23: \
reactionRate = kactDUBMdm2 * Mdm2_Ub * Mdm2DUB; \
atomicAdd(Mdm2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 24: \
reactionRate = kbinProt * Mdm2_Ub4 * Proteasome; \
atomicAdd(Mdm2_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_Ub4_Proteasome_aux, step * 1.000000 * reactionRate); \
break; \
case 25: \
reactionRate = kdegMdm2 * Mdm2_Ub4_Proteasome * kproteff; \
atomicAdd(Mdm2_Ub4_Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 4.000000 * reactionRate); \
break; \
case 26: \
reactionRate = ksynp53 * p53_mRNA; \
atomicAdd(p53_mRNA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53_mRNA_aux, step * 1.000000 * reactionRate); \
break; \
case 27: \
reactionRate = kp53Ub * E2_Ub * Mdm2_p53; \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 28: \
reactionRate = kp53PolyUb * Mdm2_p53_Ub * E2_Ub; \
atomicAdd(Mdm2_p53_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub2_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 29: \
reactionRate = kp53PolyUb * Mdm2_p53_Ub2 * E2_Ub; \
atomicAdd(Mdm2_p53_Ub2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub3_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 30: \
reactionRate = kp53PolyUb * Mdm2_p53_Ub3 * E2_Ub; \
atomicAdd(Mdm2_p53_Ub3_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub4_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 31: \
reactionRate = kactDUBp53 * Mdm2_p53_Ub4 * p53DUB; \
atomicAdd(Mdm2_p53_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub3_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 32: \
reactionRate = kactDUBp53 * Mdm2_p53_Ub3 * p53DUB; \
atomicAdd(Mdm2_p53_Ub3_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub2_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 33: \
reactionRate = kactDUBp53 * Mdm2_p53_Ub2 * p53DUB; \
atomicAdd(Mdm2_p53_Ub2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 34: \
reactionRate = kactDUBp53 * Mdm2_p53_Ub * p53DUB; \
atomicAdd(Mdm2_p53_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(p53DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 35: \
reactionRate = kphosMdm2GSK3b * Mdm2_p53_Ub4 * GSK3b; \
atomicAdd(Mdm2_p53_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P1_p53_Ub4_aux, step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_aux, step * 1.000000 * reactionRate); \
break; \
case 36: \
reactionRate = kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53; \
atomicAdd(Mdm2_p53_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P1_p53_Ub4_aux, step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, step * 1.000000 * reactionRate); \
break; \
case 37: \
reactionRate = kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53_P; \
atomicAdd(Mdm2_p53_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P1_p53_Ub4_aux, step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, step * 1.000000 * reactionRate); \
break; \
case 38: \
reactionRate = kbinProt * Mdm2_P1_p53_Ub4 * Proteasome; \
atomicAdd(Mdm2_P1_p53_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_Ub4_Proteasome_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_aux, step * 1.000000 * reactionRate); \
break; \
case 39: \
reactionRate = kdegp53 * kproteff * p53_Ub4_Proteasome * ATP / (5000 + ATP); \
atomicAdd(p53_Ub4_Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 4.000000 * reactionRate); \
atomicAdd(Proteasome_aux, step * 1.000000 * reactionRate); \
break; \
case 40: \
reactionRate = kbinMTTau * Tau; \
atomicAdd(Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(MT_Tau_aux, step * 1.000000 * reactionRate); \
break; \
case 41: \
reactionRate = krelMTTau * MT_Tau; \
atomicAdd(MT_Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_aux, step * 1.000000 * reactionRate); \
break; \
case 42: \
reactionRate = kphospTauGSK3bp53 * GSK3b_p53 * Tau; \
atomicAdd(GSK3b_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, step * 1.000000 * reactionRate); \
break; \
case 43: \
reactionRate = kphospTauGSK3bp53 * GSK3b_p53 * Tau_P1; \
atomicAdd(GSK3b_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, step * 1.000000 * reactionRate); \
atomicAdd(Tau_P2_aux, step * 1.000000 * reactionRate); \
break; \
case 44: \
reactionRate = kphospTauGSK3bp53 * GSK3b_p53_P * Tau; \
atomicAdd(GSK3b_p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, step * 1.000000 * reactionRate); \
break; \
case 45: \
reactionRate = kphospTauGSK3bp53 * GSK3b_p53_P * Tau_P1; \
atomicAdd(GSK3b_p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(Tau_P2_aux, step * 1.000000 * reactionRate); \
break; \
case 46: \
reactionRate = kphospTauGSK3b * GSK3b * Tau; \
atomicAdd(GSK3b_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_aux, step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, step * 1.000000 * reactionRate); \
break; \
case 47: \
reactionRate = kphospTauGSK3b * GSK3b * Tau_P1; \
atomicAdd(GSK3b_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_aux, step * 1.000000 * reactionRate); \
atomicAdd(Tau_P2_aux, step * 1.000000 * reactionRate); \
break; \
case 48: \
reactionRate = kdephospTau * Tau_P2 * PP1; \
atomicAdd(Tau_P2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(PP1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_P1_aux, step * 1.000000 * reactionRate); \
atomicAdd(PP1_aux, step * 1.000000 * reactionRate); \
break; \
case 49: \
reactionRate = kdephospTau * Tau_P1 * PP1; \
atomicAdd(Tau_P1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(PP1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Tau_aux, step * 1.000000 * reactionRate); \
atomicAdd(PP1_aux, step * 1.000000 * reactionRate); \
break; \
case 50: \
reactionRate = kaggTauP1 * Tau_P1 * (Tau_P1 - 1) * 0.5; \
atomicAdd(Tau_P1_aux, - step * 2.000000 * reactionRate); \
atomicAdd(AggTau_aux, step * 2.000000 * reactionRate); \
break; \
case 51: \
reactionRate = kaggTauP1 * Tau_P1 * AggTau; \
atomicAdd(Tau_P1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_aux, step * 2.000000 * reactionRate); \
break; \
case 52: \
reactionRate = kaggTauP2 * Tau_P2 * (Tau_P2 - 1) * 0.5; \
atomicAdd(Tau_P2_aux, - step * 2.000000 * reactionRate); \
atomicAdd(AggTau_aux, step * 2.000000 * reactionRate); \
break; \
case 53: \
reactionRate = kaggTauP2 * Tau_P2 * AggTau; \
atomicAdd(Tau_P2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_aux, step * 2.000000 * reactionRate); \
break; \
case 54: \
reactionRate = kaggTau * Tau * (Tau - 1) * 0.5; \
atomicAdd(Tau_aux, - step * 2.000000 * reactionRate); \
atomicAdd(AggTau_aux, step * 2.000000 * reactionRate); \
break; \
case 55: \
reactionRate = kaggTau * Tau * AggTau; \
atomicAdd(Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_aux, step * 2.000000 * reactionRate); \
break; \
case 56: \
reactionRate = ktangfor * AggTau * (AggTau - 1) * 0.5; \
atomicAdd(AggTau_aux, - step * 2.000000 * reactionRate); \
atomicAdd(NFT_aux, step * 2.000000 * reactionRate); \
break; \
case 57: \
reactionRate = ktangfor * AggTau * NFT; \
atomicAdd(AggTau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(NFT_aux, - step * 1.000000 * reactionRate); \
atomicAdd(NFT_aux, step * 2.000000 * reactionRate); \
break; \
case 58: \
reactionRate = kinhibprot * AggTau * Proteasome; \
atomicAdd(AggTau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggTau_Proteasome_aux, step * 1.000000 * reactionRate); \
break; \
case 59: \
reactionRate = kprodAbeta * Source; \
atomicAdd(Abeta_aux, step * 1.000000 * reactionRate); \
break; \
case 60: \
reactionRate = kprodAbeta2 * GSK3b_p53; \
atomicAdd(GSK3b_p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Abeta_aux, step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_aux, step * 1.000000 * reactionRate); \
break; \
case 61: \
reactionRate = kprodAbeta2 * GSK3b_p53_P; \
atomicAdd(GSK3b_p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Abeta_aux, step * 1.000000 * reactionRate); \
atomicAdd(GSK3b_p53_P_aux, step * 1.000000 * reactionRate); \
break; \
case 62: \
reactionRate = kinhibprot * AbetaDimer * Proteasome; \
atomicAdd(AbetaDimer_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggAbeta_Proteasome_aux, step * 1.000000 * reactionRate); \
break; \
case 63: \
reactionRate = kdegAbeta * Abeta; \
atomicAdd(Abeta_aux, - step * 1.000000 * reactionRate); \
break; \
case 64: \
reactionRate = ksynp53mRNAAbeta * Abeta; \
atomicAdd(Abeta_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_mRNA_aux, step * 1.000000 * reactionRate); \
atomicAdd(Abeta_aux, step * 1.000000 * reactionRate); \
break; \
case 65: \
reactionRate = kdam * IR; \
atomicAdd(IR_aux, - step * 1.000000 * reactionRate); \
atomicAdd(IR_aux, step * 1.000000 * reactionRate); \
atomicAdd(damDNA_aux, step * 1.000000 * reactionRate); \
break; \
case 66: \
reactionRate = krepair * damDNA; \
atomicAdd(damDNA_aux, - step * 1.000000 * reactionRate); \
break; \
case 67: \
reactionRate = kactATM * damDNA * ATMI; \
atomicAdd(damDNA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(ATMI_aux, - step * 1.000000 * reactionRate); \
atomicAdd(damDNA_aux, step * 1.000000 * reactionRate); \
atomicAdd(ATMA_aux, step * 1.000000 * reactionRate); \
break; \
case 68: \
reactionRate = kphosp53 * p53 * ATMA; \
atomicAdd(p53_aux, - step * 1.000000 * reactionRate); \
atomicAdd(ATMA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(ATMA_aux, step * 1.000000 * reactionRate); \
break; \
case 69: \
reactionRate = kdephosp53 * p53_P; \
atomicAdd(p53_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(p53_aux, step * 1.000000 * reactionRate); \
break; \
case 70: \
reactionRate = kphosMdm2 * Mdm2 * ATMA; \
atomicAdd(Mdm2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(ATMA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(ATMA_aux, step * 1.000000 * reactionRate); \
break; \
case 71: \
reactionRate = kdephosMdm2 * Mdm2_P; \
atomicAdd(Mdm2_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_aux, step * 1.000000 * reactionRate); \
break; \
case 72: \
reactionRate = kMdm2PUb * Mdm2_P * E2_Ub; \
atomicAdd(Mdm2_P_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 73: \
reactionRate = kMdm2PolyUb * Mdm2_P_Ub * E2_Ub; \
atomicAdd(Mdm2_P_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub2_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 74: \
reactionRate = kMdm2PolyUb * Mdm2_P_Ub2 * E2_Ub; \
atomicAdd(Mdm2_P_Ub2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub3_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 75: \
reactionRate = kMdm2PolyUb * Mdm2_P_Ub3 * E2_Ub; \
atomicAdd(Mdm2_P_Ub3_aux, - step * 1.000000 * reactionRate); \
atomicAdd(E2_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub4_aux, step * 1.000000 * reactionRate); \
atomicAdd(E2_aux, step * 1.000000 * reactionRate); \
break; \
case 76: \
reactionRate = kactDUBMdm2 * Mdm2_P_Ub4 * Mdm2DUB; \
atomicAdd(Mdm2_P_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub3_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 77: \
reactionRate = kactDUBMdm2 * Mdm2_P_Ub3 * Mdm2DUB; \
atomicAdd(Mdm2_P_Ub3_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub2_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 78: \
reactionRate = kactDUBMdm2 * Mdm2_P_Ub2 * Mdm2DUB; \
atomicAdd(Mdm2_P_Ub2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 79: \
reactionRate = kactDUBMdm2 * Mdm2_P_Ub * Mdm2DUB; \
atomicAdd(Mdm2_P_Ub_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_aux, step * 1.000000 * reactionRate); \
atomicAdd(Mdm2DUB_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 1.000000 * reactionRate); \
break; \
case 80: \
reactionRate = kbinProt * Mdm2_P_Ub4 * Proteasome; \
atomicAdd(Mdm2_P_Ub4_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Mdm2_P_Ub4_Proteasome_aux, step * 1.000000 * reactionRate); \
break; \
case 81: \
reactionRate = kdegMdm2 * Mdm2_P_Ub4_Proteasome * kproteff; \
atomicAdd(Mdm2_P_Ub4_Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, step * 1.000000 * reactionRate); \
atomicAdd(Ub_aux, step * 4.000000 * reactionRate); \
break; \
case 82: \
reactionRate = kinactATM * ATMA; \
atomicAdd(ATMA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(ATMI_aux, step * 1.000000 * reactionRate); \
break; \
case 83: \
reactionRate = kgenROSAbeta * Abeta; \
atomicAdd(Abeta_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Abeta_aux, step * 1.000000 * reactionRate); \
atomicAdd(ROS_aux, step * 1.000000 * reactionRate); \
break; \
case 84: \
reactionRate = kgenROSPlaque * AbetaPlaque; \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, step * 1.000000 * reactionRate); \
atomicAdd(ROS_aux, step * 1.000000 * reactionRate); \
break; \
case 85: \
reactionRate = kgenROSAbeta * AggAbeta_Proteasome; \
atomicAdd(AggAbeta_Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AggAbeta_Proteasome_aux, step * 1.000000 * reactionRate); \
atomicAdd(ROS_aux, step * 1.000000 * reactionRate); \
break; \
case 86: \
reactionRate = kdamROS * ROS; \
atomicAdd(ROS_aux, - step * 1.000000 * reactionRate); \
atomicAdd(ROS_aux, step * 1.000000 * reactionRate); \
atomicAdd(damDNA_aux, step * 1.000000 * reactionRate); \
break; \
case 87: \
reactionRate = ksynTau * Source; \
atomicAdd(Tau_aux, step * 1.000000 * reactionRate); \
break; \
case 88: \
reactionRate = kbinTauProt * Tau * Proteasome; \
atomicAdd(Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_Tau_aux, step * 1.000000 * reactionRate); \
break; \
case 89: \
reactionRate = kdegTau20SProt * Proteasome_Tau; \
atomicAdd(Proteasome_Tau_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Proteasome_aux, step * 1.000000 * reactionRate); \
break; \
case 90: \
reactionRate = kaggAbeta * Abeta * (Abeta - 1) * 0.5; \
atomicAdd(Abeta_aux, - step * 2.000000 * reactionRate); \
atomicAdd(AbetaDimer_aux, step * 1.000000 * reactionRate); \
break; \
case 91: \
reactionRate = kpf * AbetaDimer * (AbetaDimer - 1) * 0.5; \
atomicAdd(AbetaDimer_aux, - step * 2.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, step * 1.000000 * reactionRate); \
break; \
case 92: \
reactionRate = kpg * AbetaDimer * pow(AbetaPlaque, 2) / (pow(kpghalf, 2) + pow(AbetaPlaque, 2)); \
atomicAdd(AbetaDimer_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, step * 2.000000 * reactionRate); \
break; \
case 93: \
reactionRate = kdisaggAbeta * AbetaDimer; \
atomicAdd(AbetaDimer_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Abeta_aux, step * 2.000000 * reactionRate); \
break; \
case 94: \
reactionRate = kdisaggAbeta1 * AbetaPlaque; \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaDimer_aux, step * 1.000000 * reactionRate); \
atomicAdd(disaggPlaque1_aux, step * 1.000000 * reactionRate); \
break; \
case 95: \
reactionRate = kdisaggAbeta2 * antiAb * AbetaPlaque; \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaDimer_aux, step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, step * 1.000000 * reactionRate); \
atomicAdd(disaggPlaque2_aux, step * 1.000000 * reactionRate); \
break; \
case 96: \
reactionRate = kbinAbantiAb * Abeta * antiAb; \
atomicAdd(Abeta_aux, - step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, - step * 1.000000 * reactionRate); \
atomicAdd(Abeta_antiAb_aux, step * 1.000000 * reactionRate); \
break; \
case 97: \
reactionRate = kbinAbantiAb * AbetaDimer * antiAb; \
atomicAdd(AbetaDimer_aux, - step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaDimer_antiAb_aux, step * 1.000000 * reactionRate); \
break; \
case 98: \
reactionRate = 10 * kdegAbeta * Abeta_antiAb; \
atomicAdd(Abeta_antiAb_aux, - step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, step * 1.000000 * reactionRate); \
break; \
case 99: \
reactionRate = 10 * kdegAbeta * AbetaDimer_antiAb; \
atomicAdd(AbetaDimer_antiAb_aux, - step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, step * 1.000000 * reactionRate); \
break; \
case 100: \
reactionRate = kactglia1 * GliaI * AbetaPlaque; \
atomicAdd(GliaI_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaM1_aux, step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, step * 1.000000 * reactionRate); \
break; \
case 101: \
reactionRate = kactglia1 * GliaM1 * AbetaPlaque; \
atomicAdd(GliaM1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaM2_aux, step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, step * 1.000000 * reactionRate); \
break; \
case 102: \
reactionRate = kactglia2 * GliaM2 * antiAb; \
atomicAdd(GliaM2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaA_aux, step * 1.000000 * reactionRate); \
atomicAdd(antiAb_aux, step * 1.000000 * reactionRate); \
break; \
case 103: \
reactionRate = kinactglia1 * GliaA; \
atomicAdd(GliaA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaM2_aux, step * 1.000000 * reactionRate); \
break; \
case 104: \
reactionRate = kinactglia2 * GliaM2; \
atomicAdd(GliaM2_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaM1_aux, step * 1.000000 * reactionRate); \
break; \
case 105: \
reactionRate = kinactglia2 * GliaM1; \
atomicAdd(GliaM1_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaI_aux, step * 1.000000 * reactionRate); \
break; \
case 106: \
reactionRate = kbinAbetaGlia * AbetaPlaque * GliaA; \
atomicAdd(AbetaPlaque_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_GliaA_aux, step * 1.000000 * reactionRate); \
break; \
case 107: \
reactionRate = krelAbetaGlia * AbetaPlaque_GliaA; \
atomicAdd(AbetaPlaque_GliaA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_aux, step * 1.000000 * reactionRate); \
atomicAdd(GliaA_aux, step * 1.000000 * reactionRate); \
break; \
case 108: \
reactionRate = kdegAbetaGlia * AbetaPlaque_GliaA; \
atomicAdd(AbetaPlaque_GliaA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(GliaA_aux, step * 1.000000 * reactionRate); \
atomicAdd(degAbetaGlia_aux, step * 1.000000 * reactionRate); \
break; \
case 109: \
reactionRate = kgenROSGlia * AbetaPlaque_GliaA; \
atomicAdd(AbetaPlaque_GliaA_aux, - step * 1.000000 * reactionRate); \
atomicAdd(AbetaPlaque_GliaA_aux, step * 1.000000 * reactionRate); \
atomicAdd(ROS_aux, step * 1.000000 * reactionRate); \
break; \
case 110: \
reactionRate = kdegAntiAb * antiAb; \
atomicAdd(antiAb_aux, - step * 1.000000 * reactionRate); \
break; \
case 111: \
reactionRate = kremROS * ROS; \
atomicAdd(ROS_aux, - step * 1.000000 * reactionRate); \
break; \
} \


__global__
void simulate (float step, int numSimulations, float Mdm2, float* Mdm2_aux, float p53, float* p53_aux, float Mdm2_p53, float* Mdm2_p53_aux, float Mdm2_mRNA, float* Mdm2_mRNA_aux, float p53_mRNA, float* p53_mRNA_aux, float ATMA, float* ATMA_aux, float ATMI, float* ATMI_aux, float p53_P, float* p53_P_aux, float Mdm2_P, float* Mdm2_P_aux, float IR, float* IR_aux, float ROS, float* ROS_aux, float damDNA, float* damDNA_aux, float E1, float* E1_aux, float E2, float* E2_aux, float E1_Ub, float* E1_Ub_aux, float E2_Ub, float* E2_Ub_aux, float Proteasome, float* Proteasome_aux, float Ub, float* Ub_aux, float p53DUB, float* p53DUB_aux, float Mdm2DUB, float* Mdm2DUB_aux, float DUB, float* DUB_aux, float Mdm2_p53_Ub, float* Mdm2_p53_Ub_aux, float Mdm2_p53_Ub2, float* Mdm2_p53_Ub2_aux, float Mdm2_p53_Ub3, float* Mdm2_p53_Ub3_aux, float Mdm2_p53_Ub4, float* Mdm2_p53_Ub4_aux, float Mdm2_P1_p53_Ub4, float* Mdm2_P1_p53_Ub4_aux, float Mdm2_Ub, float* Mdm2_Ub_aux, float Mdm2_Ub2, float* Mdm2_Ub2_aux, float Mdm2_Ub3, float* Mdm2_Ub3_aux, float Mdm2_Ub4, float* Mdm2_Ub4_aux, float Mdm2_P_Ub, float* Mdm2_P_Ub_aux, float Mdm2_P_Ub2, float* Mdm2_P_Ub2_aux, float Mdm2_P_Ub3, float* Mdm2_P_Ub3_aux, float Mdm2_P_Ub4, float* Mdm2_P_Ub4_aux, float p53_Ub4_Proteasome, float* p53_Ub4_Proteasome_aux, float Mdm2_Ub4_Proteasome, float* Mdm2_Ub4_Proteasome_aux, float Mdm2_P_Ub4_Proteasome, float* Mdm2_P_Ub4_Proteasome_aux, float GSK3b, float* GSK3b_aux, float GSK3b_p53, float* GSK3b_p53_aux, float GSK3b_p53_P, float* GSK3b_p53_P_aux, float Abeta, float* Abeta_aux, float AggAbeta_Proteasome, float* AggAbeta_Proteasome_aux, float AbetaPlaque, float* AbetaPlaque_aux, float Tau, float* Tau_aux, float Tau_P1, float* Tau_P1_aux, float Tau_P2, float* Tau_P2_aux, float MT_Tau, float* MT_Tau_aux, float AggTau, float* AggTau_aux, float AggTau_Proteasome, float* AggTau_Proteasome_aux, float Proteasome_Tau, float* Proteasome_Tau_aux, float PP1, float* PP1_aux, float NFT, float* NFT_aux, float ATP, float* ATP_aux, float ADP, float* ADP_aux, float AMP, float* AMP_aux, float AbetaDimer, float* AbetaDimer_aux, float AbetaPlaque_GliaA, float* AbetaPlaque_GliaA_aux, float GliaI, float* GliaI_aux, float GliaM1, float* GliaM1_aux, float GliaM2, float* GliaM2_aux, float GliaA, float* GliaA_aux, float antiAb, float* antiAb_aux, float Abeta_antiAb, float* Abeta_antiAb_aux, float AbetaDimer_antiAb, float* AbetaDimer_antiAb_aux, float degAbetaGlia, float* degAbetaGlia_aux, float disaggPlaque1, float* disaggPlaque1_aux, float disaggPlaque2, float* disaggPlaque2_aux, float Source, float* Source_aux, float Sink, float* Sink_aux) {
float reactionRate;
for(int i = 0; i < numSimulations; i++){
Mdm2 = *Mdm2_aux;
p53 = *p53_aux;
Mdm2_p53 = *Mdm2_p53_aux;
Mdm2_mRNA = *Mdm2_mRNA_aux;
p53_mRNA = *p53_mRNA_aux;
ATMA = *ATMA_aux;
ATMI = *ATMI_aux;
p53_P = *p53_P_aux;
Mdm2_P = *Mdm2_P_aux;
IR = *IR_aux;
ROS = *ROS_aux;
damDNA = *damDNA_aux;
E1 = *E1_aux;
E2 = *E2_aux;
E1_Ub = *E1_Ub_aux;
E2_Ub = *E2_Ub_aux;
Proteasome = *Proteasome_aux;
Ub = *Ub_aux;
p53DUB = *p53DUB_aux;
Mdm2DUB = *Mdm2DUB_aux;
DUB = *DUB_aux;
Mdm2_p53_Ub = *Mdm2_p53_Ub_aux;
Mdm2_p53_Ub2 = *Mdm2_p53_Ub2_aux;
Mdm2_p53_Ub3 = *Mdm2_p53_Ub3_aux;
Mdm2_p53_Ub4 = *Mdm2_p53_Ub4_aux;
Mdm2_P1_p53_Ub4 = *Mdm2_P1_p53_Ub4_aux;
Mdm2_Ub = *Mdm2_Ub_aux;
Mdm2_Ub2 = *Mdm2_Ub2_aux;
Mdm2_Ub3 = *Mdm2_Ub3_aux;
Mdm2_Ub4 = *Mdm2_Ub4_aux;
Mdm2_P_Ub = *Mdm2_P_Ub_aux;
Mdm2_P_Ub2 = *Mdm2_P_Ub2_aux;
Mdm2_P_Ub3 = *Mdm2_P_Ub3_aux;
Mdm2_P_Ub4 = *Mdm2_P_Ub4_aux;
p53_Ub4_Proteasome = *p53_Ub4_Proteasome_aux;
Mdm2_Ub4_Proteasome = *Mdm2_Ub4_Proteasome_aux;
Mdm2_P_Ub4_Proteasome = *Mdm2_P_Ub4_Proteasome_aux;
GSK3b = *GSK3b_aux;
GSK3b_p53 = *GSK3b_p53_aux;
GSK3b_p53_P = *GSK3b_p53_P_aux;
Abeta = *Abeta_aux;
AggAbeta_Proteasome = *AggAbeta_Proteasome_aux;
AbetaPlaque = *AbetaPlaque_aux;
Tau = *Tau_aux;
Tau_P1 = *Tau_P1_aux;
Tau_P2 = *Tau_P2_aux;
MT_Tau = *MT_Tau_aux;
AggTau = *AggTau_aux;
AggTau_Proteasome = *AggTau_Proteasome_aux;
Proteasome_Tau = *Proteasome_Tau_aux;
PP1 = *PP1_aux;
NFT = *NFT_aux;
ATP = *ATP_aux;
ADP = *ADP_aux;
AMP = *AMP_aux;
AbetaDimer = *AbetaDimer_aux;
AbetaPlaque_GliaA = *AbetaPlaque_GliaA_aux;
GliaI = *GliaI_aux;
GliaM1 = *GliaM1_aux;
GliaM2 = *GliaM2_aux;
GliaA = *GliaA_aux;
antiAb = *antiAb_aux;
Abeta_antiAb = *Abeta_antiAb_aux;
AbetaDimer_antiAb = *AbetaDimer_antiAb_aux;
degAbetaGlia = *degAbetaGlia_aux;
disaggPlaque1 = *disaggPlaque1_aux;
disaggPlaque2 = *disaggPlaque2_aux;
Source = *Source_aux;
Sink = *Sink_aux;
simulateStepReaction(threadIdx.x);
__syncthreads();
}
}

int main()
{
hipError_t cudaStatus;
float Mdm2 = 5.000000;
float* dev_Mdm2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2, &Mdm2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53 = 5.000000;
float* dev_p53 = 0;
cudaStatus = hipMalloc(&dev_p53, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53, &p53, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53 = 95.000000;
float* dev_Mdm2_p53 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53, &Mdm2_p53, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_mRNA = 10.000000;
float* dev_Mdm2_mRNA = 0;
cudaStatus = hipMalloc(&dev_Mdm2_mRNA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_mRNA, &Mdm2_mRNA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53_mRNA = 10.000000;
float* dev_p53_mRNA = 0;
cudaStatus = hipMalloc(&dev_p53_mRNA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53_mRNA, &p53_mRNA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ATMA = 0.000000;
float* dev_ATMA = 0;
cudaStatus = hipMalloc(&dev_ATMA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ATMA, &ATMA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ATMI = 200.000000;
float* dev_ATMI = 0;
cudaStatus = hipMalloc(&dev_ATMI, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ATMI, &ATMI, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53_P = 0.000000;
float* dev_p53_P = 0;
cudaStatus = hipMalloc(&dev_p53_P, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53_P, &p53_P, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P = 0.000000;
float* dev_Mdm2_P = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P, &Mdm2_P, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float IR = 0.000000;
float* dev_IR = 0;
cudaStatus = hipMalloc(&dev_IR, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_IR, &IR, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ROS = 0.000000;
float* dev_ROS = 0;
cudaStatus = hipMalloc(&dev_ROS, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ROS, &ROS, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float damDNA = 0.000000;
float* dev_damDNA = 0;
cudaStatus = hipMalloc(&dev_damDNA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_damDNA, &damDNA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E1 = 100.000000;
float* dev_E1 = 0;
cudaStatus = hipMalloc(&dev_E1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E1, &E1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E2 = 100.000000;
float* dev_E2 = 0;
cudaStatus = hipMalloc(&dev_E2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E2, &E2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E1_Ub = 0.000000;
float* dev_E1_Ub = 0;
cudaStatus = hipMalloc(&dev_E1_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E1_Ub, &E1_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E2_Ub = 0.000000;
float* dev_E2_Ub = 0;
cudaStatus = hipMalloc(&dev_E2_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E2_Ub, &E2_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Proteasome = 500.000000;
float* dev_Proteasome = 0;
cudaStatus = hipMalloc(&dev_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Proteasome, &Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Ub = 4000.000000;
float* dev_Ub = 0;
cudaStatus = hipMalloc(&dev_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Ub, &Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53DUB = 200.000000;
float* dev_p53DUB = 0;
cudaStatus = hipMalloc(&dev_p53DUB, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53DUB, &p53DUB, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2DUB = 200.000000;
float* dev_Mdm2DUB = 0;
cudaStatus = hipMalloc(&dev_Mdm2DUB, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2DUB, &Mdm2DUB, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float DUB = 200.000000;
float* dev_DUB = 0;
cudaStatus = hipMalloc(&dev_DUB, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_DUB, &DUB, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub = 0.000000;
float* dev_Mdm2_p53_Ub = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub, &Mdm2_p53_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub2 = 0.000000;
float* dev_Mdm2_p53_Ub2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub2, &Mdm2_p53_Ub2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub3 = 0.000000;
float* dev_Mdm2_p53_Ub3 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub3, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub3, &Mdm2_p53_Ub3, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub4 = 0.000000;
float* dev_Mdm2_p53_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub4, &Mdm2_p53_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P1_p53_Ub4 = 0.000000;
float* dev_Mdm2_P1_p53_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P1_p53_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P1_p53_Ub4, &Mdm2_P1_p53_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub = 0.000000;
float* dev_Mdm2_Ub = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub, &Mdm2_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub2 = 0.000000;
float* dev_Mdm2_Ub2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub2, &Mdm2_Ub2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub3 = 0.000000;
float* dev_Mdm2_Ub3 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub3, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub3, &Mdm2_Ub3, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub4 = 0.000000;
float* dev_Mdm2_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub4, &Mdm2_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub = 0.000000;
float* dev_Mdm2_P_Ub = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub, &Mdm2_P_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub2 = 0.000000;
float* dev_Mdm2_P_Ub2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub2, &Mdm2_P_Ub2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub3 = 0.000000;
float* dev_Mdm2_P_Ub3 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub3, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub3, &Mdm2_P_Ub3, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub4 = 0.000000;
float* dev_Mdm2_P_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub4, &Mdm2_P_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53_Ub4_Proteasome = 0.000000;
float* dev_p53_Ub4_Proteasome = 0;
cudaStatus = hipMalloc(&dev_p53_Ub4_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53_Ub4_Proteasome, &p53_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub4_Proteasome = 0.000000;
float* dev_Mdm2_Ub4_Proteasome = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub4_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub4_Proteasome, &Mdm2_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub4_Proteasome = 0.000000;
float* dev_Mdm2_P_Ub4_Proteasome = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub4_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub4_Proteasome, &Mdm2_P_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GSK3b = 500.000000;
float* dev_GSK3b = 0;
cudaStatus = hipMalloc(&dev_GSK3b, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GSK3b, &GSK3b, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GSK3b_p53 = 0.000000;
float* dev_GSK3b_p53 = 0;
cudaStatus = hipMalloc(&dev_GSK3b_p53, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GSK3b_p53, &GSK3b_p53, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GSK3b_p53_P = 0.000000;
float* dev_GSK3b_p53_P = 0;
cudaStatus = hipMalloc(&dev_GSK3b_p53_P, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GSK3b_p53_P, &GSK3b_p53_P, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Abeta = 0.000000;
float* dev_Abeta = 0;
cudaStatus = hipMalloc(&dev_Abeta, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Abeta, &Abeta, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AggAbeta_Proteasome = 0.000000;
float* dev_AggAbeta_Proteasome = 0;
cudaStatus = hipMalloc(&dev_AggAbeta_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AggAbeta_Proteasome, &AggAbeta_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaPlaque = 0.000000;
float* dev_AbetaPlaque = 0;
cudaStatus = hipMalloc(&dev_AbetaPlaque, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaPlaque, &AbetaPlaque, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Tau = 0.000000;
float* dev_Tau = 0;
cudaStatus = hipMalloc(&dev_Tau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Tau, &Tau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Tau_P1 = 0.000000;
float* dev_Tau_P1 = 0;
cudaStatus = hipMalloc(&dev_Tau_P1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Tau_P1, &Tau_P1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Tau_P2 = 0.000000;
float* dev_Tau_P2 = 0;
cudaStatus = hipMalloc(&dev_Tau_P2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Tau_P2, &Tau_P2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float MT_Tau = 100.000000;
float* dev_MT_Tau = 0;
cudaStatus = hipMalloc(&dev_MT_Tau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_MT_Tau, &MT_Tau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AggTau = 0.000000;
float* dev_AggTau = 0;
cudaStatus = hipMalloc(&dev_AggTau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AggTau, &AggTau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AggTau_Proteasome = 0.000000;
float* dev_AggTau_Proteasome = 0;
cudaStatus = hipMalloc(&dev_AggTau_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AggTau_Proteasome, &AggTau_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Proteasome_Tau = 0.000000;
float* dev_Proteasome_Tau = 0;
cudaStatus = hipMalloc(&dev_Proteasome_Tau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Proteasome_Tau, &Proteasome_Tau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float PP1 = 50.000000;
float* dev_PP1 = 0;
cudaStatus = hipMalloc(&dev_PP1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_PP1, &PP1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float NFT = 0.000000;
float* dev_NFT = 0;
cudaStatus = hipMalloc(&dev_NFT, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_NFT, &NFT, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ATP = 10000.000000;
float* dev_ATP = 0;
cudaStatus = hipMalloc(&dev_ATP, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ATP, &ATP, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ADP = 1000.000000;
float* dev_ADP = 0;
cudaStatus = hipMalloc(&dev_ADP, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ADP, &ADP, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AMP = 1000.000000;
float* dev_AMP = 0;
cudaStatus = hipMalloc(&dev_AMP, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AMP, &AMP, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaDimer = 0.000000;
float* dev_AbetaDimer = 0;
cudaStatus = hipMalloc(&dev_AbetaDimer, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaDimer, &AbetaDimer, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaPlaque_GliaA = 0.000000;
float* dev_AbetaPlaque_GliaA = 0;
cudaStatus = hipMalloc(&dev_AbetaPlaque_GliaA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaPlaque_GliaA, &AbetaPlaque_GliaA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaI = 100.000000;
float* dev_GliaI = 0;
cudaStatus = hipMalloc(&dev_GliaI, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaI, &GliaI, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaM1 = 0.000000;
float* dev_GliaM1 = 0;
cudaStatus = hipMalloc(&dev_GliaM1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaM1, &GliaM1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaM2 = 0.000000;
float* dev_GliaM2 = 0;
cudaStatus = hipMalloc(&dev_GliaM2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaM2, &GliaM2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaA = 0.000000;
float* dev_GliaA = 0;
cudaStatus = hipMalloc(&dev_GliaA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaA, &GliaA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float antiAb = 0.000000;
float* dev_antiAb = 0;
cudaStatus = hipMalloc(&dev_antiAb, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_antiAb, &antiAb, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Abeta_antiAb = 0.000000;
float* dev_Abeta_antiAb = 0;
cudaStatus = hipMalloc(&dev_Abeta_antiAb, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Abeta_antiAb, &Abeta_antiAb, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaDimer_antiAb = 0.000000;
float* dev_AbetaDimer_antiAb = 0;
cudaStatus = hipMalloc(&dev_AbetaDimer_antiAb, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaDimer_antiAb, &AbetaDimer_antiAb, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float degAbetaGlia = 0.000000;
float* dev_degAbetaGlia = 0;
cudaStatus = hipMalloc(&dev_degAbetaGlia, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_degAbetaGlia, &degAbetaGlia, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float disaggPlaque1 = 0.000000;
float* dev_disaggPlaque1 = 0;
cudaStatus = hipMalloc(&dev_disaggPlaque1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_disaggPlaque1, &disaggPlaque1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float disaggPlaque2 = 0.000000;
float* dev_disaggPlaque2 = 0;
cudaStatus = hipMalloc(&dev_disaggPlaque2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_disaggPlaque2, &disaggPlaque2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Source = 1.000000;
float* dev_Source = 0;
cudaStatus = hipMalloc(&dev_Source, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Source, &Source, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Sink = 1.000000;
float* dev_Sink = 0;
cudaStatus = hipMalloc(&dev_Sink, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Sink, &Sink, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
simulate<<<1, 112>>>(0.010000, 100000, 0, dev_Mdm2, 0, dev_p53, 0, dev_Mdm2_p53, 0, dev_Mdm2_mRNA, 0, dev_p53_mRNA, 0, dev_ATMA, 0, dev_ATMI, 0, dev_p53_P, 0, dev_Mdm2_P, 0, dev_IR, 0, dev_ROS, 0, dev_damDNA, 0, dev_E1, 0, dev_E2, 0, dev_E1_Ub, 0, dev_E2_Ub, 0, dev_Proteasome, 0, dev_Ub, 0, dev_p53DUB, 0, dev_Mdm2DUB, 0, dev_DUB, 0, dev_Mdm2_p53_Ub, 0, dev_Mdm2_p53_Ub2, 0, dev_Mdm2_p53_Ub3, 0, dev_Mdm2_p53_Ub4, 0, dev_Mdm2_P1_p53_Ub4, 0, dev_Mdm2_Ub, 0, dev_Mdm2_Ub2, 0, dev_Mdm2_Ub3, 0, dev_Mdm2_Ub4, 0, dev_Mdm2_P_Ub, 0, dev_Mdm2_P_Ub2, 0, dev_Mdm2_P_Ub3, 0, dev_Mdm2_P_Ub4, 0, dev_p53_Ub4_Proteasome, 0, dev_Mdm2_Ub4_Proteasome, 0, dev_Mdm2_P_Ub4_Proteasome, 0, dev_GSK3b, 0, dev_GSK3b_p53, 0, dev_GSK3b_p53_P, 0, dev_Abeta, 0, dev_AggAbeta_Proteasome, 0, dev_AbetaPlaque, 0, dev_Tau, 0, dev_Tau_P1, 0, dev_Tau_P2, 0, dev_MT_Tau, 0, dev_AggTau, 0, dev_AggTau_Proteasome, 0, dev_Proteasome_Tau, 0, dev_PP1, 0, dev_NFT, 0, dev_ATP, 0, dev_ADP, 0, dev_AMP, 0, dev_AbetaDimer, 0, dev_AbetaPlaque_GliaA, 0, dev_GliaI, 0, dev_GliaM1, 0, dev_GliaM2, 0, dev_GliaA, 0, dev_antiAb, 0, dev_Abeta_antiAb, 0, dev_AbetaDimer_antiAb, 0, dev_degAbetaGlia, 0, dev_disaggPlaque1, 0, dev_disaggPlaque2, 0, dev_Source, 0, dev_Sink);

cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess) {fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));goto Error;}

cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess) {fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);goto Error;}cudaStatus = hipMemcpy(&Mdm2, dev_Mdm2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53, dev_p53, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53, dev_Mdm2_p53, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_mRNA, dev_Mdm2_mRNA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53_mRNA, dev_p53_mRNA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ATMA, dev_ATMA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ATMI, dev_ATMI, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53_P, dev_p53_P, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P, dev_Mdm2_P, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&IR, dev_IR, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ROS, dev_ROS, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&damDNA, dev_damDNA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E1, dev_E1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E2, dev_E2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E1_Ub, dev_E1_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E2_Ub, dev_E2_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Proteasome, dev_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Ub, dev_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53DUB, dev_p53DUB, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2DUB, dev_Mdm2DUB, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&DUB, dev_DUB, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub, dev_Mdm2_p53_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub2, dev_Mdm2_p53_Ub2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub3, dev_Mdm2_p53_Ub3, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub4, dev_Mdm2_p53_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P1_p53_Ub4, dev_Mdm2_P1_p53_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub, dev_Mdm2_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub2, dev_Mdm2_Ub2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub3, dev_Mdm2_Ub3, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub4, dev_Mdm2_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub, dev_Mdm2_P_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub2, dev_Mdm2_P_Ub2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub3, dev_Mdm2_P_Ub3, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub4, dev_Mdm2_P_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53_Ub4_Proteasome, dev_p53_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub4_Proteasome, dev_Mdm2_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub4_Proteasome, dev_Mdm2_P_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GSK3b, dev_GSK3b, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GSK3b_p53, dev_GSK3b_p53, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GSK3b_p53_P, dev_GSK3b_p53_P, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Abeta, dev_Abeta, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AggAbeta_Proteasome, dev_AggAbeta_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaPlaque, dev_AbetaPlaque, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Tau, dev_Tau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Tau_P1, dev_Tau_P1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Tau_P2, dev_Tau_P2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&MT_Tau, dev_MT_Tau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AggTau, dev_AggTau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AggTau_Proteasome, dev_AggTau_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Proteasome_Tau, dev_Proteasome_Tau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&PP1, dev_PP1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&NFT, dev_NFT, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ATP, dev_ATP, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ADP, dev_ADP, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AMP, dev_AMP, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaDimer, dev_AbetaDimer, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaPlaque_GliaA, dev_AbetaPlaque_GliaA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaI, dev_GliaI, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaM1, dev_GliaM1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaM2, dev_GliaM2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaA, dev_GliaA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&antiAb, dev_antiAb, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Abeta_antiAb, dev_Abeta_antiAb, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaDimer_antiAb, dev_AbetaDimer_antiAb, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&degAbetaGlia, dev_degAbetaGlia, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&disaggPlaque1, dev_disaggPlaque1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&disaggPlaque2, dev_disaggPlaque2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Source, dev_Source, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Sink, dev_Sink, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess) {fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));goto Error;}

cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess) {fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);goto Error;}printf("Mdm2 = %f\n", Mdm2);
printf("p53 = %f\n", p53);
printf("Mdm2_p53 = %f\n", Mdm2_p53);
printf("Mdm2_mRNA = %f\n", Mdm2_mRNA);
printf("p53_mRNA = %f\n", p53_mRNA);
printf("ATMA = %f\n", ATMA);
printf("ATMI = %f\n", ATMI);
printf("p53_P = %f\n", p53_P);
printf("Mdm2_P = %f\n", Mdm2_P);
printf("IR = %f\n", IR);
printf("ROS = %f\n", ROS);
printf("damDNA = %f\n", damDNA);
printf("E1 = %f\n", E1);
printf("E2 = %f\n", E2);
printf("E1_Ub = %f\n", E1_Ub);
printf("E2_Ub = %f\n", E2_Ub);
printf("Proteasome = %f\n", Proteasome);
printf("Ub = %f\n", Ub);
printf("p53DUB = %f\n", p53DUB);
printf("Mdm2DUB = %f\n", Mdm2DUB);
printf("DUB = %f\n", DUB);
printf("Mdm2_p53_Ub = %f\n", Mdm2_p53_Ub);
printf("Mdm2_p53_Ub2 = %f\n", Mdm2_p53_Ub2);
printf("Mdm2_p53_Ub3 = %f\n", Mdm2_p53_Ub3);
printf("Mdm2_p53_Ub4 = %f\n", Mdm2_p53_Ub4);
printf("Mdm2_P1_p53_Ub4 = %f\n", Mdm2_P1_p53_Ub4);
printf("Mdm2_Ub = %f\n", Mdm2_Ub);
printf("Mdm2_Ub2 = %f\n", Mdm2_Ub2);
printf("Mdm2_Ub3 = %f\n", Mdm2_Ub3);
printf("Mdm2_Ub4 = %f\n", Mdm2_Ub4);
printf("Mdm2_P_Ub = %f\n", Mdm2_P_Ub);
printf("Mdm2_P_Ub2 = %f\n", Mdm2_P_Ub2);
printf("Mdm2_P_Ub3 = %f\n", Mdm2_P_Ub3);
printf("Mdm2_P_Ub4 = %f\n", Mdm2_P_Ub4);
printf("p53_Ub4_Proteasome = %f\n", p53_Ub4_Proteasome);
printf("Mdm2_Ub4_Proteasome = %f\n", Mdm2_Ub4_Proteasome);
printf("Mdm2_P_Ub4_Proteasome = %f\n", Mdm2_P_Ub4_Proteasome);
printf("GSK3b = %f\n", GSK3b);
printf("GSK3b_p53 = %f\n", GSK3b_p53);
printf("GSK3b_p53_P = %f\n", GSK3b_p53_P);
printf("Abeta = %f\n", Abeta);
printf("AggAbeta_Proteasome = %f\n", AggAbeta_Proteasome);
printf("AbetaPlaque = %f\n", AbetaPlaque);
printf("Tau = %f\n", Tau);
printf("Tau_P1 = %f\n", Tau_P1);
printf("Tau_P2 = %f\n", Tau_P2);
printf("MT_Tau = %f\n", MT_Tau);
printf("AggTau = %f\n", AggTau);
printf("AggTau_Proteasome = %f\n", AggTau_Proteasome);
printf("Proteasome_Tau = %f\n", Proteasome_Tau);
printf("PP1 = %f\n", PP1);
printf("NFT = %f\n", NFT);
printf("ATP = %f\n", ATP);
printf("ADP = %f\n", ADP);
printf("AMP = %f\n", AMP);
printf("AbetaDimer = %f\n", AbetaDimer);
printf("AbetaPlaque_GliaA = %f\n", AbetaPlaque_GliaA);
printf("GliaI = %f\n", GliaI);
printf("GliaM1 = %f\n", GliaM1);
printf("GliaM2 = %f\n", GliaM2);
printf("GliaA = %f\n", GliaA);
printf("antiAb = %f\n", antiAb);
printf("Abeta_antiAb = %f\n", Abeta_antiAb);
printf("AbetaDimer_antiAb = %f\n", AbetaDimer_antiAb);
printf("degAbetaGlia = %f\n", degAbetaGlia);
printf("disaggPlaque1 = %f\n", disaggPlaque1);
printf("disaggPlaque2 = %f\n", disaggPlaque2);
printf("Source = %f\n", Source);
printf("Sink = %f\n", Sink);
Error:
hipFree(dev_Mdm2);
hipFree(dev_p53);
hipFree(dev_Mdm2_p53);
hipFree(dev_Mdm2_mRNA);
hipFree(dev_p53_mRNA);
hipFree(dev_ATMA);
hipFree(dev_ATMI);
hipFree(dev_p53_P);
hipFree(dev_Mdm2_P);
hipFree(dev_IR);
hipFree(dev_ROS);
hipFree(dev_damDNA);
hipFree(dev_E1);
hipFree(dev_E2);
hipFree(dev_E1_Ub);
hipFree(dev_E2_Ub);
hipFree(dev_Proteasome);
hipFree(dev_Ub);
hipFree(dev_p53DUB);
hipFree(dev_Mdm2DUB);
hipFree(dev_DUB);
hipFree(dev_Mdm2_p53_Ub);
hipFree(dev_Mdm2_p53_Ub2);
hipFree(dev_Mdm2_p53_Ub3);
hipFree(dev_Mdm2_p53_Ub4);
hipFree(dev_Mdm2_P1_p53_Ub4);
hipFree(dev_Mdm2_Ub);
hipFree(dev_Mdm2_Ub2);
hipFree(dev_Mdm2_Ub3);
hipFree(dev_Mdm2_Ub4);
hipFree(dev_Mdm2_P_Ub);
hipFree(dev_Mdm2_P_Ub2);
hipFree(dev_Mdm2_P_Ub3);
hipFree(dev_Mdm2_P_Ub4);
hipFree(dev_p53_Ub4_Proteasome);
hipFree(dev_Mdm2_Ub4_Proteasome);
hipFree(dev_Mdm2_P_Ub4_Proteasome);
hipFree(dev_GSK3b);
hipFree(dev_GSK3b_p53);
hipFree(dev_GSK3b_p53_P);
hipFree(dev_Abeta);
hipFree(dev_AggAbeta_Proteasome);
hipFree(dev_AbetaPlaque);
hipFree(dev_Tau);
hipFree(dev_Tau_P1);
hipFree(dev_Tau_P2);
hipFree(dev_MT_Tau);
hipFree(dev_AggTau);
hipFree(dev_AggTau_Proteasome);
hipFree(dev_Proteasome_Tau);
hipFree(dev_PP1);
hipFree(dev_NFT);
hipFree(dev_ATP);
hipFree(dev_ADP);
hipFree(dev_AMP);
hipFree(dev_AbetaDimer);
hipFree(dev_AbetaPlaque_GliaA);
hipFree(dev_GliaI);
hipFree(dev_GliaM1);
hipFree(dev_GliaM2);
hipFree(dev_GliaA);
hipFree(dev_antiAb);
hipFree(dev_Abeta_antiAb);
hipFree(dev_AbetaDimer_antiAb);
hipFree(dev_degAbetaGlia);
hipFree(dev_disaggPlaque1);
hipFree(dev_disaggPlaque2);
hipFree(dev_Source);
hipFree(dev_Sink);

    return 0;
}