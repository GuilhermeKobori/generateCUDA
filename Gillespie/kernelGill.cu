#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define pow powf

#define SEED 23
#define Mdm2 species[0]
#define Mdm2_id 0
#define p53 species[1]
#define p53_id 1
#define Mdm2_p53 species[2]
#define Mdm2_p53_id 2
#define Mdm2_mRNA species[3]
#define Mdm2_mRNA_id 3
#define p53_mRNA species[4]
#define p53_mRNA_id 4
#define ATMA species[5]
#define ATMA_id 5
#define ATMI species[6]
#define ATMI_id 6
#define p53_P species[7]
#define p53_P_id 7
#define Mdm2_P species[8]
#define Mdm2_P_id 8
#define IR species[9]
#define IR_id 9
#define ROS species[10]
#define ROS_id 10
#define damDNA species[11]
#define damDNA_id 11
#define E1 species[12]
#define E1_id 12
#define E2 species[13]
#define E2_id 13
#define E1_Ub species[14]
#define E1_Ub_id 14
#define E2_Ub species[15]
#define E2_Ub_id 15
#define Proteasome species[16]
#define Proteasome_id 16
#define Ub species[17]
#define Ub_id 17
#define p53DUB species[18]
#define p53DUB_id 18
#define Mdm2DUB species[19]
#define Mdm2DUB_id 19
#define DUB species[20]
#define DUB_id 20
#define Mdm2_p53_Ub species[21]
#define Mdm2_p53_Ub_id 21
#define Mdm2_p53_Ub2 species[22]
#define Mdm2_p53_Ub2_id 22
#define Mdm2_p53_Ub3 species[23]
#define Mdm2_p53_Ub3_id 23
#define Mdm2_p53_Ub4 species[24]
#define Mdm2_p53_Ub4_id 24
#define Mdm2_P1_p53_Ub4 species[25]
#define Mdm2_P1_p53_Ub4_id 25
#define Mdm2_Ub species[26]
#define Mdm2_Ub_id 26
#define Mdm2_Ub2 species[27]
#define Mdm2_Ub2_id 27
#define Mdm2_Ub3 species[28]
#define Mdm2_Ub3_id 28
#define Mdm2_Ub4 species[29]
#define Mdm2_Ub4_id 29
#define Mdm2_P_Ub species[30]
#define Mdm2_P_Ub_id 30
#define Mdm2_P_Ub2 species[31]
#define Mdm2_P_Ub2_id 31
#define Mdm2_P_Ub3 species[32]
#define Mdm2_P_Ub3_id 32
#define Mdm2_P_Ub4 species[33]
#define Mdm2_P_Ub4_id 33
#define p53_Ub4_Proteasome species[34]
#define p53_Ub4_Proteasome_id 34
#define Mdm2_Ub4_Proteasome species[35]
#define Mdm2_Ub4_Proteasome_id 35
#define Mdm2_P_Ub4_Proteasome species[36]
#define Mdm2_P_Ub4_Proteasome_id 36
#define GSK3b species[37]
#define GSK3b_id 37
#define GSK3b_p53 species[38]
#define GSK3b_p53_id 38
#define GSK3b_p53_P species[39]
#define GSK3b_p53_P_id 39
#define Abeta species[40]
#define Abeta_id 40
#define AggAbeta_Proteasome species[41]
#define AggAbeta_Proteasome_id 41
#define AbetaPlaque species[42]
#define AbetaPlaque_id 42
#define Tau species[43]
#define Tau_id 43
#define Tau_P1 species[44]
#define Tau_P1_id 44
#define Tau_P2 species[45]
#define Tau_P2_id 45
#define MT_Tau species[46]
#define MT_Tau_id 46
#define AggTau species[47]
#define AggTau_id 47
#define AggTau_Proteasome species[48]
#define AggTau_Proteasome_id 48
#define Proteasome_Tau species[49]
#define Proteasome_Tau_id 49
#define PP1 species[50]
#define PP1_id 50
#define NFT species[51]
#define NFT_id 51
#define ATP species[52]
#define ATP_id 52
#define ADP species[53]
#define ADP_id 53
#define AMP species[54]
#define AMP_id 54
#define AbetaDimer species[55]
#define AbetaDimer_id 55
#define AbetaPlaque_GliaA species[56]
#define AbetaPlaque_GliaA_id 56
#define GliaI species[57]
#define GliaI_id 57
#define GliaM1 species[58]
#define GliaM1_id 58
#define GliaM2 species[59]
#define GliaM2_id 59
#define GliaA species[60]
#define GliaA_id 60
#define antiAb species[61]
#define antiAb_id 61
#define Abeta_antiAb species[62]
#define Abeta_antiAb_id 62
#define AbetaDimer_antiAb species[63]
#define AbetaDimer_antiAb_id 63
#define degAbetaGlia species[64]
#define degAbetaGlia_id 64
#define disaggPlaque1 species[65]
#define disaggPlaque1_id 65
#define disaggPlaque2 species[66]
#define disaggPlaque2_id 66
#define Source species[67]
#define Source_id 67
#define Sink species[68]
#define Sink_id 68
#define cell 1.0000000000
#define ksynp53mRNA 0.0010000000
#define kdegp53mRNA 0.0001000000
#define ksynMdm2mRNA 0.0005000000
#define kdegMdm2mRNA 0.0005000000
#define ksynMdm2mRNAGSK3bp53 0.0007000000
#define ksynp53 0.0070000000
#define kdegp53 0.0050000000
#define kbinMdm2p53 0.0011550000
#define krelMdm2p53 0.0000115500
#define kbinGSK3bp53 0.0000020000
#define krelGSK3bp53 0.0020000000
#define ksynMdm2 0.0004950000
#define kdegMdm2 0.0100000000
#define kbinE1Ub 0.0002000000
#define kbinE2Ub 0.0010000000
#define kp53Ub 0.0000500000
#define kp53PolyUb 0.0100000000
#define kbinProt 0.0000020000
#define kactDUBp53 0.0000001000
#define kactDUBProtp53 0.0001000000
#define kactDUBMdm2 0.0000001000
#define kMdm2Ub 0.0000045600
#define kMdm2PUb 0.0000068400
#define kMdm2PolyUb 0.0045600000
#define kdam 0.0800000000
#define krepair 0.0000200000
#define kactATM 0.0001000000
#define kinactATM 0.0005000000
#define kphosp53 0.0002000000
#define kdephosp53 0.5000000000
#define kphosMdm2 2.0000000000
#define kdephosMdm2 0.5000000000
#define kphosMdm2GSK3b 0.0050000000
#define kphosMdm2GSK3bp53 0.5000000000
#define kphospTauGSK3bp53 0.1000000000
#define kphospTauGSK3b 0.0002000000
#define kdephospTau 0.0100000000
#define kbinMTTau 0.1000000000
#define krelMTTau 0.0001000000
#define ksynTau 0.0000800000
#define kbinTauProt 0.0000001925
#define kdegTau20SProt 0.0100000000
#define kaggTau 0.0000000100
#define kaggTauP1 0.0000000100
#define kaggTauP2 0.0000001000
#define ktangfor 0.0010000000
#define kinhibprot 0.0000001000
#define ksynp53mRNAAbeta 0.0000100000
#define kdamROS 0.0000100000
#define kgenROSAbeta 0.0000200000
#define kgenROSPlaque 0.0000100000
#define kgenROSGlia 0.0000100000
#define kproteff 1.0000000000
#define kremROS 0.0000700000
#define kprodAbeta 0.0000186000
#define kprodAbeta2 0.0000186000
#define kdegAbeta 0.0000150000
#define kaggAbeta 0.0000030000
#define kdisaggAbeta 0.0000010000
#define kdisaggAbeta1 0.0002000000
#define kdisaggAbeta2 0.0000010000
#define kdegAbetaGlia 0.0050000000
#define kpf 0.2000000000
#define kpg 0.1500000000
#define kpghalf 10.0000000000
#define kactglia1 0.0000006000
#define kactglia2 0.0000006000
#define kinactglia1 0.0000050000
#define kinactglia2 0.0000050000
#define kbinAbetaGlia 0.0000100000
#define krelAbetaGlia 0.0000500000
#define kdegAntiAb 0.0000027500
#define kbinAbantiAb 0.0000010000

__global__
void simulate(int numberOfExecutions, float* output, hiprandState *state, float step, float endTime, float segmentSize, float* species_global) {
	int reaction, stepCount = 0;
	int indexMin, indexMax;
	float time = numberOfExecutions * segmentSize;
	float sum_p, timeStep, random;
	float cummulative_p[112];
	int triggerEvent0 = 0;
	if (time >= 345600) { triggerEvent0 = 1; }
	float species[69];
	if (numberOfExecutions == 0) {
		species[0] = species_global[0];
	}
	else {
		species[0] = species_global[Mdm2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[1] = species_global[1];
	}
	else {
		species[1] = species_global[p53_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[2] = species_global[2];
	}
	else {
		species[2] = species_global[Mdm2_p53_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[3] = species_global[3];
	}
	else {
		species[3] = species_global[Mdm2_mRNA_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[4] = species_global[4];
	}
	else {
		species[4] = species_global[p53_mRNA_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[5] = species_global[5];
	}
	else {
		species[5] = species_global[ATMA_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[6] = species_global[6];
	}
	else {
		species[6] = species_global[ATMI_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[7] = species_global[7];
	}
	else {
		species[7] = species_global[p53_P_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[8] = species_global[8];
	}
	else {
		species[8] = species_global[Mdm2_P_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[9] = species_global[9];
	}
	else {
		species[9] = species_global[IR_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[10] = species_global[10];
	}
	else {
		species[10] = species_global[ROS_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[11] = species_global[11];
	}
	else {
		species[11] = species_global[damDNA_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[12] = species_global[12];
	}
	else {
		species[12] = species_global[E1_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[13] = species_global[13];
	}
	else {
		species[13] = species_global[E2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[14] = species_global[14];
	}
	else {
		species[14] = species_global[E1_Ub_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[15] = species_global[15];
	}
	else {
		species[15] = species_global[E2_Ub_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[16] = species_global[16];
	}
	else {
		species[16] = species_global[Proteasome_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[17] = species_global[17];
	}
	else {
		species[17] = species_global[Ub_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[18] = species_global[18];
	}
	else {
		species[18] = species_global[p53DUB_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[19] = species_global[19];
	}
	else {
		species[19] = species_global[Mdm2DUB_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[20] = species_global[20];
	}
	else {
		species[20] = species_global[DUB_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[21] = species_global[21];
	}
	else {
		species[21] = species_global[Mdm2_p53_Ub_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[22] = species_global[22];
	}
	else {
		species[22] = species_global[Mdm2_p53_Ub2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[23] = species_global[23];
	}
	else {
		species[23] = species_global[Mdm2_p53_Ub3_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[24] = species_global[24];
	}
	else {
		species[24] = species_global[Mdm2_p53_Ub4_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[25] = species_global[25];
	}
	else {
		species[25] = species_global[Mdm2_P1_p53_Ub4_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[26] = species_global[26];
	}
	else {
		species[26] = species_global[Mdm2_Ub_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[27] = species_global[27];
	}
	else {
		species[27] = species_global[Mdm2_Ub2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[28] = species_global[28];
	}
	else {
		species[28] = species_global[Mdm2_Ub3_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[29] = species_global[29];
	}
	else {
		species[29] = species_global[Mdm2_Ub4_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[30] = species_global[30];
	}
	else {
		species[30] = species_global[Mdm2_P_Ub_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[31] = species_global[31];
	}
	else {
		species[31] = species_global[Mdm2_P_Ub2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[32] = species_global[32];
	}
	else {
		species[32] = species_global[Mdm2_P_Ub3_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[33] = species_global[33];
	}
	else {
		species[33] = species_global[Mdm2_P_Ub4_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[34] = species_global[34];
	}
	else {
		species[34] = species_global[p53_Ub4_Proteasome_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[35] = species_global[35];
	}
	else {
		species[35] = species_global[Mdm2_Ub4_Proteasome_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[36] = species_global[36];
	}
	else {
		species[36] = species_global[Mdm2_P_Ub4_Proteasome_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[37] = species_global[37];
	}
	else {
		species[37] = species_global[GSK3b_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[38] = species_global[38];
	}
	else {
		species[38] = species_global[GSK3b_p53_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[39] = species_global[39];
	}
	else {
		species[39] = species_global[GSK3b_p53_P_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[40] = species_global[40];
	}
	else {
		species[40] = species_global[Abeta_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[41] = species_global[41];
	}
	else {
		species[41] = species_global[AggAbeta_Proteasome_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[42] = species_global[42];
	}
	else {
		species[42] = species_global[AbetaPlaque_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[43] = species_global[43];
	}
	else {
		species[43] = species_global[Tau_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[44] = species_global[44];
	}
	else {
		species[44] = species_global[Tau_P1_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[45] = species_global[45];
	}
	else {
		species[45] = species_global[Tau_P2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[46] = species_global[46];
	}
	else {
		species[46] = species_global[MT_Tau_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[47] = species_global[47];
	}
	else {
		species[47] = species_global[AggTau_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[48] = species_global[48];
	}
	else {
		species[48] = species_global[AggTau_Proteasome_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[49] = species_global[49];
	}
	else {
		species[49] = species_global[Proteasome_Tau_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[50] = species_global[50];
	}
	else {
		species[50] = species_global[PP1_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[51] = species_global[51];
	}
	else {
		species[51] = species_global[NFT_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[52] = species_global[52];
	}
	else {
		species[52] = species_global[ATP_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[53] = species_global[53];
	}
	else {
		species[53] = species_global[ADP_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[54] = species_global[54];
	}
	else {
		species[54] = species_global[AMP_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[55] = species_global[55];
	}
	else {
		species[55] = species_global[AbetaDimer_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[56] = species_global[56];
	}
	else {
		species[56] = species_global[AbetaPlaque_GliaA_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[57] = species_global[57];
	}
	else {
		species[57] = species_global[GliaI_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[58] = species_global[58];
	}
	else {
		species[58] = species_global[GliaM1_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[59] = species_global[59];
	}
	else {
		species[59] = species_global[GliaM2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[60] = species_global[60];
	}
	else {
		species[60] = species_global[GliaA_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[61] = species_global[61];
	}
	else {
		species[61] = species_global[antiAb_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[62] = species_global[62];
	}
	else {
		species[62] = species_global[Abeta_antiAb_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[63] = species_global[63];
	}
	else {
		species[63] = species_global[AbetaDimer_antiAb_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[64] = species_global[64];
	}
	else {
		species[64] = species_global[degAbetaGlia_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[65] = species_global[65];
	}
	else {
		species[65] = species_global[disaggPlaque1_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[66] = species_global[66];
	}
	else {
		species[66] = species_global[disaggPlaque2_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[67] = species_global[67];
	}
	else {
		species[67] = species_global[Source_id * 32 + threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[68] = species_global[68];
	}
	else {
		species[68] = species_global[Sink_id * 32 + threadIdx.x];
	}
	int reactionsSpecies[112][5];
	int reactionsValues[112][5];
	for (int i = 0; i < 112; i++) {
		for (int j = 0; j < 5; j++) {
			reactionsSpecies[i][j] = -1;
			reactionsValues[i][j] = 0;
		}
	}
	reactionsSpecies[0][0] = p53_mRNA_id;
	reactionsValues[0][0] = 1.0000000000;
	reactionsSpecies[1][0] = p53_mRNA_id;
	reactionsValues[1][0] = -1.0000000000;
	reactionsSpecies[2][0] = Mdm2_mRNA_id;
	reactionsValues[2][0] = -1.0000000000;
	reactionsSpecies[2][1] = Mdm2_mRNA_id;
	reactionsValues[2][1] = 1.0000000000;
	reactionsSpecies[2][2] = Mdm2_id;
	reactionsValues[2][2] = 1.0000000000;
	reactionsSpecies[3][0] = p53_id;
	reactionsValues[3][0] = -1.0000000000;
	reactionsSpecies[3][1] = p53_id;
	reactionsValues[3][1] = 1.0000000000;
	reactionsSpecies[3][2] = Mdm2_mRNA_id;
	reactionsValues[3][2] = 1.0000000000;
	reactionsSpecies[4][0] = p53_P_id;
	reactionsValues[4][0] = -1.0000000000;
	reactionsSpecies[4][1] = p53_P_id;
	reactionsValues[4][1] = 1.0000000000;
	reactionsSpecies[4][2] = Mdm2_mRNA_id;
	reactionsValues[4][2] = 1.0000000000;
	reactionsSpecies[5][0] = GSK3b_p53_id;
	reactionsValues[5][0] = -1.0000000000;
	reactionsSpecies[5][1] = GSK3b_p53_id;
	reactionsValues[5][1] = 1.0000000000;
	reactionsSpecies[5][2] = Mdm2_mRNA_id;
	reactionsValues[5][2] = 1.0000000000;
	reactionsSpecies[6][0] = GSK3b_p53_P_id;
	reactionsValues[6][0] = -1.0000000000;
	reactionsSpecies[6][1] = GSK3b_p53_P_id;
	reactionsValues[6][1] = 1.0000000000;
	reactionsSpecies[6][2] = Mdm2_mRNA_id;
	reactionsValues[6][2] = 1.0000000000;
	reactionsSpecies[7][0] = Mdm2_mRNA_id;
	reactionsValues[7][0] = -1.0000000000;
	reactionsSpecies[8][0] = p53_id;
	reactionsValues[8][0] = -1.0000000000;
	reactionsSpecies[8][1] = Mdm2_id;
	reactionsValues[8][1] = -1.0000000000;
	reactionsSpecies[8][2] = Mdm2_p53_id;
	reactionsValues[8][2] = 1.0000000000;
	reactionsSpecies[9][0] = Mdm2_p53_id;
	reactionsValues[9][0] = -1.0000000000;
	reactionsSpecies[9][1] = p53_id;
	reactionsValues[9][1] = 1.0000000000;
	reactionsSpecies[9][2] = Mdm2_id;
	reactionsValues[9][2] = 1.0000000000;
	reactionsSpecies[10][0] = GSK3b_id;
	reactionsValues[10][0] = -1.0000000000;
	reactionsSpecies[10][1] = p53_id;
	reactionsValues[10][1] = -1.0000000000;
	reactionsSpecies[10][2] = GSK3b_p53_id;
	reactionsValues[10][2] = 1.0000000000;
	reactionsSpecies[11][0] = GSK3b_p53_id;
	reactionsValues[11][0] = -1.0000000000;
	reactionsSpecies[11][1] = GSK3b_id;
	reactionsValues[11][1] = 1.0000000000;
	reactionsSpecies[11][2] = p53_id;
	reactionsValues[11][2] = 1.0000000000;
	reactionsSpecies[12][0] = GSK3b_id;
	reactionsValues[12][0] = -1.0000000000;
	reactionsSpecies[12][1] = p53_P_id;
	reactionsValues[12][1] = -1.0000000000;
	reactionsSpecies[12][2] = GSK3b_p53_P_id;
	reactionsValues[12][2] = 1.0000000000;
	reactionsSpecies[13][0] = GSK3b_p53_P_id;
	reactionsValues[13][0] = -1.0000000000;
	reactionsSpecies[13][1] = GSK3b_id;
	reactionsValues[13][1] = 1.0000000000;
	reactionsSpecies[13][2] = p53_P_id;
	reactionsValues[13][2] = 1.0000000000;
	reactionsSpecies[14][0] = E1_id;
	reactionsValues[14][0] = -1.0000000000;
	reactionsSpecies[14][1] = Ub_id;
	reactionsValues[14][1] = -1.0000000000;
	reactionsSpecies[14][2] = E1_Ub_id;
	reactionsValues[14][2] = 1.0000000000;
	reactionsSpecies[15][0] = E2_id;
	reactionsValues[15][0] = -1.0000000000;
	reactionsSpecies[15][1] = E1_Ub_id;
	reactionsValues[15][1] = -1.0000000000;
	reactionsSpecies[15][2] = E2_Ub_id;
	reactionsValues[15][2] = 1.0000000000;
	reactionsSpecies[15][3] = E1_id;
	reactionsValues[15][3] = 1.0000000000;
	reactionsSpecies[16][0] = Mdm2_id;
	reactionsValues[16][0] = -1.0000000000;
	reactionsSpecies[16][1] = E2_Ub_id;
	reactionsValues[16][1] = -1.0000000000;
	reactionsSpecies[16][2] = Mdm2_Ub_id;
	reactionsValues[16][2] = 1.0000000000;
	reactionsSpecies[16][3] = E2_id;
	reactionsValues[16][3] = 1.0000000000;
	reactionsSpecies[17][0] = Mdm2_Ub_id;
	reactionsValues[17][0] = -1.0000000000;
	reactionsSpecies[17][1] = E2_Ub_id;
	reactionsValues[17][1] = -1.0000000000;
	reactionsSpecies[17][2] = Mdm2_Ub2_id;
	reactionsValues[17][2] = 1.0000000000;
	reactionsSpecies[17][3] = E2_id;
	reactionsValues[17][3] = 1.0000000000;
	reactionsSpecies[18][0] = Mdm2_Ub2_id;
	reactionsValues[18][0] = -1.0000000000;
	reactionsSpecies[18][1] = E2_Ub_id;
	reactionsValues[18][1] = -1.0000000000;
	reactionsSpecies[18][2] = Mdm2_Ub3_id;
	reactionsValues[18][2] = 1.0000000000;
	reactionsSpecies[18][3] = E2_id;
	reactionsValues[18][3] = 1.0000000000;
	reactionsSpecies[19][0] = Mdm2_Ub3_id;
	reactionsValues[19][0] = -1.0000000000;
	reactionsSpecies[19][1] = E2_Ub_id;
	reactionsValues[19][1] = -1.0000000000;
	reactionsSpecies[19][2] = Mdm2_Ub4_id;
	reactionsValues[19][2] = 1.0000000000;
	reactionsSpecies[19][3] = E2_id;
	reactionsValues[19][3] = 1.0000000000;
	reactionsSpecies[20][0] = Mdm2_Ub4_id;
	reactionsValues[20][0] = -1.0000000000;
	reactionsSpecies[20][1] = Mdm2DUB_id;
	reactionsValues[20][1] = -1.0000000000;
	reactionsSpecies[20][2] = Mdm2_Ub3_id;
	reactionsValues[20][2] = 1.0000000000;
	reactionsSpecies[20][3] = Mdm2DUB_id;
	reactionsValues[20][3] = 1.0000000000;
	reactionsSpecies[20][4] = Ub_id;
	reactionsValues[20][4] = 1.0000000000;
	reactionsSpecies[21][0] = Mdm2_Ub3_id;
	reactionsValues[21][0] = -1.0000000000;
	reactionsSpecies[21][1] = Mdm2DUB_id;
	reactionsValues[21][1] = -1.0000000000;
	reactionsSpecies[21][2] = Mdm2_Ub2_id;
	reactionsValues[21][2] = 1.0000000000;
	reactionsSpecies[21][3] = Mdm2DUB_id;
	reactionsValues[21][3] = 1.0000000000;
	reactionsSpecies[21][4] = Ub_id;
	reactionsValues[21][4] = 1.0000000000;
	reactionsSpecies[22][0] = Mdm2_Ub2_id;
	reactionsValues[22][0] = -1.0000000000;
	reactionsSpecies[22][1] = Mdm2DUB_id;
	reactionsValues[22][1] = -1.0000000000;
	reactionsSpecies[22][2] = Mdm2_Ub_id;
	reactionsValues[22][2] = 1.0000000000;
	reactionsSpecies[22][3] = Mdm2DUB_id;
	reactionsValues[22][3] = 1.0000000000;
	reactionsSpecies[22][4] = Ub_id;
	reactionsValues[22][4] = 1.0000000000;
	reactionsSpecies[23][0] = Mdm2_Ub_id;
	reactionsValues[23][0] = -1.0000000000;
	reactionsSpecies[23][1] = Mdm2DUB_id;
	reactionsValues[23][1] = -1.0000000000;
	reactionsSpecies[23][2] = Mdm2_id;
	reactionsValues[23][2] = 1.0000000000;
	reactionsSpecies[23][3] = Mdm2DUB_id;
	reactionsValues[23][3] = 1.0000000000;
	reactionsSpecies[23][4] = Ub_id;
	reactionsValues[23][4] = 1.0000000000;
	reactionsSpecies[24][0] = Mdm2_Ub4_id;
	reactionsValues[24][0] = -1.0000000000;
	reactionsSpecies[24][1] = Proteasome_id;
	reactionsValues[24][1] = -1.0000000000;
	reactionsSpecies[24][2] = Mdm2_Ub4_Proteasome_id;
	reactionsValues[24][2] = 1.0000000000;
	reactionsSpecies[25][0] = Mdm2_Ub4_Proteasome_id;
	reactionsValues[25][0] = -1.0000000000;
	reactionsSpecies[25][1] = Proteasome_id;
	reactionsValues[25][1] = 1.0000000000;
	reactionsSpecies[25][2] = Ub_id;
	reactionsValues[25][2] = 4.0000000000;
	reactionsSpecies[26][0] = p53_mRNA_id;
	reactionsValues[26][0] = -1.0000000000;
	reactionsSpecies[26][1] = p53_id;
	reactionsValues[26][1] = 1.0000000000;
	reactionsSpecies[26][2] = p53_mRNA_id;
	reactionsValues[26][2] = 1.0000000000;
	reactionsSpecies[27][0] = E2_Ub_id;
	reactionsValues[27][0] = -1.0000000000;
	reactionsSpecies[27][1] = Mdm2_p53_id;
	reactionsValues[27][1] = -1.0000000000;
	reactionsSpecies[27][2] = Mdm2_p53_Ub_id;
	reactionsValues[27][2] = 1.0000000000;
	reactionsSpecies[27][3] = E2_id;
	reactionsValues[27][3] = 1.0000000000;
	reactionsSpecies[28][0] = Mdm2_p53_Ub_id;
	reactionsValues[28][0] = -1.0000000000;
	reactionsSpecies[28][1] = E2_Ub_id;
	reactionsValues[28][1] = -1.0000000000;
	reactionsSpecies[28][2] = Mdm2_p53_Ub2_id;
	reactionsValues[28][2] = 1.0000000000;
	reactionsSpecies[28][3] = E2_id;
	reactionsValues[28][3] = 1.0000000000;
	reactionsSpecies[29][0] = Mdm2_p53_Ub2_id;
	reactionsValues[29][0] = -1.0000000000;
	reactionsSpecies[29][1] = E2_Ub_id;
	reactionsValues[29][1] = -1.0000000000;
	reactionsSpecies[29][2] = Mdm2_p53_Ub3_id;
	reactionsValues[29][2] = 1.0000000000;
	reactionsSpecies[29][3] = E2_id;
	reactionsValues[29][3] = 1.0000000000;
	reactionsSpecies[30][0] = Mdm2_p53_Ub3_id;
	reactionsValues[30][0] = -1.0000000000;
	reactionsSpecies[30][1] = E2_Ub_id;
	reactionsValues[30][1] = -1.0000000000;
	reactionsSpecies[30][2] = Mdm2_p53_Ub4_id;
	reactionsValues[30][2] = 1.0000000000;
	reactionsSpecies[30][3] = E2_id;
	reactionsValues[30][3] = 1.0000000000;
	reactionsSpecies[31][0] = Mdm2_p53_Ub4_id;
	reactionsValues[31][0] = -1.0000000000;
	reactionsSpecies[31][1] = p53DUB_id;
	reactionsValues[31][1] = -1.0000000000;
	reactionsSpecies[31][2] = Mdm2_p53_Ub3_id;
	reactionsValues[31][2] = 1.0000000000;
	reactionsSpecies[31][3] = p53DUB_id;
	reactionsValues[31][3] = 1.0000000000;
	reactionsSpecies[31][4] = Ub_id;
	reactionsValues[31][4] = 1.0000000000;
	reactionsSpecies[32][0] = Mdm2_p53_Ub3_id;
	reactionsValues[32][0] = -1.0000000000;
	reactionsSpecies[32][1] = p53DUB_id;
	reactionsValues[32][1] = -1.0000000000;
	reactionsSpecies[32][2] = Mdm2_p53_Ub2_id;
	reactionsValues[32][2] = 1.0000000000;
	reactionsSpecies[32][3] = p53DUB_id;
	reactionsValues[32][3] = 1.0000000000;
	reactionsSpecies[32][4] = Ub_id;
	reactionsValues[32][4] = 1.0000000000;
	reactionsSpecies[33][0] = Mdm2_p53_Ub2_id;
	reactionsValues[33][0] = -1.0000000000;
	reactionsSpecies[33][1] = p53DUB_id;
	reactionsValues[33][1] = -1.0000000000;
	reactionsSpecies[33][2] = Mdm2_p53_Ub_id;
	reactionsValues[33][2] = 1.0000000000;
	reactionsSpecies[33][3] = p53DUB_id;
	reactionsValues[33][3] = 1.0000000000;
	reactionsSpecies[33][4] = Ub_id;
	reactionsValues[33][4] = 1.0000000000;
	reactionsSpecies[34][0] = Mdm2_p53_Ub_id;
	reactionsValues[34][0] = -1.0000000000;
	reactionsSpecies[34][1] = p53DUB_id;
	reactionsValues[34][1] = -1.0000000000;
	reactionsSpecies[34][2] = Mdm2_p53_id;
	reactionsValues[34][2] = 1.0000000000;
	reactionsSpecies[34][3] = p53DUB_id;
	reactionsValues[34][3] = 1.0000000000;
	reactionsSpecies[34][4] = Ub_id;
	reactionsValues[34][4] = 1.0000000000;
	reactionsSpecies[35][0] = Mdm2_p53_Ub4_id;
	reactionsValues[35][0] = -1.0000000000;
	reactionsSpecies[35][1] = GSK3b_id;
	reactionsValues[35][1] = -1.0000000000;
	reactionsSpecies[35][2] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[35][2] = 1.0000000000;
	reactionsSpecies[35][3] = GSK3b_id;
	reactionsValues[35][3] = 1.0000000000;
	reactionsSpecies[36][0] = Mdm2_p53_Ub4_id;
	reactionsValues[36][0] = -1.0000000000;
	reactionsSpecies[36][1] = GSK3b_p53_id;
	reactionsValues[36][1] = -1.0000000000;
	reactionsSpecies[36][2] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[36][2] = 1.0000000000;
	reactionsSpecies[36][3] = GSK3b_p53_id;
	reactionsValues[36][3] = 1.0000000000;
	reactionsSpecies[37][0] = Mdm2_p53_Ub4_id;
	reactionsValues[37][0] = -1.0000000000;
	reactionsSpecies[37][1] = GSK3b_p53_P_id;
	reactionsValues[37][1] = -1.0000000000;
	reactionsSpecies[37][2] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[37][2] = 1.0000000000;
	reactionsSpecies[37][3] = GSK3b_p53_P_id;
	reactionsValues[37][3] = 1.0000000000;
	reactionsSpecies[38][0] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[38][0] = -1.0000000000;
	reactionsSpecies[38][1] = Proteasome_id;
	reactionsValues[38][1] = -1.0000000000;
	reactionsSpecies[38][2] = p53_Ub4_Proteasome_id;
	reactionsValues[38][2] = 1.0000000000;
	reactionsSpecies[38][3] = Mdm2_id;
	reactionsValues[38][3] = 1.0000000000;
	reactionsSpecies[39][0] = p53_Ub4_Proteasome_id;
	reactionsValues[39][0] = -1.0000000000;
	reactionsSpecies[39][1] = Ub_id;
	reactionsValues[39][1] = 4.0000000000;
	reactionsSpecies[39][2] = Proteasome_id;
	reactionsValues[39][2] = 1.0000000000;
	reactionsSpecies[40][0] = Tau_id;
	reactionsValues[40][0] = -1.0000000000;
	reactionsSpecies[40][1] = MT_Tau_id;
	reactionsValues[40][1] = 1.0000000000;
	reactionsSpecies[41][0] = MT_Tau_id;
	reactionsValues[41][0] = -1.0000000000;
	reactionsSpecies[41][1] = Tau_id;
	reactionsValues[41][1] = 1.0000000000;
	reactionsSpecies[42][0] = GSK3b_p53_id;
	reactionsValues[42][0] = -1.0000000000;
	reactionsSpecies[42][1] = Tau_id;
	reactionsValues[42][1] = -1.0000000000;
	reactionsSpecies[42][2] = GSK3b_p53_id;
	reactionsValues[42][2] = 1.0000000000;
	reactionsSpecies[42][3] = Tau_P1_id;
	reactionsValues[42][3] = 1.0000000000;
	reactionsSpecies[43][0] = GSK3b_p53_id;
	reactionsValues[43][0] = -1.0000000000;
	reactionsSpecies[43][1] = Tau_P1_id;
	reactionsValues[43][1] = -1.0000000000;
	reactionsSpecies[43][2] = GSK3b_p53_id;
	reactionsValues[43][2] = 1.0000000000;
	reactionsSpecies[43][3] = Tau_P2_id;
	reactionsValues[43][3] = 1.0000000000;
	reactionsSpecies[44][0] = GSK3b_p53_P_id;
	reactionsValues[44][0] = -1.0000000000;
	reactionsSpecies[44][1] = Tau_id;
	reactionsValues[44][1] = -1.0000000000;
	reactionsSpecies[44][2] = GSK3b_p53_P_id;
	reactionsValues[44][2] = 1.0000000000;
	reactionsSpecies[44][3] = Tau_P1_id;
	reactionsValues[44][3] = 1.0000000000;
	reactionsSpecies[45][0] = GSK3b_p53_P_id;
	reactionsValues[45][0] = -1.0000000000;
	reactionsSpecies[45][1] = Tau_P1_id;
	reactionsValues[45][1] = -1.0000000000;
	reactionsSpecies[45][2] = GSK3b_p53_P_id;
	reactionsValues[45][2] = 1.0000000000;
	reactionsSpecies[45][3] = Tau_P2_id;
	reactionsValues[45][3] = 1.0000000000;
	reactionsSpecies[46][0] = GSK3b_id;
	reactionsValues[46][0] = -1.0000000000;
	reactionsSpecies[46][1] = Tau_id;
	reactionsValues[46][1] = -1.0000000000;
	reactionsSpecies[46][2] = GSK3b_id;
	reactionsValues[46][2] = 1.0000000000;
	reactionsSpecies[46][3] = Tau_P1_id;
	reactionsValues[46][3] = 1.0000000000;
	reactionsSpecies[47][0] = GSK3b_id;
	reactionsValues[47][0] = -1.0000000000;
	reactionsSpecies[47][1] = Tau_P1_id;
	reactionsValues[47][1] = -1.0000000000;
	reactionsSpecies[47][2] = GSK3b_id;
	reactionsValues[47][2] = 1.0000000000;
	reactionsSpecies[47][3] = Tau_P2_id;
	reactionsValues[47][3] = 1.0000000000;
	reactionsSpecies[48][0] = Tau_P2_id;
	reactionsValues[48][0] = -1.0000000000;
	reactionsSpecies[48][1] = PP1_id;
	reactionsValues[48][1] = -1.0000000000;
	reactionsSpecies[48][2] = Tau_P1_id;
	reactionsValues[48][2] = 1.0000000000;
	reactionsSpecies[48][3] = PP1_id;
	reactionsValues[48][3] = 1.0000000000;
	reactionsSpecies[49][0] = Tau_P1_id;
	reactionsValues[49][0] = -1.0000000000;
	reactionsSpecies[49][1] = PP1_id;
	reactionsValues[49][1] = -1.0000000000;
	reactionsSpecies[49][2] = Tau_id;
	reactionsValues[49][2] = 1.0000000000;
	reactionsSpecies[49][3] = PP1_id;
	reactionsValues[49][3] = 1.0000000000;
	reactionsSpecies[50][0] = Tau_P1_id;
	reactionsValues[50][0] = -2.0000000000;
	reactionsSpecies[50][1] = AggTau_id;
	reactionsValues[50][1] = 2.0000000000;
	reactionsSpecies[51][0] = Tau_P1_id;
	reactionsValues[51][0] = -1.0000000000;
	reactionsSpecies[51][1] = AggTau_id;
	reactionsValues[51][1] = -1.0000000000;
	reactionsSpecies[51][2] = AggTau_id;
	reactionsValues[51][2] = 2.0000000000;
	reactionsSpecies[52][0] = Tau_P2_id;
	reactionsValues[52][0] = -2.0000000000;
	reactionsSpecies[52][1] = AggTau_id;
	reactionsValues[52][1] = 2.0000000000;
	reactionsSpecies[53][0] = Tau_P2_id;
	reactionsValues[53][0] = -1.0000000000;
	reactionsSpecies[53][1] = AggTau_id;
	reactionsValues[53][1] = -1.0000000000;
	reactionsSpecies[53][2] = AggTau_id;
	reactionsValues[53][2] = 2.0000000000;
	reactionsSpecies[54][0] = Tau_id;
	reactionsValues[54][0] = -2.0000000000;
	reactionsSpecies[54][1] = AggTau_id;
	reactionsValues[54][1] = 2.0000000000;
	reactionsSpecies[55][0] = Tau_id;
	reactionsValues[55][0] = -1.0000000000;
	reactionsSpecies[55][1] = AggTau_id;
	reactionsValues[55][1] = -1.0000000000;
	reactionsSpecies[55][2] = AggTau_id;
	reactionsValues[55][2] = 2.0000000000;
	reactionsSpecies[56][0] = AggTau_id;
	reactionsValues[56][0] = -2.0000000000;
	reactionsSpecies[56][1] = NFT_id;
	reactionsValues[56][1] = 2.0000000000;
	reactionsSpecies[57][0] = AggTau_id;
	reactionsValues[57][0] = -1.0000000000;
	reactionsSpecies[57][1] = NFT_id;
	reactionsValues[57][1] = -1.0000000000;
	reactionsSpecies[57][2] = NFT_id;
	reactionsValues[57][2] = 2.0000000000;
	reactionsSpecies[58][0] = AggTau_id;
	reactionsValues[58][0] = -1.0000000000;
	reactionsSpecies[58][1] = Proteasome_id;
	reactionsValues[58][1] = -1.0000000000;
	reactionsSpecies[58][2] = AggTau_Proteasome_id;
	reactionsValues[58][2] = 1.0000000000;
	reactionsSpecies[59][0] = Abeta_id;
	reactionsValues[59][0] = 1.0000000000;
	reactionsSpecies[60][0] = GSK3b_p53_id;
	reactionsValues[60][0] = -1.0000000000;
	reactionsSpecies[60][1] = Abeta_id;
	reactionsValues[60][1] = 1.0000000000;
	reactionsSpecies[60][2] = GSK3b_p53_id;
	reactionsValues[60][2] = 1.0000000000;
	reactionsSpecies[61][0] = GSK3b_p53_P_id;
	reactionsValues[61][0] = -1.0000000000;
	reactionsSpecies[61][1] = Abeta_id;
	reactionsValues[61][1] = 1.0000000000;
	reactionsSpecies[61][2] = GSK3b_p53_P_id;
	reactionsValues[61][2] = 1.0000000000;
	reactionsSpecies[62][0] = AbetaDimer_id;
	reactionsValues[62][0] = -1.0000000000;
	reactionsSpecies[62][1] = Proteasome_id;
	reactionsValues[62][1] = -1.0000000000;
	reactionsSpecies[62][2] = AggAbeta_Proteasome_id;
	reactionsValues[62][2] = 1.0000000000;
	reactionsSpecies[63][0] = Abeta_id;
	reactionsValues[63][0] = -1.0000000000;
	reactionsSpecies[64][0] = Abeta_id;
	reactionsValues[64][0] = -1.0000000000;
	reactionsSpecies[64][1] = p53_mRNA_id;
	reactionsValues[64][1] = 1.0000000000;
	reactionsSpecies[64][2] = Abeta_id;
	reactionsValues[64][2] = 1.0000000000;
	reactionsSpecies[65][0] = IR_id;
	reactionsValues[65][0] = -1.0000000000;
	reactionsSpecies[65][1] = IR_id;
	reactionsValues[65][1] = 1.0000000000;
	reactionsSpecies[65][2] = damDNA_id;
	reactionsValues[65][2] = 1.0000000000;
	reactionsSpecies[66][0] = damDNA_id;
	reactionsValues[66][0] = -1.0000000000;
	reactionsSpecies[67][0] = damDNA_id;
	reactionsValues[67][0] = -1.0000000000;
	reactionsSpecies[67][1] = ATMI_id;
	reactionsValues[67][1] = -1.0000000000;
	reactionsSpecies[67][2] = damDNA_id;
	reactionsValues[67][2] = 1.0000000000;
	reactionsSpecies[67][3] = ATMA_id;
	reactionsValues[67][3] = 1.0000000000;
	reactionsSpecies[68][0] = p53_id;
	reactionsValues[68][0] = -1.0000000000;
	reactionsSpecies[68][1] = ATMA_id;
	reactionsValues[68][1] = -1.0000000000;
	reactionsSpecies[68][2] = p53_P_id;
	reactionsValues[68][2] = 1.0000000000;
	reactionsSpecies[68][3] = ATMA_id;
	reactionsValues[68][3] = 1.0000000000;
	reactionsSpecies[69][0] = p53_P_id;
	reactionsValues[69][0] = -1.0000000000;
	reactionsSpecies[69][1] = p53_id;
	reactionsValues[69][1] = 1.0000000000;
	reactionsSpecies[70][0] = Mdm2_id;
	reactionsValues[70][0] = -1.0000000000;
	reactionsSpecies[70][1] = ATMA_id;
	reactionsValues[70][1] = -1.0000000000;
	reactionsSpecies[70][2] = Mdm2_P_id;
	reactionsValues[70][2] = 1.0000000000;
	reactionsSpecies[70][3] = ATMA_id;
	reactionsValues[70][3] = 1.0000000000;
	reactionsSpecies[71][0] = Mdm2_P_id;
	reactionsValues[71][0] = -1.0000000000;
	reactionsSpecies[71][1] = Mdm2_id;
	reactionsValues[71][1] = 1.0000000000;
	reactionsSpecies[72][0] = Mdm2_P_id;
	reactionsValues[72][0] = -1.0000000000;
	reactionsSpecies[72][1] = E2_Ub_id;
	reactionsValues[72][1] = -1.0000000000;
	reactionsSpecies[72][2] = Mdm2_P_Ub_id;
	reactionsValues[72][2] = 1.0000000000;
	reactionsSpecies[72][3] = E2_id;
	reactionsValues[72][3] = 1.0000000000;
	reactionsSpecies[73][0] = Mdm2_P_Ub_id;
	reactionsValues[73][0] = -1.0000000000;
	reactionsSpecies[73][1] = E2_Ub_id;
	reactionsValues[73][1] = -1.0000000000;
	reactionsSpecies[73][2] = Mdm2_P_Ub2_id;
	reactionsValues[73][2] = 1.0000000000;
	reactionsSpecies[73][3] = E2_id;
	reactionsValues[73][3] = 1.0000000000;
	reactionsSpecies[74][0] = Mdm2_P_Ub2_id;
	reactionsValues[74][0] = -1.0000000000;
	reactionsSpecies[74][1] = E2_Ub_id;
	reactionsValues[74][1] = -1.0000000000;
	reactionsSpecies[74][2] = Mdm2_P_Ub3_id;
	reactionsValues[74][2] = 1.0000000000;
	reactionsSpecies[74][3] = E2_id;
	reactionsValues[74][3] = 1.0000000000;
	reactionsSpecies[75][0] = Mdm2_P_Ub3_id;
	reactionsValues[75][0] = -1.0000000000;
	reactionsSpecies[75][1] = E2_Ub_id;
	reactionsValues[75][1] = -1.0000000000;
	reactionsSpecies[75][2] = Mdm2_P_Ub4_id;
	reactionsValues[75][2] = 1.0000000000;
	reactionsSpecies[75][3] = E2_id;
	reactionsValues[75][3] = 1.0000000000;
	reactionsSpecies[76][0] = Mdm2_P_Ub4_id;
	reactionsValues[76][0] = -1.0000000000;
	reactionsSpecies[76][1] = Mdm2DUB_id;
	reactionsValues[76][1] = -1.0000000000;
	reactionsSpecies[76][2] = Mdm2_P_Ub3_id;
	reactionsValues[76][2] = 1.0000000000;
	reactionsSpecies[76][3] = Mdm2DUB_id;
	reactionsValues[76][3] = 1.0000000000;
	reactionsSpecies[76][4] = Ub_id;
	reactionsValues[76][4] = 1.0000000000;
	reactionsSpecies[77][0] = Mdm2_P_Ub3_id;
	reactionsValues[77][0] = -1.0000000000;
	reactionsSpecies[77][1] = Mdm2DUB_id;
	reactionsValues[77][1] = -1.0000000000;
	reactionsSpecies[77][2] = Mdm2_P_Ub2_id;
	reactionsValues[77][2] = 1.0000000000;
	reactionsSpecies[77][3] = Mdm2DUB_id;
	reactionsValues[77][3] = 1.0000000000;
	reactionsSpecies[77][4] = Ub_id;
	reactionsValues[77][4] = 1.0000000000;
	reactionsSpecies[78][0] = Mdm2_P_Ub2_id;
	reactionsValues[78][0] = -1.0000000000;
	reactionsSpecies[78][1] = Mdm2DUB_id;
	reactionsValues[78][1] = -1.0000000000;
	reactionsSpecies[78][2] = Mdm2_P_Ub_id;
	reactionsValues[78][2] = 1.0000000000;
	reactionsSpecies[78][3] = Mdm2DUB_id;
	reactionsValues[78][3] = 1.0000000000;
	reactionsSpecies[78][4] = Ub_id;
	reactionsValues[78][4] = 1.0000000000;
	reactionsSpecies[79][0] = Mdm2_P_Ub_id;
	reactionsValues[79][0] = -1.0000000000;
	reactionsSpecies[79][1] = Mdm2DUB_id;
	reactionsValues[79][1] = -1.0000000000;
	reactionsSpecies[79][2] = Mdm2_P_id;
	reactionsValues[79][2] = 1.0000000000;
	reactionsSpecies[79][3] = Mdm2DUB_id;
	reactionsValues[79][3] = 1.0000000000;
	reactionsSpecies[79][4] = Ub_id;
	reactionsValues[79][4] = 1.0000000000;
	reactionsSpecies[80][0] = Mdm2_P_Ub4_id;
	reactionsValues[80][0] = -1.0000000000;
	reactionsSpecies[80][1] = Proteasome_id;
	reactionsValues[80][1] = -1.0000000000;
	reactionsSpecies[80][2] = Mdm2_P_Ub4_Proteasome_id;
	reactionsValues[80][2] = 1.0000000000;
	reactionsSpecies[81][0] = Mdm2_P_Ub4_Proteasome_id;
	reactionsValues[81][0] = -1.0000000000;
	reactionsSpecies[81][1] = Proteasome_id;
	reactionsValues[81][1] = 1.0000000000;
	reactionsSpecies[81][2] = Ub_id;
	reactionsValues[81][2] = 4.0000000000;
	reactionsSpecies[82][0] = ATMA_id;
	reactionsValues[82][0] = -1.0000000000;
	reactionsSpecies[82][1] = ATMI_id;
	reactionsValues[82][1] = 1.0000000000;
	reactionsSpecies[83][0] = Abeta_id;
	reactionsValues[83][0] = -1.0000000000;
	reactionsSpecies[83][1] = Abeta_id;
	reactionsValues[83][1] = 1.0000000000;
	reactionsSpecies[83][2] = ROS_id;
	reactionsValues[83][2] = 1.0000000000;
	reactionsSpecies[84][0] = AbetaPlaque_id;
	reactionsValues[84][0] = -1.0000000000;
	reactionsSpecies[84][1] = AbetaPlaque_id;
	reactionsValues[84][1] = 1.0000000000;
	reactionsSpecies[84][2] = ROS_id;
	reactionsValues[84][2] = 1.0000000000;
	reactionsSpecies[85][0] = AggAbeta_Proteasome_id;
	reactionsValues[85][0] = -1.0000000000;
	reactionsSpecies[85][1] = AggAbeta_Proteasome_id;
	reactionsValues[85][1] = 1.0000000000;
	reactionsSpecies[85][2] = ROS_id;
	reactionsValues[85][2] = 1.0000000000;
	reactionsSpecies[86][0] = ROS_id;
	reactionsValues[86][0] = -1.0000000000;
	reactionsSpecies[86][1] = ROS_id;
	reactionsValues[86][1] = 1.0000000000;
	reactionsSpecies[86][2] = damDNA_id;
	reactionsValues[86][2] = 1.0000000000;
	reactionsSpecies[87][0] = Tau_id;
	reactionsValues[87][0] = 1.0000000000;
	reactionsSpecies[88][0] = Tau_id;
	reactionsValues[88][0] = -1.0000000000;
	reactionsSpecies[88][1] = Proteasome_id;
	reactionsValues[88][1] = -1.0000000000;
	reactionsSpecies[88][2] = Proteasome_Tau_id;
	reactionsValues[88][2] = 1.0000000000;
	reactionsSpecies[89][0] = Proteasome_Tau_id;
	reactionsValues[89][0] = -1.0000000000;
	reactionsSpecies[89][1] = Proteasome_id;
	reactionsValues[89][1] = 1.0000000000;
	reactionsSpecies[90][0] = Abeta_id;
	reactionsValues[90][0] = -2.0000000000;
	reactionsSpecies[90][1] = AbetaDimer_id;
	reactionsValues[90][1] = 1.0000000000;
	reactionsSpecies[91][0] = AbetaDimer_id;
	reactionsValues[91][0] = -2.0000000000;
	reactionsSpecies[91][1] = AbetaPlaque_id;
	reactionsValues[91][1] = 1.0000000000;
	reactionsSpecies[92][0] = AbetaDimer_id;
	reactionsValues[92][0] = -1.0000000000;
	reactionsSpecies[92][1] = AbetaPlaque_id;
	reactionsValues[92][1] = -1.0000000000;
	reactionsSpecies[92][2] = AbetaPlaque_id;
	reactionsValues[92][2] = 2.0000000000;
	reactionsSpecies[93][0] = AbetaDimer_id;
	reactionsValues[93][0] = -1.0000000000;
	reactionsSpecies[93][1] = Abeta_id;
	reactionsValues[93][1] = 2.0000000000;
	reactionsSpecies[94][0] = AbetaPlaque_id;
	reactionsValues[94][0] = -1.0000000000;
	reactionsSpecies[94][1] = AbetaDimer_id;
	reactionsValues[94][1] = 1.0000000000;
	reactionsSpecies[94][2] = disaggPlaque1_id;
	reactionsValues[94][2] = 1.0000000000;
	reactionsSpecies[95][0] = AbetaPlaque_id;
	reactionsValues[95][0] = -1.0000000000;
	reactionsSpecies[95][1] = antiAb_id;
	reactionsValues[95][1] = -1.0000000000;
	reactionsSpecies[95][2] = AbetaDimer_id;
	reactionsValues[95][2] = 1.0000000000;
	reactionsSpecies[95][3] = antiAb_id;
	reactionsValues[95][3] = 1.0000000000;
	reactionsSpecies[95][4] = disaggPlaque2_id;
	reactionsValues[95][4] = 1.0000000000;
	reactionsSpecies[96][0] = Abeta_id;
	reactionsValues[96][0] = -1.0000000000;
	reactionsSpecies[96][1] = antiAb_id;
	reactionsValues[96][1] = -1.0000000000;
	reactionsSpecies[96][2] = Abeta_antiAb_id;
	reactionsValues[96][2] = 1.0000000000;
	reactionsSpecies[97][0] = AbetaDimer_id;
	reactionsValues[97][0] = -1.0000000000;
	reactionsSpecies[97][1] = antiAb_id;
	reactionsValues[97][1] = -1.0000000000;
	reactionsSpecies[97][2] = AbetaDimer_antiAb_id;
	reactionsValues[97][2] = 1.0000000000;
	reactionsSpecies[98][0] = Abeta_antiAb_id;
	reactionsValues[98][0] = -1.0000000000;
	reactionsSpecies[98][1] = antiAb_id;
	reactionsValues[98][1] = 1.0000000000;
	reactionsSpecies[99][0] = AbetaDimer_antiAb_id;
	reactionsValues[99][0] = -1.0000000000;
	reactionsSpecies[99][1] = antiAb_id;
	reactionsValues[99][1] = 1.0000000000;
	reactionsSpecies[100][0] = GliaI_id;
	reactionsValues[100][0] = -1.0000000000;
	reactionsSpecies[100][1] = AbetaPlaque_id;
	reactionsValues[100][1] = -1.0000000000;
	reactionsSpecies[100][2] = GliaM1_id;
	reactionsValues[100][2] = 1.0000000000;
	reactionsSpecies[100][3] = AbetaPlaque_id;
	reactionsValues[100][3] = 1.0000000000;
	reactionsSpecies[101][0] = GliaM1_id;
	reactionsValues[101][0] = -1.0000000000;
	reactionsSpecies[101][1] = AbetaPlaque_id;
	reactionsValues[101][1] = -1.0000000000;
	reactionsSpecies[101][2] = GliaM2_id;
	reactionsValues[101][2] = 1.0000000000;
	reactionsSpecies[101][3] = AbetaPlaque_id;
	reactionsValues[101][3] = 1.0000000000;
	reactionsSpecies[102][0] = GliaM2_id;
	reactionsValues[102][0] = -1.0000000000;
	reactionsSpecies[102][1] = antiAb_id;
	reactionsValues[102][1] = -1.0000000000;
	reactionsSpecies[102][2] = GliaA_id;
	reactionsValues[102][2] = 1.0000000000;
	reactionsSpecies[102][3] = antiAb_id;
	reactionsValues[102][3] = 1.0000000000;
	reactionsSpecies[103][0] = GliaA_id;
	reactionsValues[103][0] = -1.0000000000;
	reactionsSpecies[103][1] = GliaM2_id;
	reactionsValues[103][1] = 1.0000000000;
	reactionsSpecies[104][0] = GliaM2_id;
	reactionsValues[104][0] = -1.0000000000;
	reactionsSpecies[104][1] = GliaM1_id;
	reactionsValues[104][1] = 1.0000000000;
	reactionsSpecies[105][0] = GliaM1_id;
	reactionsValues[105][0] = -1.0000000000;
	reactionsSpecies[105][1] = GliaI_id;
	reactionsValues[105][1] = 1.0000000000;
	reactionsSpecies[106][0] = AbetaPlaque_id;
	reactionsValues[106][0] = -1.0000000000;
	reactionsSpecies[106][1] = GliaA_id;
	reactionsValues[106][1] = -1.0000000000;
	reactionsSpecies[106][2] = AbetaPlaque_GliaA_id;
	reactionsValues[106][2] = 1.0000000000;
	reactionsSpecies[107][0] = AbetaPlaque_GliaA_id;
	reactionsValues[107][0] = -1.0000000000;
	reactionsSpecies[107][1] = AbetaPlaque_id;
	reactionsValues[107][1] = 1.0000000000;
	reactionsSpecies[107][2] = GliaA_id;
	reactionsValues[107][2] = 1.0000000000;
	reactionsSpecies[108][0] = AbetaPlaque_GliaA_id;
	reactionsValues[108][0] = -1.0000000000;
	reactionsSpecies[108][1] = GliaA_id;
	reactionsValues[108][1] = 1.0000000000;
	reactionsSpecies[108][2] = degAbetaGlia_id;
	reactionsValues[108][2] = 1.0000000000;
	reactionsSpecies[109][0] = AbetaPlaque_GliaA_id;
	reactionsValues[109][0] = -1.0000000000;
	reactionsSpecies[109][1] = AbetaPlaque_GliaA_id;
	reactionsValues[109][1] = 1.0000000000;
	reactionsSpecies[109][2] = ROS_id;
	reactionsValues[109][2] = 1.0000000000;
	reactionsSpecies[110][0] = antiAb_id;
	reactionsValues[110][0] = -1.0000000000;
	reactionsSpecies[111][0] = ROS_id;
	reactionsValues[111][0] = -1.0000000000;
	hiprandState localState = state[threadIdx.x];
	while (time < endTime && time < (numberOfExecutions + 1)*segmentSize) {
		cummulative_p[0] = ksynp53mRNA * Source;
		cummulative_p[1] = cummulative_p[0] + kdegp53mRNA * p53_mRNA;
		cummulative_p[2] = cummulative_p[1] + ksynMdm2 * Mdm2_mRNA;
		cummulative_p[3] = cummulative_p[2] + ksynMdm2mRNA * p53;
		cummulative_p[4] = cummulative_p[3] + ksynMdm2mRNA * p53_P;
		cummulative_p[5] = cummulative_p[4] + ksynMdm2mRNAGSK3bp53 * GSK3b_p53;
		cummulative_p[6] = cummulative_p[5] + ksynMdm2mRNAGSK3bp53 * GSK3b_p53_P;
		cummulative_p[7] = cummulative_p[6] + kdegMdm2mRNA * Mdm2_mRNA;
		cummulative_p[8] = cummulative_p[7] + kbinMdm2p53 * p53 * Mdm2;
		cummulative_p[9] = cummulative_p[8] + krelMdm2p53 * Mdm2_p53;
		cummulative_p[10] = cummulative_p[9] + kbinGSK3bp53 * GSK3b * p53;
		cummulative_p[11] = cummulative_p[10] + krelGSK3bp53 * GSK3b_p53;
		cummulative_p[12] = cummulative_p[11] + kbinGSK3bp53 * GSK3b * p53_P;
		cummulative_p[13] = cummulative_p[12] + krelGSK3bp53 * GSK3b_p53_P;
		cummulative_p[14] = cummulative_p[13] + kbinE1Ub * E1 * Ub * ATP / (5000 + ATP);
		cummulative_p[15] = cummulative_p[14] + kbinE2Ub * E2 * E1_Ub;
		cummulative_p[16] = cummulative_p[15] + kMdm2Ub * Mdm2 * E2_Ub;
		cummulative_p[17] = cummulative_p[16] + kMdm2PolyUb * Mdm2_Ub * E2_Ub;
		cummulative_p[18] = cummulative_p[17] + kMdm2PolyUb * Mdm2_Ub2 * E2_Ub;
		cummulative_p[19] = cummulative_p[18] + kMdm2PolyUb * Mdm2_Ub3 * E2_Ub;
		cummulative_p[20] = cummulative_p[19] + kactDUBMdm2 * Mdm2_Ub4 * Mdm2DUB;
		cummulative_p[21] = cummulative_p[20] + kactDUBMdm2 * Mdm2_Ub3 * Mdm2DUB;
		cummulative_p[22] = cummulative_p[21] + kactDUBMdm2 * Mdm2_Ub2 * Mdm2DUB;
		cummulative_p[23] = cummulative_p[22] + kactDUBMdm2 * Mdm2_Ub * Mdm2DUB;
		cummulative_p[24] = cummulative_p[23] + kbinProt * Mdm2_Ub4 * Proteasome;
		cummulative_p[25] = cummulative_p[24] + kdegMdm2 * Mdm2_Ub4_Proteasome * kproteff;
		cummulative_p[26] = cummulative_p[25] + ksynp53 * p53_mRNA;
		cummulative_p[27] = cummulative_p[26] + kp53Ub * E2_Ub * Mdm2_p53;
		cummulative_p[28] = cummulative_p[27] + kp53PolyUb * Mdm2_p53_Ub * E2_Ub;
		cummulative_p[29] = cummulative_p[28] + kp53PolyUb * Mdm2_p53_Ub2 * E2_Ub;
		cummulative_p[30] = cummulative_p[29] + kp53PolyUb * Mdm2_p53_Ub3 * E2_Ub;
		cummulative_p[31] = cummulative_p[30] + kactDUBp53 * Mdm2_p53_Ub4 * p53DUB;
		cummulative_p[32] = cummulative_p[31] + kactDUBp53 * Mdm2_p53_Ub3 * p53DUB;
		cummulative_p[33] = cummulative_p[32] + kactDUBp53 * Mdm2_p53_Ub2 * p53DUB;
		cummulative_p[34] = cummulative_p[33] + kactDUBp53 * Mdm2_p53_Ub * p53DUB;
		cummulative_p[35] = cummulative_p[34] + kphosMdm2GSK3b * Mdm2_p53_Ub4 * GSK3b;
		cummulative_p[36] = cummulative_p[35] + kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53;
		cummulative_p[37] = cummulative_p[36] + kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53_P;
		cummulative_p[38] = cummulative_p[37] + kbinProt * Mdm2_P1_p53_Ub4 * Proteasome;
		cummulative_p[39] = cummulative_p[38] + kdegp53 * kproteff * p53_Ub4_Proteasome * ATP / (5000 + ATP);
		cummulative_p[40] = cummulative_p[39] + kbinMTTau * Tau;
		cummulative_p[41] = cummulative_p[40] + krelMTTau * MT_Tau;
		cummulative_p[42] = cummulative_p[41] + kphospTauGSK3bp53 * GSK3b_p53 * Tau;
		cummulative_p[43] = cummulative_p[42] + kphospTauGSK3bp53 * GSK3b_p53 * Tau_P1;
		cummulative_p[44] = cummulative_p[43] + kphospTauGSK3bp53 * GSK3b_p53_P * Tau;
		cummulative_p[45] = cummulative_p[44] + kphospTauGSK3bp53 * GSK3b_p53_P * Tau_P1;
		cummulative_p[46] = cummulative_p[45] + kphospTauGSK3b * GSK3b * Tau;
		cummulative_p[47] = cummulative_p[46] + kphospTauGSK3b * GSK3b * Tau_P1;
		cummulative_p[48] = cummulative_p[47] + kdephospTau * Tau_P2 * PP1;
		cummulative_p[49] = cummulative_p[48] + kdephospTau * Tau_P1 * PP1;
		cummulative_p[50] = cummulative_p[49] + kaggTauP1 * Tau_P1 * (Tau_P1 - 1) * 0.5;
		cummulative_p[51] = cummulative_p[50] + kaggTauP1 * Tau_P1 * AggTau;
		cummulative_p[52] = cummulative_p[51] + kaggTauP2 * Tau_P2 * (Tau_P2 - 1) * 0.5;
		cummulative_p[53] = cummulative_p[52] + kaggTauP2 * Tau_P2 * AggTau;
		cummulative_p[54] = cummulative_p[53] + kaggTau * Tau * (Tau - 1) * 0.5;
		cummulative_p[55] = cummulative_p[54] + kaggTau * Tau * AggTau;
		cummulative_p[56] = cummulative_p[55] + ktangfor * AggTau * (AggTau - 1) * 0.5;
		cummulative_p[57] = cummulative_p[56] + ktangfor * AggTau * NFT;
		cummulative_p[58] = cummulative_p[57] + kinhibprot * AggTau * Proteasome;
		cummulative_p[59] = cummulative_p[58] + kprodAbeta * Source;
		cummulative_p[60] = cummulative_p[59] + kprodAbeta2 * GSK3b_p53;
		cummulative_p[61] = cummulative_p[60] + kprodAbeta2 * GSK3b_p53_P;
		cummulative_p[62] = cummulative_p[61] + kinhibprot * AbetaDimer * Proteasome;
		cummulative_p[63] = cummulative_p[62] + kdegAbeta * Abeta;
		cummulative_p[64] = cummulative_p[63] + ksynp53mRNAAbeta * Abeta;
		cummulative_p[65] = cummulative_p[64] + kdam * IR;
		cummulative_p[66] = cummulative_p[65] + krepair * damDNA;
		cummulative_p[67] = cummulative_p[66] + kactATM * damDNA * ATMI;
		cummulative_p[68] = cummulative_p[67] + kphosp53 * p53 * ATMA;
		cummulative_p[69] = cummulative_p[68] + kdephosp53 * p53_P;
		cummulative_p[70] = cummulative_p[69] + kphosMdm2 * Mdm2 * ATMA;
		cummulative_p[71] = cummulative_p[70] + kdephosMdm2 * Mdm2_P;
		cummulative_p[72] = cummulative_p[71] + kMdm2PUb * Mdm2_P * E2_Ub;
		cummulative_p[73] = cummulative_p[72] + kMdm2PolyUb * Mdm2_P_Ub * E2_Ub;
		cummulative_p[74] = cummulative_p[73] + kMdm2PolyUb * Mdm2_P_Ub2 * E2_Ub;
		cummulative_p[75] = cummulative_p[74] + kMdm2PolyUb * Mdm2_P_Ub3 * E2_Ub;
		cummulative_p[76] = cummulative_p[75] + kactDUBMdm2 * Mdm2_P_Ub4 * Mdm2DUB;
		cummulative_p[77] = cummulative_p[76] + kactDUBMdm2 * Mdm2_P_Ub3 * Mdm2DUB;
		cummulative_p[78] = cummulative_p[77] + kactDUBMdm2 * Mdm2_P_Ub2 * Mdm2DUB;
		cummulative_p[79] = cummulative_p[78] + kactDUBMdm2 * Mdm2_P_Ub * Mdm2DUB;
		cummulative_p[80] = cummulative_p[79] + kbinProt * Mdm2_P_Ub4 * Proteasome;
		cummulative_p[81] = cummulative_p[80] + kdegMdm2 * Mdm2_P_Ub4_Proteasome * kproteff;
		cummulative_p[82] = cummulative_p[81] + kinactATM * ATMA;
		cummulative_p[83] = cummulative_p[82] + kgenROSAbeta * Abeta;
		cummulative_p[84] = cummulative_p[83] + kgenROSPlaque * AbetaPlaque;
		cummulative_p[85] = cummulative_p[84] + kgenROSAbeta * AggAbeta_Proteasome;
		cummulative_p[86] = cummulative_p[85] + kdamROS * ROS;
		cummulative_p[87] = cummulative_p[86] + ksynTau * Source;
		cummulative_p[88] = cummulative_p[87] + kbinTauProt * Tau * Proteasome;
		cummulative_p[89] = cummulative_p[88] + kdegTau20SProt * Proteasome_Tau;
		cummulative_p[90] = cummulative_p[89] + kaggAbeta * Abeta * (Abeta - 1) * 0.5;
		cummulative_p[91] = cummulative_p[90] + kpf * AbetaDimer * (AbetaDimer - 1) * 0.5;
		cummulative_p[92] = cummulative_p[91] + kpg * AbetaDimer * pow(AbetaPlaque, 2) / (pow(kpghalf, 2) + pow(AbetaPlaque, 2));
		cummulative_p[93] = cummulative_p[92] + kdisaggAbeta * AbetaDimer;
		cummulative_p[94] = cummulative_p[93] + kdisaggAbeta1 * AbetaPlaque;
		cummulative_p[95] = cummulative_p[94] + kdisaggAbeta2 * antiAb * AbetaPlaque;
		cummulative_p[96] = cummulative_p[95] + kbinAbantiAb * Abeta * antiAb;
		cummulative_p[97] = cummulative_p[96] + kbinAbantiAb * AbetaDimer * antiAb;
		cummulative_p[98] = cummulative_p[97] + 10 * kdegAbeta * Abeta_antiAb;
		cummulative_p[99] = cummulative_p[98] + 10 * kdegAbeta * AbetaDimer_antiAb;
		cummulative_p[100] = cummulative_p[99] + kactglia1 * GliaI * AbetaPlaque;
		cummulative_p[101] = cummulative_p[100] + kactglia1 * GliaM1 * AbetaPlaque;
		cummulative_p[102] = cummulative_p[101] + kactglia2 * GliaM2 * antiAb;
		cummulative_p[103] = cummulative_p[102] + kinactglia1 * GliaA;
		cummulative_p[104] = cummulative_p[103] + kinactglia2 * GliaM2;
		cummulative_p[105] = cummulative_p[104] + kinactglia2 * GliaM1;
		cummulative_p[106] = cummulative_p[105] + kbinAbetaGlia * AbetaPlaque * GliaA;
		cummulative_p[107] = cummulative_p[106] + krelAbetaGlia * AbetaPlaque_GliaA;
		cummulative_p[108] = cummulative_p[107] + kdegAbetaGlia * AbetaPlaque_GliaA;
		cummulative_p[109] = cummulative_p[108] + kgenROSGlia * AbetaPlaque_GliaA;
		cummulative_p[110] = cummulative_p[109] + kdegAntiAb * antiAb;
		cummulative_p[111] = cummulative_p[110] + kremROS * ROS;
		if (time >= segmentSize * numberOfExecutions + step * stepCount) {
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 0], species[0]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 1], species[1]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 2], species[2]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 3], species[3]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 4], species[4]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 5], species[5]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 6], species[6]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 7], species[7]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 8], species[8]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 9], species[9]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 10], species[10]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 11], species[11]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 12], species[12]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 13], species[13]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 14], species[14]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 15], species[15]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 16], species[16]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 17], species[17]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 18], species[18]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 19], species[19]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 20], species[20]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 21], species[21]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 22], species[22]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 23], species[23]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 24], species[24]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 25], species[25]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 26], species[26]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 27], species[27]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 28], species[28]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 29], species[29]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 30], species[30]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 31], species[31]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 32], species[32]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 33], species[33]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 34], species[34]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 35], species[35]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 36], species[36]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 37], species[37]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 38], species[38]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 39], species[39]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 40], species[40]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 41], species[41]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 42], species[42]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 43], species[43]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 44], species[44]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 45], species[45]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 46], species[46]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 47], species[47]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 48], species[48]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 49], species[49]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 50], species[50]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 51], species[51]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 52], species[52]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 53], species[53]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 54], species[54]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 55], species[55]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 56], species[56]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 57], species[57]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 58], species[58]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 59], species[59]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 60], species[60]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 61], species[61]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 62], species[62]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 63], species[63]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 64], species[64]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 65], species[65]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 66], species[66]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 67], species[67]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 68], species[68]);
			stepCount++;
		}
		sum_p = cummulative_p[111];
		random = hiprand_uniform(&localState);
		if (sum_p > 0) timeStep = -log(random) / sum_p;
		else break;
		random = hiprand_uniform(&localState);
		random *= sum_p;
		indexMin = 0;
		indexMax = 111;
		while (indexMax > indexMin) {
			reaction = (indexMin + indexMax) / 2;
			if (cummulative_p[reaction] <= random) {
				if (cummulative_p[reaction + 1] > random) {
					reaction++;
					break;
				}
				else {
					indexMin = reaction;
				}
			}
			else {
				indexMax = reaction;
			}
		}
		for (int i = 0; i < 5; i++) {
			if (reactionsSpecies[reaction][i] == -1) { break; }
			species[reactionsSpecies[reaction][i]] += reactionsValues[reaction][i];
		}
		if (triggerEvent0 == 0 && time >= 345600) {
			triggerEvent0 = 1;
			antiAb += 50;
		}
		time += timeStep;
	}
	species_global[Mdm2_id * 32 + threadIdx.x] = species[0];
	species_global[p53_id * 32 + threadIdx.x] = species[1];
	species_global[Mdm2_p53_id * 32 + threadIdx.x] = species[2];
	species_global[Mdm2_mRNA_id * 32 + threadIdx.x] = species[3];
	species_global[p53_mRNA_id * 32 + threadIdx.x] = species[4];
	species_global[ATMA_id * 32 + threadIdx.x] = species[5];
	species_global[ATMI_id * 32 + threadIdx.x] = species[6];
	species_global[p53_P_id * 32 + threadIdx.x] = species[7];
	species_global[Mdm2_P_id * 32 + threadIdx.x] = species[8];
	species_global[IR_id * 32 + threadIdx.x] = species[9];
	species_global[ROS_id * 32 + threadIdx.x] = species[10];
	species_global[damDNA_id * 32 + threadIdx.x] = species[11];
	species_global[E1_id * 32 + threadIdx.x] = species[12];
	species_global[E2_id * 32 + threadIdx.x] = species[13];
	species_global[E1_Ub_id * 32 + threadIdx.x] = species[14];
	species_global[E2_Ub_id * 32 + threadIdx.x] = species[15];
	species_global[Proteasome_id * 32 + threadIdx.x] = species[16];
	species_global[Ub_id * 32 + threadIdx.x] = species[17];
	species_global[p53DUB_id * 32 + threadIdx.x] = species[18];
	species_global[Mdm2DUB_id * 32 + threadIdx.x] = species[19];
	species_global[DUB_id * 32 + threadIdx.x] = species[20];
	species_global[Mdm2_p53_Ub_id * 32 + threadIdx.x] = species[21];
	species_global[Mdm2_p53_Ub2_id * 32 + threadIdx.x] = species[22];
	species_global[Mdm2_p53_Ub3_id * 32 + threadIdx.x] = species[23];
	species_global[Mdm2_p53_Ub4_id * 32 + threadIdx.x] = species[24];
	species_global[Mdm2_P1_p53_Ub4_id * 32 + threadIdx.x] = species[25];
	species_global[Mdm2_Ub_id * 32 + threadIdx.x] = species[26];
	species_global[Mdm2_Ub2_id * 32 + threadIdx.x] = species[27];
	species_global[Mdm2_Ub3_id * 32 + threadIdx.x] = species[28];
	species_global[Mdm2_Ub4_id * 32 + threadIdx.x] = species[29];
	species_global[Mdm2_P_Ub_id * 32 + threadIdx.x] = species[30];
	species_global[Mdm2_P_Ub2_id * 32 + threadIdx.x] = species[31];
	species_global[Mdm2_P_Ub3_id * 32 + threadIdx.x] = species[32];
	species_global[Mdm2_P_Ub4_id * 32 + threadIdx.x] = species[33];
	species_global[p53_Ub4_Proteasome_id * 32 + threadIdx.x] = species[34];
	species_global[Mdm2_Ub4_Proteasome_id * 32 + threadIdx.x] = species[35];
	species_global[Mdm2_P_Ub4_Proteasome_id * 32 + threadIdx.x] = species[36];
	species_global[GSK3b_id * 32 + threadIdx.x] = species[37];
	species_global[GSK3b_p53_id * 32 + threadIdx.x] = species[38];
	species_global[GSK3b_p53_P_id * 32 + threadIdx.x] = species[39];
	species_global[Abeta_id * 32 + threadIdx.x] = species[40];
	species_global[AggAbeta_Proteasome_id * 32 + threadIdx.x] = species[41];
	species_global[AbetaPlaque_id * 32 + threadIdx.x] = species[42];
	species_global[Tau_id * 32 + threadIdx.x] = species[43];
	species_global[Tau_P1_id * 32 + threadIdx.x] = species[44];
	species_global[Tau_P2_id * 32 + threadIdx.x] = species[45];
	species_global[MT_Tau_id * 32 + threadIdx.x] = species[46];
	species_global[AggTau_id * 32 + threadIdx.x] = species[47];
	species_global[AggTau_Proteasome_id * 32 + threadIdx.x] = species[48];
	species_global[Proteasome_Tau_id * 32 + threadIdx.x] = species[49];
	species_global[PP1_id * 32 + threadIdx.x] = species[50];
	species_global[NFT_id * 32 + threadIdx.x] = species[51];
	species_global[ATP_id * 32 + threadIdx.x] = species[52];
	species_global[ADP_id * 32 + threadIdx.x] = species[53];
	species_global[AMP_id * 32 + threadIdx.x] = species[54];
	species_global[AbetaDimer_id * 32 + threadIdx.x] = species[55];
	species_global[AbetaPlaque_GliaA_id * 32 + threadIdx.x] = species[56];
	species_global[GliaI_id * 32 + threadIdx.x] = species[57];
	species_global[GliaM1_id * 32 + threadIdx.x] = species[58];
	species_global[GliaM2_id * 32 + threadIdx.x] = species[59];
	species_global[GliaA_id * 32 + threadIdx.x] = species[60];
	species_global[antiAb_id * 32 + threadIdx.x] = species[61];
	species_global[Abeta_antiAb_id * 32 + threadIdx.x] = species[62];
	species_global[AbetaDimer_antiAb_id * 32 + threadIdx.x] = species[63];
	species_global[degAbetaGlia_id * 32 + threadIdx.x] = species[64];
	species_global[disaggPlaque1_id * 32 + threadIdx.x] = species[65];
	species_global[disaggPlaque2_id * 32 + threadIdx.x] = species[66];
	species_global[Source_id * 32 + threadIdx.x] = species[67];
	species_global[Sink_id * 32 + threadIdx.x] = species[68];
	state[threadIdx.x] = localState;
}

__global__
void initCurand(hiprandState* state, unsigned long long seed) {
	hiprand_init(seed, threadIdx.x, 0, &state[threadIdx.x]);
}

int main()
{
	hipError_t cudaStatus;
	float* output;
	float* dev_output;
	output = (float*)malloc(334 * 69 * sizeof(float));
	for (int i = 0; i < 334 * 69; i++) {
		output[i] = 0;
	}
	cudaStatus = hipMalloc(&dev_output, 334 * 69 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_output, output, 334 * 69 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* species_global;
	cudaStatus = hipMalloc(&species_global, 69 * 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float init_species[69];
	init_species[0] = 5.0000000000;
	init_species[1] = 5.0000000000;
	init_species[2] = 95.0000000000;
	init_species[3] = 10.0000000000;
	init_species[4] = 10.0000000000;
	init_species[5] = 0.0000000000;
	init_species[6] = 200.0000000000;
	init_species[7] = 0.0000000000;
	init_species[8] = 0.0000000000;
	init_species[9] = 0.0000000000;
	init_species[10] = 0.0000000000;
	init_species[11] = 0.0000000000;
	init_species[12] = 100.0000000000;
	init_species[13] = 100.0000000000;
	init_species[14] = 0.0000000000;
	init_species[15] = 0.0000000000;
	init_species[16] = 500.0000000000;
	init_species[17] = 4000.0000000000;
	init_species[18] = 200.0000000000;
	init_species[19] = 200.0000000000;
	init_species[20] = 200.0000000000;
	init_species[21] = 0.0000000000;
	init_species[22] = 0.0000000000;
	init_species[23] = 0.0000000000;
	init_species[24] = 0.0000000000;
	init_species[25] = 0.0000000000;
	init_species[26] = 0.0000000000;
	init_species[27] = 0.0000000000;
	init_species[28] = 0.0000000000;
	init_species[29] = 0.0000000000;
	init_species[30] = 0.0000000000;
	init_species[31] = 0.0000000000;
	init_species[32] = 0.0000000000;
	init_species[33] = 0.0000000000;
	init_species[34] = 0.0000000000;
	init_species[35] = 0.0000000000;
	init_species[36] = 0.0000000000;
	init_species[37] = 500.0000000000;
	init_species[38] = 0.0000000000;
	init_species[39] = 0.0000000000;
	init_species[40] = 0.0000000000;
	init_species[41] = 0.0000000000;
	init_species[42] = 0.0000000000;
	init_species[43] = 0.0000000000;
	init_species[44] = 0.0000000000;
	init_species[45] = 0.0000000000;
	init_species[46] = 100.0000000000;
	init_species[47] = 0.0000000000;
	init_species[48] = 0.0000000000;
	init_species[49] = 0.0000000000;
	init_species[50] = 50.0000000000;
	init_species[51] = 0.0000000000;
	init_species[52] = 10000.0000000000;
	init_species[53] = 1000.0000000000;
	init_species[54] = 1000.0000000000;
	init_species[55] = 0.0000000000;
	init_species[56] = 0.0000000000;
	init_species[57] = 100.0000000000;
	init_species[58] = 0.0000000000;
	init_species[59] = 0.0000000000;
	init_species[60] = 0.0000000000;
	init_species[61] = 0.0000000000;
	init_species[62] = 0.0000000000;
	init_species[63] = 0.0000000000;
	init_species[64] = 0.0000000000;
	init_species[65] = 0.0000000000;
	init_species[66] = 0.0000000000;
	init_species[67] = 1.0000000000;
	init_species[68] = 1.0000000000;
	cudaStatus = hipMemcpy(species_global, &init_species, sizeof(float) * 69, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	hiprandState *devStates;
	CUDA_CALL(hipMalloc((void **)&devStates, 32 * sizeof(hiprandState)));
	initCurand << <1, 32 >> > (devStates, SEED);
	hipEvent_t start, stop;
	float milliseconds;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	printf("GO\n");
	for (int i = 0; i < 10; i++) {
		hipEventRecord(start);

		simulate << <1, 32 >> > (i, dev_output, devStates, 60.0000000000, 20000.0000000000, 2000, species_global); cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess) { fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

		cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
		hipEventRecord(stop);
		printf("%d\n", i);
		hipEventSynchronize(stop);
		milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("TIME: %lf\n", milliseconds);
	}


	cudaStatus = hipMemcpy(output, dev_output, 334 * 69 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	FILE* results = fopen("results.csv", "w");
	if (results == NULL) {
		printf("Error acesssing results!");
		exit(1);
	}
	fprintf(results, "time");
	fprintf(results, ", Mdm2");
	fprintf(results, ", p53");
	fprintf(results, ", Mdm2_p53");
	fprintf(results, ", Mdm2_mRNA");
	fprintf(results, ", p53_mRNA");
	fprintf(results, ", ATMA");
	fprintf(results, ", ATMI");
	fprintf(results, ", p53_P");
	fprintf(results, ", Mdm2_P");
	fprintf(results, ", IR");
	fprintf(results, ", ROS");
	fprintf(results, ", damDNA");
	fprintf(results, ", E1");
	fprintf(results, ", E2");
	fprintf(results, ", E1_Ub");
	fprintf(results, ", E2_Ub");
	fprintf(results, ", Proteasome");
	fprintf(results, ", Ub");
	fprintf(results, ", p53DUB");
	fprintf(results, ", Mdm2DUB");
	fprintf(results, ", DUB");
	fprintf(results, ", Mdm2_p53_Ub");
	fprintf(results, ", Mdm2_p53_Ub2");
	fprintf(results, ", Mdm2_p53_Ub3");
	fprintf(results, ", Mdm2_p53_Ub4");
	fprintf(results, ", Mdm2_P1_p53_Ub4");
	fprintf(results, ", Mdm2_Ub");
	fprintf(results, ", Mdm2_Ub2");
	fprintf(results, ", Mdm2_Ub3");
	fprintf(results, ", Mdm2_Ub4");
	fprintf(results, ", Mdm2_P_Ub");
	fprintf(results, ", Mdm2_P_Ub2");
	fprintf(results, ", Mdm2_P_Ub3");
	fprintf(results, ", Mdm2_P_Ub4");
	fprintf(results, ", p53_Ub4_Proteasome");
	fprintf(results, ", Mdm2_Ub4_Proteasome");
	fprintf(results, ", Mdm2_P_Ub4_Proteasome");
	fprintf(results, ", GSK3b");
	fprintf(results, ", GSK3b_p53");
	fprintf(results, ", GSK3b_p53_P");
	fprintf(results, ", Abeta");
	fprintf(results, ", AggAbeta_Proteasome");
	fprintf(results, ", AbetaPlaque");
	fprintf(results, ", Tau");
	fprintf(results, ", Tau_P1");
	fprintf(results, ", Tau_P2");
	fprintf(results, ", MT_Tau");
	fprintf(results, ", AggTau");
	fprintf(results, ", AggTau_Proteasome");
	fprintf(results, ", Proteasome_Tau");
	fprintf(results, ", PP1");
	fprintf(results, ", NFT");
	fprintf(results, ", ATP");
	fprintf(results, ", ADP");
	fprintf(results, ", AMP");
	fprintf(results, ", AbetaDimer");
	fprintf(results, ", AbetaPlaque_GliaA");
	fprintf(results, ", GliaI");
	fprintf(results, ", GliaM1");
	fprintf(results, ", GliaM2");
	fprintf(results, ", GliaA");
	fprintf(results, ", antiAb");
	fprintf(results, ", Abeta_antiAb");
	fprintf(results, ", AbetaDimer_antiAb");
	fprintf(results, ", degAbetaGlia");
	fprintf(results, ", disaggPlaque1");
	fprintf(results, ", disaggPlaque2");
	fprintf(results, ", Source");
	fprintf(results, ", Sink");
	fprintf(results, "\n");
	for (int i = 0; i < 334; i++) {
		fprintf(results, "%.10lf", 60.0000000000*i);
		for (int j = 0; j < 69; j++) {
			fprintf(results, ", %.10lf", output[69 * i + j] / 32);
		}
		fprintf(results, "\n");
	}
	fprintf(results, "\n");
Error:
	hipFree(dev_output);
	hipFree(species_global);

	return 0;
}
