#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define pow powf

#define SEED 23
#define Mdm2 species[0]
#define Mdm2_id 0
#define p53 species[1]
#define p53_id 1
#define Mdm2_p53 species[2]
#define Mdm2_p53_id 2
#define Mdm2_mRNA species[3]
#define Mdm2_mRNA_id 3
#define p53_mRNA species[4]
#define p53_mRNA_id 4
#define ATMA species[5]
#define ATMA_id 5
#define ATMI species[6]
#define ATMI_id 6
#define p53_P species[7]
#define p53_P_id 7
#define Mdm2_P species[8]
#define Mdm2_P_id 8
#define IR species[9]
#define IR_id 9
#define ROS species[10]
#define ROS_id 10
#define damDNA species[11]
#define damDNA_id 11
#define E1 species[12]
#define E1_id 12
#define E2 species[13]
#define E2_id 13
#define E1_Ub species[14]
#define E1_Ub_id 14
#define E2_Ub species[15]
#define E2_Ub_id 15
#define Proteasome species[16]
#define Proteasome_id 16
#define Ub species[17]
#define Ub_id 17
#define p53DUB species[18]
#define p53DUB_id 18
#define Mdm2DUB species[19]
#define Mdm2DUB_id 19
#define DUB species[20]
#define DUB_id 20
#define Mdm2_p53_Ub species[21]
#define Mdm2_p53_Ub_id 21
#define Mdm2_p53_Ub2 species[22]
#define Mdm2_p53_Ub2_id 22
#define Mdm2_p53_Ub3 species[23]
#define Mdm2_p53_Ub3_id 23
#define Mdm2_p53_Ub4 species[24]
#define Mdm2_p53_Ub4_id 24
#define Mdm2_P1_p53_Ub4 species[25]
#define Mdm2_P1_p53_Ub4_id 25
#define Mdm2_Ub species[26]
#define Mdm2_Ub_id 26
#define Mdm2_Ub2 species[27]
#define Mdm2_Ub2_id 27
#define Mdm2_Ub3 species[28]
#define Mdm2_Ub3_id 28
#define Mdm2_Ub4 species[29]
#define Mdm2_Ub4_id 29
#define Mdm2_P_Ub species[30]
#define Mdm2_P_Ub_id 30
#define Mdm2_P_Ub2 species[31]
#define Mdm2_P_Ub2_id 31
#define Mdm2_P_Ub3 species[32]
#define Mdm2_P_Ub3_id 32
#define Mdm2_P_Ub4 species[33]
#define Mdm2_P_Ub4_id 33
#define p53_Ub4_Proteasome species[34]
#define p53_Ub4_Proteasome_id 34
#define Mdm2_Ub4_Proteasome species[35]
#define Mdm2_Ub4_Proteasome_id 35
#define Mdm2_P_Ub4_Proteasome species[36]
#define Mdm2_P_Ub4_Proteasome_id 36
#define GSK3b species[37]
#define GSK3b_id 37
#define GSK3b_p53 species[38]
#define GSK3b_p53_id 38
#define GSK3b_p53_P species[39]
#define GSK3b_p53_P_id 39
#define Abeta species[40]
#define Abeta_id 40
#define AggAbeta_Proteasome species[41]
#define AggAbeta_Proteasome_id 41
#define AbetaPlaque species[42]
#define AbetaPlaque_id 42
#define Tau species[43]
#define Tau_id 43
#define Tau_P1 species[44]
#define Tau_P1_id 44
#define Tau_P2 species[45]
#define Tau_P2_id 45
#define MT_Tau species[46]
#define MT_Tau_id 46
#define AggTau species[47]
#define AggTau_id 47
#define AggTau_Proteasome species[48]
#define AggTau_Proteasome_id 48
#define Proteasome_Tau species[49]
#define Proteasome_Tau_id 49
#define PP1 species[50]
#define PP1_id 50
#define NFT species[51]
#define NFT_id 51
#define ATP species[52]
#define ATP_id 52
#define ADP species[53]
#define ADP_id 53
#define AMP species[54]
#define AMP_id 54
#define AbetaDimer species[55]
#define AbetaDimer_id 55
#define AbetaPlaque_GliaA species[56]
#define AbetaPlaque_GliaA_id 56
#define GliaI species[57]
#define GliaI_id 57
#define GliaM1 species[58]
#define GliaM1_id 58
#define GliaM2 species[59]
#define GliaM2_id 59
#define GliaA species[60]
#define GliaA_id 60
#define antiAb species[61]
#define antiAb_id 61
#define Abeta_antiAb species[62]
#define Abeta_antiAb_id 62
#define AbetaDimer_antiAb species[63]
#define AbetaDimer_antiAb_id 63
#define degAbetaGlia species[64]
#define degAbetaGlia_id 64
#define disaggPlaque1 species[65]
#define disaggPlaque1_id 65
#define disaggPlaque2 species[66]
#define disaggPlaque2_id 66
#define Source species[67]
#define Source_id 67
#define Sink species[68]
#define Sink_id 68
#define cell 1.0000000000
#define ksynp53mRNA 0.0010000000
#define kdegp53mRNA 0.0001000000
#define ksynMdm2mRNA 0.0005000000
#define kdegMdm2mRNA 0.0005000000
#define ksynMdm2mRNAGSK3bp53 0.0007000000
#define ksynp53 0.0070000000
#define kdegp53 0.0050000000
#define kbinMdm2p53 0.0011550000
#define krelMdm2p53 0.0000115500
#define kbinGSK3bp53 0.0000020000
#define krelGSK3bp53 0.0020000000
#define ksynMdm2 0.0004950000
#define kdegMdm2 0.0100000000
#define kbinE1Ub 0.0002000000
#define kbinE2Ub 0.0010000000
#define kp53Ub 0.0000500000
#define kp53PolyUb 0.0100000000
#define kbinProt 0.0000020000
#define kactDUBp53 0.0000001000
#define kactDUBProtp53 0.0001000000
#define kactDUBMdm2 0.0000001000
#define kMdm2Ub 0.0000045600
#define kMdm2PUb 0.0000068400
#define kMdm2PolyUb 0.0045600000
#define kdam 0.0800000000
#define krepair 0.0000200000
#define kactATM 0.0001000000
#define kinactATM 0.0005000000
#define kphosp53 0.0002000000
#define kdephosp53 0.5000000000
#define kphosMdm2 2.0000000000
#define kdephosMdm2 0.5000000000
#define kphosMdm2GSK3b 0.0050000000
#define kphosMdm2GSK3bp53 0.5000000000
#define kphospTauGSK3bp53 0.1000000000
#define kphospTauGSK3b 0.0002000000
#define kdephospTau 0.0100000000
#define kbinMTTau 0.1000000000
#define krelMTTau 0.0001000000
#define ksynTau 0.0000800000
#define kbinTauProt 0.0000001925
#define kdegTau20SProt 0.0100000000
#define kaggTau 0.0000000100
#define kaggTauP1 0.0000000100
#define kaggTauP2 0.0000001000
#define ktangfor 0.0010000000
#define kinhibprot 0.0000001000
#define ksynp53mRNAAbeta 0.0000100000
#define kdamROS 0.0000100000
#define kgenROSAbeta 0.0000200000
#define kgenROSPlaque 0.0000100000
#define kgenROSGlia 0.0000100000
#define kproteff 1.0000000000
#define kremROS 0.0000700000
#define kprodAbeta 0.0000186000
#define kprodAbeta2 0.0000186000
#define kdegAbeta 0.0000150000
#define kaggAbeta 0.0000030000
#define kdisaggAbeta 0.0000010000
#define kdisaggAbeta1 0.0002000000
#define kdisaggAbeta2 0.0000010000
#define kdegAbetaGlia 0.0050000000
#define kpf 0.2000000000
#define kpg 0.1500000000
#define kpghalf 10.0000000000
#define kactglia1 0.0000006000
#define kactglia2 0.0000006000
#define kinactglia1 0.0000050000
#define kinactglia2 0.0000050000
#define kbinAbetaGlia 0.0000100000
#define krelAbetaGlia 0.0000500000
#define kdegAntiAb 0.0000027500
#define kbinAbantiAb 0.0000010000

__global__
void simulate(int numberOfExecutions, float* output, hiprandState *state, float step, float endTime, float segmentSize, float* Mdm2_aux, float* Mdm2_global, float* p53_aux, float* p53_global, float* Mdm2_p53_aux, float* Mdm2_p53_global, float* Mdm2_mRNA_aux, float* Mdm2_mRNA_global, float* p53_mRNA_aux, float* p53_mRNA_global, float* ATMA_aux, float* ATMA_global, float* ATMI_aux, float* ATMI_global, float* p53_P_aux, float* p53_P_global, float* Mdm2_P_aux, float* Mdm2_P_global, float* IR_aux, float* IR_global, float* ROS_aux, float* ROS_global, float* damDNA_aux, float* damDNA_global, float* E1_aux, float* E1_global, float* E2_aux, float* E2_global, float* E1_Ub_aux, float* E1_Ub_global, float* E2_Ub_aux, float* E2_Ub_global, float* Proteasome_aux, float* Proteasome_global, float* Ub_aux, float* Ub_global, float* p53DUB_aux, float* p53DUB_global, float* Mdm2DUB_aux, float* Mdm2DUB_global, float* DUB_aux, float* DUB_global, float* Mdm2_p53_Ub_aux, float* Mdm2_p53_Ub_global, float* Mdm2_p53_Ub2_aux, float* Mdm2_p53_Ub2_global, float* Mdm2_p53_Ub3_aux, float* Mdm2_p53_Ub3_global, float* Mdm2_p53_Ub4_aux, float* Mdm2_p53_Ub4_global, float* Mdm2_P1_p53_Ub4_aux, float* Mdm2_P1_p53_Ub4_global, float* Mdm2_Ub_aux, float* Mdm2_Ub_global, float* Mdm2_Ub2_aux, float* Mdm2_Ub2_global, float* Mdm2_Ub3_aux, float* Mdm2_Ub3_global, float* Mdm2_Ub4_aux, float* Mdm2_Ub4_global, float* Mdm2_P_Ub_aux, float* Mdm2_P_Ub_global, float* Mdm2_P_Ub2_aux, float* Mdm2_P_Ub2_global, float* Mdm2_P_Ub3_aux, float* Mdm2_P_Ub3_global, float* Mdm2_P_Ub4_aux, float* Mdm2_P_Ub4_global, float* p53_Ub4_Proteasome_aux, float* p53_Ub4_Proteasome_global, float* Mdm2_Ub4_Proteasome_aux, float* Mdm2_Ub4_Proteasome_global, float* Mdm2_P_Ub4_Proteasome_aux, float* Mdm2_P_Ub4_Proteasome_global, float* GSK3b_aux, float* GSK3b_global, float* GSK3b_p53_aux, float* GSK3b_p53_global, float* GSK3b_p53_P_aux, float* GSK3b_p53_P_global, float* Abeta_aux, float* Abeta_global, float* AggAbeta_Proteasome_aux, float* AggAbeta_Proteasome_global, float* AbetaPlaque_aux, float* AbetaPlaque_global, float* Tau_aux, float* Tau_global, float* Tau_P1_aux, float* Tau_P1_global, float* Tau_P2_aux, float* Tau_P2_global, float* MT_Tau_aux, float* MT_Tau_global, float* AggTau_aux, float* AggTau_global, float* AggTau_Proteasome_aux, float* AggTau_Proteasome_global, float* Proteasome_Tau_aux, float* Proteasome_Tau_global, float* PP1_aux, float* PP1_global, float* NFT_aux, float* NFT_global, float* ATP_aux, float* ATP_global, float* ADP_aux, float* ADP_global, float* AMP_aux, float* AMP_global, float* AbetaDimer_aux, float* AbetaDimer_global, float* AbetaPlaque_GliaA_aux, float* AbetaPlaque_GliaA_global, float* GliaI_aux, float* GliaI_global, float* GliaM1_aux, float* GliaM1_global, float* GliaM2_aux, float* GliaM2_global, float* GliaA_aux, float* GliaA_global, float* antiAb_aux, float* antiAb_global, float* Abeta_antiAb_aux, float* Abeta_antiAb_global, float* AbetaDimer_antiAb_aux, float* AbetaDimer_antiAb_global, float* degAbetaGlia_aux, float* degAbetaGlia_global, float* disaggPlaque1_aux, float* disaggPlaque1_global, float* disaggPlaque2_aux, float* disaggPlaque2_global, float* Source_aux, float* Source_global, float* Sink_aux, float* Sink_global) {
	int reaction, stepCount = 0;
	int indexMin, indexMax;
	float time = numberOfExecutions * segmentSize;
	float sum_p, timeStep, random;
	float cummulative_p[112];
	int triggerEvent0 = 0;
	if (time >= 345600) { triggerEvent0 = 1; }
	float species[69];
	if (numberOfExecutions == 0) {
		species[0] = *Mdm2_aux;
	}
	else {
		species[0] = Mdm2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[1] = *p53_aux;
	}
	else {
		species[1] = p53_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[2] = *Mdm2_p53_aux;
	}
	else {
		species[2] = Mdm2_p53_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[3] = *Mdm2_mRNA_aux;
	}
	else {
		species[3] = Mdm2_mRNA_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[4] = *p53_mRNA_aux;
	}
	else {
		species[4] = p53_mRNA_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[5] = *ATMA_aux;
	}
	else {
		species[5] = ATMA_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[6] = *ATMI_aux;
	}
	else {
		species[6] = ATMI_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[7] = *p53_P_aux;
	}
	else {
		species[7] = p53_P_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[8] = *Mdm2_P_aux;
	}
	else {
		species[8] = Mdm2_P_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[9] = *IR_aux;
	}
	else {
		species[9] = IR_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[10] = *ROS_aux;
	}
	else {
		species[10] = ROS_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[11] = *damDNA_aux;
	}
	else {
		species[11] = damDNA_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[12] = *E1_aux;
	}
	else {
		species[12] = E1_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[13] = *E2_aux;
	}
	else {
		species[13] = E2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[14] = *E1_Ub_aux;
	}
	else {
		species[14] = E1_Ub_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[15] = *E2_Ub_aux;
	}
	else {
		species[15] = E2_Ub_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[16] = *Proteasome_aux;
	}
	else {
		species[16] = Proteasome_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[17] = *Ub_aux;
	}
	else {
		species[17] = Ub_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[18] = *p53DUB_aux;
	}
	else {
		species[18] = p53DUB_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[19] = *Mdm2DUB_aux;
	}
	else {
		species[19] = Mdm2DUB_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[20] = *DUB_aux;
	}
	else {
		species[20] = DUB_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[21] = *Mdm2_p53_Ub_aux;
	}
	else {
		species[21] = Mdm2_p53_Ub_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[22] = *Mdm2_p53_Ub2_aux;
	}
	else {
		species[22] = Mdm2_p53_Ub2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[23] = *Mdm2_p53_Ub3_aux;
	}
	else {
		species[23] = Mdm2_p53_Ub3_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[24] = *Mdm2_p53_Ub4_aux;
	}
	else {
		species[24] = Mdm2_p53_Ub4_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[25] = *Mdm2_P1_p53_Ub4_aux;
	}
	else {
		species[25] = Mdm2_P1_p53_Ub4_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[26] = *Mdm2_Ub_aux;
	}
	else {
		species[26] = Mdm2_Ub_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[27] = *Mdm2_Ub2_aux;
	}
	else {
		species[27] = Mdm2_Ub2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[28] = *Mdm2_Ub3_aux;
	}
	else {
		species[28] = Mdm2_Ub3_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[29] = *Mdm2_Ub4_aux;
	}
	else {
		species[29] = Mdm2_Ub4_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[30] = *Mdm2_P_Ub_aux;
	}
	else {
		species[30] = Mdm2_P_Ub_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[31] = *Mdm2_P_Ub2_aux;
	}
	else {
		species[31] = Mdm2_P_Ub2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[32] = *Mdm2_P_Ub3_aux;
	}
	else {
		species[32] = Mdm2_P_Ub3_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[33] = *Mdm2_P_Ub4_aux;
	}
	else {
		species[33] = Mdm2_P_Ub4_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[34] = *p53_Ub4_Proteasome_aux;
	}
	else {
		species[34] = p53_Ub4_Proteasome_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[35] = *Mdm2_Ub4_Proteasome_aux;
	}
	else {
		species[35] = Mdm2_Ub4_Proteasome_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[36] = *Mdm2_P_Ub4_Proteasome_aux;
	}
	else {
		species[36] = Mdm2_P_Ub4_Proteasome_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[37] = *GSK3b_aux;
	}
	else {
		species[37] = GSK3b_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[38] = *GSK3b_p53_aux;
	}
	else {
		species[38] = GSK3b_p53_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[39] = *GSK3b_p53_P_aux;
	}
	else {
		species[39] = GSK3b_p53_P_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[40] = *Abeta_aux;
	}
	else {
		species[40] = Abeta_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[41] = *AggAbeta_Proteasome_aux;
	}
	else {
		species[41] = AggAbeta_Proteasome_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[42] = *AbetaPlaque_aux;
	}
	else {
		species[42] = AbetaPlaque_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[43] = *Tau_aux;
	}
	else {
		species[43] = Tau_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[44] = *Tau_P1_aux;
	}
	else {
		species[44] = Tau_P1_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[45] = *Tau_P2_aux;
	}
	else {
		species[45] = Tau_P2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[46] = *MT_Tau_aux;
	}
	else {
		species[46] = MT_Tau_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[47] = *AggTau_aux;
	}
	else {
		species[47] = AggTau_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[48] = *AggTau_Proteasome_aux;
	}
	else {
		species[48] = AggTau_Proteasome_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[49] = *Proteasome_Tau_aux;
	}
	else {
		species[49] = Proteasome_Tau_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[50] = *PP1_aux;
	}
	else {
		species[50] = PP1_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[51] = *NFT_aux;
	}
	else {
		species[51] = NFT_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[52] = *ATP_aux;
	}
	else {
		species[52] = ATP_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[53] = *ADP_aux;
	}
	else {
		species[53] = ADP_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[54] = *AMP_aux;
	}
	else {
		species[54] = AMP_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[55] = *AbetaDimer_aux;
	}
	else {
		species[55] = AbetaDimer_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[56] = *AbetaPlaque_GliaA_aux;
	}
	else {
		species[56] = AbetaPlaque_GliaA_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[57] = *GliaI_aux;
	}
	else {
		species[57] = GliaI_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[58] = *GliaM1_aux;
	}
	else {
		species[58] = GliaM1_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[59] = *GliaM2_aux;
	}
	else {
		species[59] = GliaM2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[60] = *GliaA_aux;
	}
	else {
		species[60] = GliaA_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[61] = *antiAb_aux;
	}
	else {
		species[61] = antiAb_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[62] = *Abeta_antiAb_aux;
	}
	else {
		species[62] = Abeta_antiAb_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[63] = *AbetaDimer_antiAb_aux;
	}
	else {
		species[63] = AbetaDimer_antiAb_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[64] = *degAbetaGlia_aux;
	}
	else {
		species[64] = degAbetaGlia_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[65] = *disaggPlaque1_aux;
	}
	else {
		species[65] = disaggPlaque1_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[66] = *disaggPlaque2_aux;
	}
	else {
		species[66] = disaggPlaque2_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[67] = *Source_aux;
	}
	else {
		species[67] = Source_global[threadIdx.x];
	}
	if (numberOfExecutions == 0) {
		species[68] = *Sink_aux;
	}
	else {
		species[68] = Sink_global[threadIdx.x];
	}
	int reactionsSpecies[112][5];
	int reactionsValues[112][5];
	for (int i = 0; i < 112; i++) {
		for (int j = 0; j < 5; j++) {
			reactionsSpecies[i][j] = -1;
			reactionsValues[i][j] = 0;
		}
	}
	reactionsSpecies[0][0] = p53_mRNA_id;
	reactionsValues[0][0] = 1.0000000000;
	reactionsSpecies[1][0] = p53_mRNA_id;
	reactionsValues[1][0] = -1.0000000000;
	reactionsSpecies[2][0] = Mdm2_mRNA_id;
	reactionsValues[2][0] = -1.0000000000;
	reactionsSpecies[2][1] = Mdm2_mRNA_id;
	reactionsValues[2][1] = 1.0000000000;
	reactionsSpecies[2][2] = Mdm2_id;
	reactionsValues[2][2] = 1.0000000000;
	reactionsSpecies[3][0] = p53_id;
	reactionsValues[3][0] = -1.0000000000;
	reactionsSpecies[3][1] = p53_id;
	reactionsValues[3][1] = 1.0000000000;
	reactionsSpecies[3][2] = Mdm2_mRNA_id;
	reactionsValues[3][2] = 1.0000000000;
	reactionsSpecies[4][0] = p53_P_id;
	reactionsValues[4][0] = -1.0000000000;
	reactionsSpecies[4][1] = p53_P_id;
	reactionsValues[4][1] = 1.0000000000;
	reactionsSpecies[4][2] = Mdm2_mRNA_id;
	reactionsValues[4][2] = 1.0000000000;
	reactionsSpecies[5][0] = GSK3b_p53_id;
	reactionsValues[5][0] = -1.0000000000;
	reactionsSpecies[5][1] = GSK3b_p53_id;
	reactionsValues[5][1] = 1.0000000000;
	reactionsSpecies[5][2] = Mdm2_mRNA_id;
	reactionsValues[5][2] = 1.0000000000;
	reactionsSpecies[6][0] = GSK3b_p53_P_id;
	reactionsValues[6][0] = -1.0000000000;
	reactionsSpecies[6][1] = GSK3b_p53_P_id;
	reactionsValues[6][1] = 1.0000000000;
	reactionsSpecies[6][2] = Mdm2_mRNA_id;
	reactionsValues[6][2] = 1.0000000000;
	reactionsSpecies[7][0] = Mdm2_mRNA_id;
	reactionsValues[7][0] = -1.0000000000;
	reactionsSpecies[8][0] = p53_id;
	reactionsValues[8][0] = -1.0000000000;
	reactionsSpecies[8][1] = Mdm2_id;
	reactionsValues[8][1] = -1.0000000000;
	reactionsSpecies[8][2] = Mdm2_p53_id;
	reactionsValues[8][2] = 1.0000000000;
	reactionsSpecies[9][0] = Mdm2_p53_id;
	reactionsValues[9][0] = -1.0000000000;
	reactionsSpecies[9][1] = p53_id;
	reactionsValues[9][1] = 1.0000000000;
	reactionsSpecies[9][2] = Mdm2_id;
	reactionsValues[9][2] = 1.0000000000;
	reactionsSpecies[10][0] = GSK3b_id;
	reactionsValues[10][0] = -1.0000000000;
	reactionsSpecies[10][1] = p53_id;
	reactionsValues[10][1] = -1.0000000000;
	reactionsSpecies[10][2] = GSK3b_p53_id;
	reactionsValues[10][2] = 1.0000000000;
	reactionsSpecies[11][0] = GSK3b_p53_id;
	reactionsValues[11][0] = -1.0000000000;
	reactionsSpecies[11][1] = GSK3b_id;
	reactionsValues[11][1] = 1.0000000000;
	reactionsSpecies[11][2] = p53_id;
	reactionsValues[11][2] = 1.0000000000;
	reactionsSpecies[12][0] = GSK3b_id;
	reactionsValues[12][0] = -1.0000000000;
	reactionsSpecies[12][1] = p53_P_id;
	reactionsValues[12][1] = -1.0000000000;
	reactionsSpecies[12][2] = GSK3b_p53_P_id;
	reactionsValues[12][2] = 1.0000000000;
	reactionsSpecies[13][0] = GSK3b_p53_P_id;
	reactionsValues[13][0] = -1.0000000000;
	reactionsSpecies[13][1] = GSK3b_id;
	reactionsValues[13][1] = 1.0000000000;
	reactionsSpecies[13][2] = p53_P_id;
	reactionsValues[13][2] = 1.0000000000;
	reactionsSpecies[14][0] = E1_id;
	reactionsValues[14][0] = -1.0000000000;
	reactionsSpecies[14][1] = Ub_id;
	reactionsValues[14][1] = -1.0000000000;
	reactionsSpecies[14][2] = E1_Ub_id;
	reactionsValues[14][2] = 1.0000000000;
	reactionsSpecies[15][0] = E2_id;
	reactionsValues[15][0] = -1.0000000000;
	reactionsSpecies[15][1] = E1_Ub_id;
	reactionsValues[15][1] = -1.0000000000;
	reactionsSpecies[15][2] = E2_Ub_id;
	reactionsValues[15][2] = 1.0000000000;
	reactionsSpecies[15][3] = E1_id;
	reactionsValues[15][3] = 1.0000000000;
	reactionsSpecies[16][0] = Mdm2_id;
	reactionsValues[16][0] = -1.0000000000;
	reactionsSpecies[16][1] = E2_Ub_id;
	reactionsValues[16][1] = -1.0000000000;
	reactionsSpecies[16][2] = Mdm2_Ub_id;
	reactionsValues[16][2] = 1.0000000000;
	reactionsSpecies[16][3] = E2_id;
	reactionsValues[16][3] = 1.0000000000;
	reactionsSpecies[17][0] = Mdm2_Ub_id;
	reactionsValues[17][0] = -1.0000000000;
	reactionsSpecies[17][1] = E2_Ub_id;
	reactionsValues[17][1] = -1.0000000000;
	reactionsSpecies[17][2] = Mdm2_Ub2_id;
	reactionsValues[17][2] = 1.0000000000;
	reactionsSpecies[17][3] = E2_id;
	reactionsValues[17][3] = 1.0000000000;
	reactionsSpecies[18][0] = Mdm2_Ub2_id;
	reactionsValues[18][0] = -1.0000000000;
	reactionsSpecies[18][1] = E2_Ub_id;
	reactionsValues[18][1] = -1.0000000000;
	reactionsSpecies[18][2] = Mdm2_Ub3_id;
	reactionsValues[18][2] = 1.0000000000;
	reactionsSpecies[18][3] = E2_id;
	reactionsValues[18][3] = 1.0000000000;
	reactionsSpecies[19][0] = Mdm2_Ub3_id;
	reactionsValues[19][0] = -1.0000000000;
	reactionsSpecies[19][1] = E2_Ub_id;
	reactionsValues[19][1] = -1.0000000000;
	reactionsSpecies[19][2] = Mdm2_Ub4_id;
	reactionsValues[19][2] = 1.0000000000;
	reactionsSpecies[19][3] = E2_id;
	reactionsValues[19][3] = 1.0000000000;
	reactionsSpecies[20][0] = Mdm2_Ub4_id;
	reactionsValues[20][0] = -1.0000000000;
	reactionsSpecies[20][1] = Mdm2DUB_id;
	reactionsValues[20][1] = -1.0000000000;
	reactionsSpecies[20][2] = Mdm2_Ub3_id;
	reactionsValues[20][2] = 1.0000000000;
	reactionsSpecies[20][3] = Mdm2DUB_id;
	reactionsValues[20][3] = 1.0000000000;
	reactionsSpecies[20][4] = Ub_id;
	reactionsValues[20][4] = 1.0000000000;
	reactionsSpecies[21][0] = Mdm2_Ub3_id;
	reactionsValues[21][0] = -1.0000000000;
	reactionsSpecies[21][1] = Mdm2DUB_id;
	reactionsValues[21][1] = -1.0000000000;
	reactionsSpecies[21][2] = Mdm2_Ub2_id;
	reactionsValues[21][2] = 1.0000000000;
	reactionsSpecies[21][3] = Mdm2DUB_id;
	reactionsValues[21][3] = 1.0000000000;
	reactionsSpecies[21][4] = Ub_id;
	reactionsValues[21][4] = 1.0000000000;
	reactionsSpecies[22][0] = Mdm2_Ub2_id;
	reactionsValues[22][0] = -1.0000000000;
	reactionsSpecies[22][1] = Mdm2DUB_id;
	reactionsValues[22][1] = -1.0000000000;
	reactionsSpecies[22][2] = Mdm2_Ub_id;
	reactionsValues[22][2] = 1.0000000000;
	reactionsSpecies[22][3] = Mdm2DUB_id;
	reactionsValues[22][3] = 1.0000000000;
	reactionsSpecies[22][4] = Ub_id;
	reactionsValues[22][4] = 1.0000000000;
	reactionsSpecies[23][0] = Mdm2_Ub_id;
	reactionsValues[23][0] = -1.0000000000;
	reactionsSpecies[23][1] = Mdm2DUB_id;
	reactionsValues[23][1] = -1.0000000000;
	reactionsSpecies[23][2] = Mdm2_id;
	reactionsValues[23][2] = 1.0000000000;
	reactionsSpecies[23][3] = Mdm2DUB_id;
	reactionsValues[23][3] = 1.0000000000;
	reactionsSpecies[23][4] = Ub_id;
	reactionsValues[23][4] = 1.0000000000;
	reactionsSpecies[24][0] = Mdm2_Ub4_id;
	reactionsValues[24][0] = -1.0000000000;
	reactionsSpecies[24][1] = Proteasome_id;
	reactionsValues[24][1] = -1.0000000000;
	reactionsSpecies[24][2] = Mdm2_Ub4_Proteasome_id;
	reactionsValues[24][2] = 1.0000000000;
	reactionsSpecies[25][0] = Mdm2_Ub4_Proteasome_id;
	reactionsValues[25][0] = -1.0000000000;
	reactionsSpecies[25][1] = Proteasome_id;
	reactionsValues[25][1] = 1.0000000000;
	reactionsSpecies[25][2] = Ub_id;
	reactionsValues[25][2] = 4.0000000000;
	reactionsSpecies[26][0] = p53_mRNA_id;
	reactionsValues[26][0] = -1.0000000000;
	reactionsSpecies[26][1] = p53_id;
	reactionsValues[26][1] = 1.0000000000;
	reactionsSpecies[26][2] = p53_mRNA_id;
	reactionsValues[26][2] = 1.0000000000;
	reactionsSpecies[27][0] = E2_Ub_id;
	reactionsValues[27][0] = -1.0000000000;
	reactionsSpecies[27][1] = Mdm2_p53_id;
	reactionsValues[27][1] = -1.0000000000;
	reactionsSpecies[27][2] = Mdm2_p53_Ub_id;
	reactionsValues[27][2] = 1.0000000000;
	reactionsSpecies[27][3] = E2_id;
	reactionsValues[27][3] = 1.0000000000;
	reactionsSpecies[28][0] = Mdm2_p53_Ub_id;
	reactionsValues[28][0] = -1.0000000000;
	reactionsSpecies[28][1] = E2_Ub_id;
	reactionsValues[28][1] = -1.0000000000;
	reactionsSpecies[28][2] = Mdm2_p53_Ub2_id;
	reactionsValues[28][2] = 1.0000000000;
	reactionsSpecies[28][3] = E2_id;
	reactionsValues[28][3] = 1.0000000000;
	reactionsSpecies[29][0] = Mdm2_p53_Ub2_id;
	reactionsValues[29][0] = -1.0000000000;
	reactionsSpecies[29][1] = E2_Ub_id;
	reactionsValues[29][1] = -1.0000000000;
	reactionsSpecies[29][2] = Mdm2_p53_Ub3_id;
	reactionsValues[29][2] = 1.0000000000;
	reactionsSpecies[29][3] = E2_id;
	reactionsValues[29][3] = 1.0000000000;
	reactionsSpecies[30][0] = Mdm2_p53_Ub3_id;
	reactionsValues[30][0] = -1.0000000000;
	reactionsSpecies[30][1] = E2_Ub_id;
	reactionsValues[30][1] = -1.0000000000;
	reactionsSpecies[30][2] = Mdm2_p53_Ub4_id;
	reactionsValues[30][2] = 1.0000000000;
	reactionsSpecies[30][3] = E2_id;
	reactionsValues[30][3] = 1.0000000000;
	reactionsSpecies[31][0] = Mdm2_p53_Ub4_id;
	reactionsValues[31][0] = -1.0000000000;
	reactionsSpecies[31][1] = p53DUB_id;
	reactionsValues[31][1] = -1.0000000000;
	reactionsSpecies[31][2] = Mdm2_p53_Ub3_id;
	reactionsValues[31][2] = 1.0000000000;
	reactionsSpecies[31][3] = p53DUB_id;
	reactionsValues[31][3] = 1.0000000000;
	reactionsSpecies[31][4] = Ub_id;
	reactionsValues[31][4] = 1.0000000000;
	reactionsSpecies[32][0] = Mdm2_p53_Ub3_id;
	reactionsValues[32][0] = -1.0000000000;
	reactionsSpecies[32][1] = p53DUB_id;
	reactionsValues[32][1] = -1.0000000000;
	reactionsSpecies[32][2] = Mdm2_p53_Ub2_id;
	reactionsValues[32][2] = 1.0000000000;
	reactionsSpecies[32][3] = p53DUB_id;
	reactionsValues[32][3] = 1.0000000000;
	reactionsSpecies[32][4] = Ub_id;
	reactionsValues[32][4] = 1.0000000000;
	reactionsSpecies[33][0] = Mdm2_p53_Ub2_id;
	reactionsValues[33][0] = -1.0000000000;
	reactionsSpecies[33][1] = p53DUB_id;
	reactionsValues[33][1] = -1.0000000000;
	reactionsSpecies[33][2] = Mdm2_p53_Ub_id;
	reactionsValues[33][2] = 1.0000000000;
	reactionsSpecies[33][3] = p53DUB_id;
	reactionsValues[33][3] = 1.0000000000;
	reactionsSpecies[33][4] = Ub_id;
	reactionsValues[33][4] = 1.0000000000;
	reactionsSpecies[34][0] = Mdm2_p53_Ub_id;
	reactionsValues[34][0] = -1.0000000000;
	reactionsSpecies[34][1] = p53DUB_id;
	reactionsValues[34][1] = -1.0000000000;
	reactionsSpecies[34][2] = Mdm2_p53_id;
	reactionsValues[34][2] = 1.0000000000;
	reactionsSpecies[34][3] = p53DUB_id;
	reactionsValues[34][3] = 1.0000000000;
	reactionsSpecies[34][4] = Ub_id;
	reactionsValues[34][4] = 1.0000000000;
	reactionsSpecies[35][0] = Mdm2_p53_Ub4_id;
	reactionsValues[35][0] = -1.0000000000;
	reactionsSpecies[35][1] = GSK3b_id;
	reactionsValues[35][1] = -1.0000000000;
	reactionsSpecies[35][2] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[35][2] = 1.0000000000;
	reactionsSpecies[35][3] = GSK3b_id;
	reactionsValues[35][3] = 1.0000000000;
	reactionsSpecies[36][0] = Mdm2_p53_Ub4_id;
	reactionsValues[36][0] = -1.0000000000;
	reactionsSpecies[36][1] = GSK3b_p53_id;
	reactionsValues[36][1] = -1.0000000000;
	reactionsSpecies[36][2] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[36][2] = 1.0000000000;
	reactionsSpecies[36][3] = GSK3b_p53_id;
	reactionsValues[36][3] = 1.0000000000;
	reactionsSpecies[37][0] = Mdm2_p53_Ub4_id;
	reactionsValues[37][0] = -1.0000000000;
	reactionsSpecies[37][1] = GSK3b_p53_P_id;
	reactionsValues[37][1] = -1.0000000000;
	reactionsSpecies[37][2] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[37][2] = 1.0000000000;
	reactionsSpecies[37][3] = GSK3b_p53_P_id;
	reactionsValues[37][3] = 1.0000000000;
	reactionsSpecies[38][0] = Mdm2_P1_p53_Ub4_id;
	reactionsValues[38][0] = -1.0000000000;
	reactionsSpecies[38][1] = Proteasome_id;
	reactionsValues[38][1] = -1.0000000000;
	reactionsSpecies[38][2] = p53_Ub4_Proteasome_id;
	reactionsValues[38][2] = 1.0000000000;
	reactionsSpecies[38][3] = Mdm2_id;
	reactionsValues[38][3] = 1.0000000000;
	reactionsSpecies[39][0] = p53_Ub4_Proteasome_id;
	reactionsValues[39][0] = -1.0000000000;
	reactionsSpecies[39][1] = Ub_id;
	reactionsValues[39][1] = 4.0000000000;
	reactionsSpecies[39][2] = Proteasome_id;
	reactionsValues[39][2] = 1.0000000000;
	reactionsSpecies[40][0] = Tau_id;
	reactionsValues[40][0] = -1.0000000000;
	reactionsSpecies[40][1] = MT_Tau_id;
	reactionsValues[40][1] = 1.0000000000;
	reactionsSpecies[41][0] = MT_Tau_id;
	reactionsValues[41][0] = -1.0000000000;
	reactionsSpecies[41][1] = Tau_id;
	reactionsValues[41][1] = 1.0000000000;
	reactionsSpecies[42][0] = GSK3b_p53_id;
	reactionsValues[42][0] = -1.0000000000;
	reactionsSpecies[42][1] = Tau_id;
	reactionsValues[42][1] = -1.0000000000;
	reactionsSpecies[42][2] = GSK3b_p53_id;
	reactionsValues[42][2] = 1.0000000000;
	reactionsSpecies[42][3] = Tau_P1_id;
	reactionsValues[42][3] = 1.0000000000;
	reactionsSpecies[43][0] = GSK3b_p53_id;
	reactionsValues[43][0] = -1.0000000000;
	reactionsSpecies[43][1] = Tau_P1_id;
	reactionsValues[43][1] = -1.0000000000;
	reactionsSpecies[43][2] = GSK3b_p53_id;
	reactionsValues[43][2] = 1.0000000000;
	reactionsSpecies[43][3] = Tau_P2_id;
	reactionsValues[43][3] = 1.0000000000;
	reactionsSpecies[44][0] = GSK3b_p53_P_id;
	reactionsValues[44][0] = -1.0000000000;
	reactionsSpecies[44][1] = Tau_id;
	reactionsValues[44][1] = -1.0000000000;
	reactionsSpecies[44][2] = GSK3b_p53_P_id;
	reactionsValues[44][2] = 1.0000000000;
	reactionsSpecies[44][3] = Tau_P1_id;
	reactionsValues[44][3] = 1.0000000000;
	reactionsSpecies[45][0] = GSK3b_p53_P_id;
	reactionsValues[45][0] = -1.0000000000;
	reactionsSpecies[45][1] = Tau_P1_id;
	reactionsValues[45][1] = -1.0000000000;
	reactionsSpecies[45][2] = GSK3b_p53_P_id;
	reactionsValues[45][2] = 1.0000000000;
	reactionsSpecies[45][3] = Tau_P2_id;
	reactionsValues[45][3] = 1.0000000000;
	reactionsSpecies[46][0] = GSK3b_id;
	reactionsValues[46][0] = -1.0000000000;
	reactionsSpecies[46][1] = Tau_id;
	reactionsValues[46][1] = -1.0000000000;
	reactionsSpecies[46][2] = GSK3b_id;
	reactionsValues[46][2] = 1.0000000000;
	reactionsSpecies[46][3] = Tau_P1_id;
	reactionsValues[46][3] = 1.0000000000;
	reactionsSpecies[47][0] = GSK3b_id;
	reactionsValues[47][0] = -1.0000000000;
	reactionsSpecies[47][1] = Tau_P1_id;
	reactionsValues[47][1] = -1.0000000000;
	reactionsSpecies[47][2] = GSK3b_id;
	reactionsValues[47][2] = 1.0000000000;
	reactionsSpecies[47][3] = Tau_P2_id;
	reactionsValues[47][3] = 1.0000000000;
	reactionsSpecies[48][0] = Tau_P2_id;
	reactionsValues[48][0] = -1.0000000000;
	reactionsSpecies[48][1] = PP1_id;
	reactionsValues[48][1] = -1.0000000000;
	reactionsSpecies[48][2] = Tau_P1_id;
	reactionsValues[48][2] = 1.0000000000;
	reactionsSpecies[48][3] = PP1_id;
	reactionsValues[48][3] = 1.0000000000;
	reactionsSpecies[49][0] = Tau_P1_id;
	reactionsValues[49][0] = -1.0000000000;
	reactionsSpecies[49][1] = PP1_id;
	reactionsValues[49][1] = -1.0000000000;
	reactionsSpecies[49][2] = Tau_id;
	reactionsValues[49][2] = 1.0000000000;
	reactionsSpecies[49][3] = PP1_id;
	reactionsValues[49][3] = 1.0000000000;
	reactionsSpecies[50][0] = Tau_P1_id;
	reactionsValues[50][0] = -2.0000000000;
	reactionsSpecies[50][1] = AggTau_id;
	reactionsValues[50][1] = 2.0000000000;
	reactionsSpecies[51][0] = Tau_P1_id;
	reactionsValues[51][0] = -1.0000000000;
	reactionsSpecies[51][1] = AggTau_id;
	reactionsValues[51][1] = -1.0000000000;
	reactionsSpecies[51][2] = AggTau_id;
	reactionsValues[51][2] = 2.0000000000;
	reactionsSpecies[52][0] = Tau_P2_id;
	reactionsValues[52][0] = -2.0000000000;
	reactionsSpecies[52][1] = AggTau_id;
	reactionsValues[52][1] = 2.0000000000;
	reactionsSpecies[53][0] = Tau_P2_id;
	reactionsValues[53][0] = -1.0000000000;
	reactionsSpecies[53][1] = AggTau_id;
	reactionsValues[53][1] = -1.0000000000;
	reactionsSpecies[53][2] = AggTau_id;
	reactionsValues[53][2] = 2.0000000000;
	reactionsSpecies[54][0] = Tau_id;
	reactionsValues[54][0] = -2.0000000000;
	reactionsSpecies[54][1] = AggTau_id;
	reactionsValues[54][1] = 2.0000000000;
	reactionsSpecies[55][0] = Tau_id;
	reactionsValues[55][0] = -1.0000000000;
	reactionsSpecies[55][1] = AggTau_id;
	reactionsValues[55][1] = -1.0000000000;
	reactionsSpecies[55][2] = AggTau_id;
	reactionsValues[55][2] = 2.0000000000;
	reactionsSpecies[56][0] = AggTau_id;
	reactionsValues[56][0] = -2.0000000000;
	reactionsSpecies[56][1] = NFT_id;
	reactionsValues[56][1] = 2.0000000000;
	reactionsSpecies[57][0] = AggTau_id;
	reactionsValues[57][0] = -1.0000000000;
	reactionsSpecies[57][1] = NFT_id;
	reactionsValues[57][1] = -1.0000000000;
	reactionsSpecies[57][2] = NFT_id;
	reactionsValues[57][2] = 2.0000000000;
	reactionsSpecies[58][0] = AggTau_id;
	reactionsValues[58][0] = -1.0000000000;
	reactionsSpecies[58][1] = Proteasome_id;
	reactionsValues[58][1] = -1.0000000000;
	reactionsSpecies[58][2] = AggTau_Proteasome_id;
	reactionsValues[58][2] = 1.0000000000;
	reactionsSpecies[59][0] = Abeta_id;
	reactionsValues[59][0] = 1.0000000000;
	reactionsSpecies[60][0] = GSK3b_p53_id;
	reactionsValues[60][0] = -1.0000000000;
	reactionsSpecies[60][1] = Abeta_id;
	reactionsValues[60][1] = 1.0000000000;
	reactionsSpecies[60][2] = GSK3b_p53_id;
	reactionsValues[60][2] = 1.0000000000;
	reactionsSpecies[61][0] = GSK3b_p53_P_id;
	reactionsValues[61][0] = -1.0000000000;
	reactionsSpecies[61][1] = Abeta_id;
	reactionsValues[61][1] = 1.0000000000;
	reactionsSpecies[61][2] = GSK3b_p53_P_id;
	reactionsValues[61][2] = 1.0000000000;
	reactionsSpecies[62][0] = AbetaDimer_id;
	reactionsValues[62][0] = -1.0000000000;
	reactionsSpecies[62][1] = Proteasome_id;
	reactionsValues[62][1] = -1.0000000000;
	reactionsSpecies[62][2] = AggAbeta_Proteasome_id;
	reactionsValues[62][2] = 1.0000000000;
	reactionsSpecies[63][0] = Abeta_id;
	reactionsValues[63][0] = -1.0000000000;
	reactionsSpecies[64][0] = Abeta_id;
	reactionsValues[64][0] = -1.0000000000;
	reactionsSpecies[64][1] = p53_mRNA_id;
	reactionsValues[64][1] = 1.0000000000;
	reactionsSpecies[64][2] = Abeta_id;
	reactionsValues[64][2] = 1.0000000000;
	reactionsSpecies[65][0] = IR_id;
	reactionsValues[65][0] = -1.0000000000;
	reactionsSpecies[65][1] = IR_id;
	reactionsValues[65][1] = 1.0000000000;
	reactionsSpecies[65][2] = damDNA_id;
	reactionsValues[65][2] = 1.0000000000;
	reactionsSpecies[66][0] = damDNA_id;
	reactionsValues[66][0] = -1.0000000000;
	reactionsSpecies[67][0] = damDNA_id;
	reactionsValues[67][0] = -1.0000000000;
	reactionsSpecies[67][1] = ATMI_id;
	reactionsValues[67][1] = -1.0000000000;
	reactionsSpecies[67][2] = damDNA_id;
	reactionsValues[67][2] = 1.0000000000;
	reactionsSpecies[67][3] = ATMA_id;
	reactionsValues[67][3] = 1.0000000000;
	reactionsSpecies[68][0] = p53_id;
	reactionsValues[68][0] = -1.0000000000;
	reactionsSpecies[68][1] = ATMA_id;
	reactionsValues[68][1] = -1.0000000000;
	reactionsSpecies[68][2] = p53_P_id;
	reactionsValues[68][2] = 1.0000000000;
	reactionsSpecies[68][3] = ATMA_id;
	reactionsValues[68][3] = 1.0000000000;
	reactionsSpecies[69][0] = p53_P_id;
	reactionsValues[69][0] = -1.0000000000;
	reactionsSpecies[69][1] = p53_id;
	reactionsValues[69][1] = 1.0000000000;
	reactionsSpecies[70][0] = Mdm2_id;
	reactionsValues[70][0] = -1.0000000000;
	reactionsSpecies[70][1] = ATMA_id;
	reactionsValues[70][1] = -1.0000000000;
	reactionsSpecies[70][2] = Mdm2_P_id;
	reactionsValues[70][2] = 1.0000000000;
	reactionsSpecies[70][3] = ATMA_id;
	reactionsValues[70][3] = 1.0000000000;
	reactionsSpecies[71][0] = Mdm2_P_id;
	reactionsValues[71][0] = -1.0000000000;
	reactionsSpecies[71][1] = Mdm2_id;
	reactionsValues[71][1] = 1.0000000000;
	reactionsSpecies[72][0] = Mdm2_P_id;
	reactionsValues[72][0] = -1.0000000000;
	reactionsSpecies[72][1] = E2_Ub_id;
	reactionsValues[72][1] = -1.0000000000;
	reactionsSpecies[72][2] = Mdm2_P_Ub_id;
	reactionsValues[72][2] = 1.0000000000;
	reactionsSpecies[72][3] = E2_id;
	reactionsValues[72][3] = 1.0000000000;
	reactionsSpecies[73][0] = Mdm2_P_Ub_id;
	reactionsValues[73][0] = -1.0000000000;
	reactionsSpecies[73][1] = E2_Ub_id;
	reactionsValues[73][1] = -1.0000000000;
	reactionsSpecies[73][2] = Mdm2_P_Ub2_id;
	reactionsValues[73][2] = 1.0000000000;
	reactionsSpecies[73][3] = E2_id;
	reactionsValues[73][3] = 1.0000000000;
	reactionsSpecies[74][0] = Mdm2_P_Ub2_id;
	reactionsValues[74][0] = -1.0000000000;
	reactionsSpecies[74][1] = E2_Ub_id;
	reactionsValues[74][1] = -1.0000000000;
	reactionsSpecies[74][2] = Mdm2_P_Ub3_id;
	reactionsValues[74][2] = 1.0000000000;
	reactionsSpecies[74][3] = E2_id;
	reactionsValues[74][3] = 1.0000000000;
	reactionsSpecies[75][0] = Mdm2_P_Ub3_id;
	reactionsValues[75][0] = -1.0000000000;
	reactionsSpecies[75][1] = E2_Ub_id;
	reactionsValues[75][1] = -1.0000000000;
	reactionsSpecies[75][2] = Mdm2_P_Ub4_id;
	reactionsValues[75][2] = 1.0000000000;
	reactionsSpecies[75][3] = E2_id;
	reactionsValues[75][3] = 1.0000000000;
	reactionsSpecies[76][0] = Mdm2_P_Ub4_id;
	reactionsValues[76][0] = -1.0000000000;
	reactionsSpecies[76][1] = Mdm2DUB_id;
	reactionsValues[76][1] = -1.0000000000;
	reactionsSpecies[76][2] = Mdm2_P_Ub3_id;
	reactionsValues[76][2] = 1.0000000000;
	reactionsSpecies[76][3] = Mdm2DUB_id;
	reactionsValues[76][3] = 1.0000000000;
	reactionsSpecies[76][4] = Ub_id;
	reactionsValues[76][4] = 1.0000000000;
	reactionsSpecies[77][0] = Mdm2_P_Ub3_id;
	reactionsValues[77][0] = -1.0000000000;
	reactionsSpecies[77][1] = Mdm2DUB_id;
	reactionsValues[77][1] = -1.0000000000;
	reactionsSpecies[77][2] = Mdm2_P_Ub2_id;
	reactionsValues[77][2] = 1.0000000000;
	reactionsSpecies[77][3] = Mdm2DUB_id;
	reactionsValues[77][3] = 1.0000000000;
	reactionsSpecies[77][4] = Ub_id;
	reactionsValues[77][4] = 1.0000000000;
	reactionsSpecies[78][0] = Mdm2_P_Ub2_id;
	reactionsValues[78][0] = -1.0000000000;
	reactionsSpecies[78][1] = Mdm2DUB_id;
	reactionsValues[78][1] = -1.0000000000;
	reactionsSpecies[78][2] = Mdm2_P_Ub_id;
	reactionsValues[78][2] = 1.0000000000;
	reactionsSpecies[78][3] = Mdm2DUB_id;
	reactionsValues[78][3] = 1.0000000000;
	reactionsSpecies[78][4] = Ub_id;
	reactionsValues[78][4] = 1.0000000000;
	reactionsSpecies[79][0] = Mdm2_P_Ub_id;
	reactionsValues[79][0] = -1.0000000000;
	reactionsSpecies[79][1] = Mdm2DUB_id;
	reactionsValues[79][1] = -1.0000000000;
	reactionsSpecies[79][2] = Mdm2_P_id;
	reactionsValues[79][2] = 1.0000000000;
	reactionsSpecies[79][3] = Mdm2DUB_id;
	reactionsValues[79][3] = 1.0000000000;
	reactionsSpecies[79][4] = Ub_id;
	reactionsValues[79][4] = 1.0000000000;
	reactionsSpecies[80][0] = Mdm2_P_Ub4_id;
	reactionsValues[80][0] = -1.0000000000;
	reactionsSpecies[80][1] = Proteasome_id;
	reactionsValues[80][1] = -1.0000000000;
	reactionsSpecies[80][2] = Mdm2_P_Ub4_Proteasome_id;
	reactionsValues[80][2] = 1.0000000000;
	reactionsSpecies[81][0] = Mdm2_P_Ub4_Proteasome_id;
	reactionsValues[81][0] = -1.0000000000;
	reactionsSpecies[81][1] = Proteasome_id;
	reactionsValues[81][1] = 1.0000000000;
	reactionsSpecies[81][2] = Ub_id;
	reactionsValues[81][2] = 4.0000000000;
	reactionsSpecies[82][0] = ATMA_id;
	reactionsValues[82][0] = -1.0000000000;
	reactionsSpecies[82][1] = ATMI_id;
	reactionsValues[82][1] = 1.0000000000;
	reactionsSpecies[83][0] = Abeta_id;
	reactionsValues[83][0] = -1.0000000000;
	reactionsSpecies[83][1] = Abeta_id;
	reactionsValues[83][1] = 1.0000000000;
	reactionsSpecies[83][2] = ROS_id;
	reactionsValues[83][2] = 1.0000000000;
	reactionsSpecies[84][0] = AbetaPlaque_id;
	reactionsValues[84][0] = -1.0000000000;
	reactionsSpecies[84][1] = AbetaPlaque_id;
	reactionsValues[84][1] = 1.0000000000;
	reactionsSpecies[84][2] = ROS_id;
	reactionsValues[84][2] = 1.0000000000;
	reactionsSpecies[85][0] = AggAbeta_Proteasome_id;
	reactionsValues[85][0] = -1.0000000000;
	reactionsSpecies[85][1] = AggAbeta_Proteasome_id;
	reactionsValues[85][1] = 1.0000000000;
	reactionsSpecies[85][2] = ROS_id;
	reactionsValues[85][2] = 1.0000000000;
	reactionsSpecies[86][0] = ROS_id;
	reactionsValues[86][0] = -1.0000000000;
	reactionsSpecies[86][1] = ROS_id;
	reactionsValues[86][1] = 1.0000000000;
	reactionsSpecies[86][2] = damDNA_id;
	reactionsValues[86][2] = 1.0000000000;
	reactionsSpecies[87][0] = Tau_id;
	reactionsValues[87][0] = 1.0000000000;
	reactionsSpecies[88][0] = Tau_id;
	reactionsValues[88][0] = -1.0000000000;
	reactionsSpecies[88][1] = Proteasome_id;
	reactionsValues[88][1] = -1.0000000000;
	reactionsSpecies[88][2] = Proteasome_Tau_id;
	reactionsValues[88][2] = 1.0000000000;
	reactionsSpecies[89][0] = Proteasome_Tau_id;
	reactionsValues[89][0] = -1.0000000000;
	reactionsSpecies[89][1] = Proteasome_id;
	reactionsValues[89][1] = 1.0000000000;
	reactionsSpecies[90][0] = Abeta_id;
	reactionsValues[90][0] = -2.0000000000;
	reactionsSpecies[90][1] = AbetaDimer_id;
	reactionsValues[90][1] = 1.0000000000;
	reactionsSpecies[91][0] = AbetaDimer_id;
	reactionsValues[91][0] = -2.0000000000;
	reactionsSpecies[91][1] = AbetaPlaque_id;
	reactionsValues[91][1] = 1.0000000000;
	reactionsSpecies[92][0] = AbetaDimer_id;
	reactionsValues[92][0] = -1.0000000000;
	reactionsSpecies[92][1] = AbetaPlaque_id;
	reactionsValues[92][1] = -1.0000000000;
	reactionsSpecies[92][2] = AbetaPlaque_id;
	reactionsValues[92][2] = 2.0000000000;
	reactionsSpecies[93][0] = AbetaDimer_id;
	reactionsValues[93][0] = -1.0000000000;
	reactionsSpecies[93][1] = Abeta_id;
	reactionsValues[93][1] = 2.0000000000;
	reactionsSpecies[94][0] = AbetaPlaque_id;
	reactionsValues[94][0] = -1.0000000000;
	reactionsSpecies[94][1] = AbetaDimer_id;
	reactionsValues[94][1] = 1.0000000000;
	reactionsSpecies[94][2] = disaggPlaque1_id;
	reactionsValues[94][2] = 1.0000000000;
	reactionsSpecies[95][0] = AbetaPlaque_id;
	reactionsValues[95][0] = -1.0000000000;
	reactionsSpecies[95][1] = antiAb_id;
	reactionsValues[95][1] = -1.0000000000;
	reactionsSpecies[95][2] = AbetaDimer_id;
	reactionsValues[95][2] = 1.0000000000;
	reactionsSpecies[95][3] = antiAb_id;
	reactionsValues[95][3] = 1.0000000000;
	reactionsSpecies[95][4] = disaggPlaque2_id;
	reactionsValues[95][4] = 1.0000000000;
	reactionsSpecies[96][0] = Abeta_id;
	reactionsValues[96][0] = -1.0000000000;
	reactionsSpecies[96][1] = antiAb_id;
	reactionsValues[96][1] = -1.0000000000;
	reactionsSpecies[96][2] = Abeta_antiAb_id;
	reactionsValues[96][2] = 1.0000000000;
	reactionsSpecies[97][0] = AbetaDimer_id;
	reactionsValues[97][0] = -1.0000000000;
	reactionsSpecies[97][1] = antiAb_id;
	reactionsValues[97][1] = -1.0000000000;
	reactionsSpecies[97][2] = AbetaDimer_antiAb_id;
	reactionsValues[97][2] = 1.0000000000;
	reactionsSpecies[98][0] = Abeta_antiAb_id;
	reactionsValues[98][0] = -1.0000000000;
	reactionsSpecies[98][1] = antiAb_id;
	reactionsValues[98][1] = 1.0000000000;
	reactionsSpecies[99][0] = AbetaDimer_antiAb_id;
	reactionsValues[99][0] = -1.0000000000;
	reactionsSpecies[99][1] = antiAb_id;
	reactionsValues[99][1] = 1.0000000000;
	reactionsSpecies[100][0] = GliaI_id;
	reactionsValues[100][0] = -1.0000000000;
	reactionsSpecies[100][1] = AbetaPlaque_id;
	reactionsValues[100][1] = -1.0000000000;
	reactionsSpecies[100][2] = GliaM1_id;
	reactionsValues[100][2] = 1.0000000000;
	reactionsSpecies[100][3] = AbetaPlaque_id;
	reactionsValues[100][3] = 1.0000000000;
	reactionsSpecies[101][0] = GliaM1_id;
	reactionsValues[101][0] = -1.0000000000;
	reactionsSpecies[101][1] = AbetaPlaque_id;
	reactionsValues[101][1] = -1.0000000000;
	reactionsSpecies[101][2] = GliaM2_id;
	reactionsValues[101][2] = 1.0000000000;
	reactionsSpecies[101][3] = AbetaPlaque_id;
	reactionsValues[101][3] = 1.0000000000;
	reactionsSpecies[102][0] = GliaM2_id;
	reactionsValues[102][0] = -1.0000000000;
	reactionsSpecies[102][1] = antiAb_id;
	reactionsValues[102][1] = -1.0000000000;
	reactionsSpecies[102][2] = GliaA_id;
	reactionsValues[102][2] = 1.0000000000;
	reactionsSpecies[102][3] = antiAb_id;
	reactionsValues[102][3] = 1.0000000000;
	reactionsSpecies[103][0] = GliaA_id;
	reactionsValues[103][0] = -1.0000000000;
	reactionsSpecies[103][1] = GliaM2_id;
	reactionsValues[103][1] = 1.0000000000;
	reactionsSpecies[104][0] = GliaM2_id;
	reactionsValues[104][0] = -1.0000000000;
	reactionsSpecies[104][1] = GliaM1_id;
	reactionsValues[104][1] = 1.0000000000;
	reactionsSpecies[105][0] = GliaM1_id;
	reactionsValues[105][0] = -1.0000000000;
	reactionsSpecies[105][1] = GliaI_id;
	reactionsValues[105][1] = 1.0000000000;
	reactionsSpecies[106][0] = AbetaPlaque_id;
	reactionsValues[106][0] = -1.0000000000;
	reactionsSpecies[106][1] = GliaA_id;
	reactionsValues[106][1] = -1.0000000000;
	reactionsSpecies[106][2] = AbetaPlaque_GliaA_id;
	reactionsValues[106][2] = 1.0000000000;
	reactionsSpecies[107][0] = AbetaPlaque_GliaA_id;
	reactionsValues[107][0] = -1.0000000000;
	reactionsSpecies[107][1] = AbetaPlaque_id;
	reactionsValues[107][1] = 1.0000000000;
	reactionsSpecies[107][2] = GliaA_id;
	reactionsValues[107][2] = 1.0000000000;
	reactionsSpecies[108][0] = AbetaPlaque_GliaA_id;
	reactionsValues[108][0] = -1.0000000000;
	reactionsSpecies[108][1] = GliaA_id;
	reactionsValues[108][1] = 1.0000000000;
	reactionsSpecies[108][2] = degAbetaGlia_id;
	reactionsValues[108][2] = 1.0000000000;
	reactionsSpecies[109][0] = AbetaPlaque_GliaA_id;
	reactionsValues[109][0] = -1.0000000000;
	reactionsSpecies[109][1] = AbetaPlaque_GliaA_id;
	reactionsValues[109][1] = 1.0000000000;
	reactionsSpecies[109][2] = ROS_id;
	reactionsValues[109][2] = 1.0000000000;
	reactionsSpecies[110][0] = antiAb_id;
	reactionsValues[110][0] = -1.0000000000;
	reactionsSpecies[111][0] = ROS_id;
	reactionsValues[111][0] = -1.0000000000;
	hiprandState localState = state[threadIdx.x];
	while (time < endTime && time < (numberOfExecutions + 1)*segmentSize) {
		cummulative_p[0] = ksynp53mRNA * Source;
		cummulative_p[1] = cummulative_p[0] + kdegp53mRNA * p53_mRNA;
		cummulative_p[2] = cummulative_p[1] + ksynMdm2 * Mdm2_mRNA;
		cummulative_p[3] = cummulative_p[2] + ksynMdm2mRNA * p53;
		cummulative_p[4] = cummulative_p[3] + ksynMdm2mRNA * p53_P;
		cummulative_p[5] = cummulative_p[4] + ksynMdm2mRNAGSK3bp53 * GSK3b_p53;
		cummulative_p[6] = cummulative_p[5] + ksynMdm2mRNAGSK3bp53 * GSK3b_p53_P;
		cummulative_p[7] = cummulative_p[6] + kdegMdm2mRNA * Mdm2_mRNA;
		cummulative_p[8] = cummulative_p[7] + kbinMdm2p53 * p53 * Mdm2;
		cummulative_p[9] = cummulative_p[8] + krelMdm2p53 * Mdm2_p53;
		cummulative_p[10] = cummulative_p[9] + kbinGSK3bp53 * GSK3b * p53;
		cummulative_p[11] = cummulative_p[10] + krelGSK3bp53 * GSK3b_p53;
		cummulative_p[12] = cummulative_p[11] + kbinGSK3bp53 * GSK3b * p53_P;
		cummulative_p[13] = cummulative_p[12] + krelGSK3bp53 * GSK3b_p53_P;
		cummulative_p[14] = cummulative_p[13] + kbinE1Ub * E1 * Ub * ATP / (5000 + ATP);
		cummulative_p[15] = cummulative_p[14] + kbinE2Ub * E2 * E1_Ub;
		cummulative_p[16] = cummulative_p[15] + kMdm2Ub * Mdm2 * E2_Ub;
		cummulative_p[17] = cummulative_p[16] + kMdm2PolyUb * Mdm2_Ub * E2_Ub;
		cummulative_p[18] = cummulative_p[17] + kMdm2PolyUb * Mdm2_Ub2 * E2_Ub;
		cummulative_p[19] = cummulative_p[18] + kMdm2PolyUb * Mdm2_Ub3 * E2_Ub;
		cummulative_p[20] = cummulative_p[19] + kactDUBMdm2 * Mdm2_Ub4 * Mdm2DUB;
		cummulative_p[21] = cummulative_p[20] + kactDUBMdm2 * Mdm2_Ub3 * Mdm2DUB;
		cummulative_p[22] = cummulative_p[21] + kactDUBMdm2 * Mdm2_Ub2 * Mdm2DUB;
		cummulative_p[23] = cummulative_p[22] + kactDUBMdm2 * Mdm2_Ub * Mdm2DUB;
		cummulative_p[24] = cummulative_p[23] + kbinProt * Mdm2_Ub4 * Proteasome;
		cummulative_p[25] = cummulative_p[24] + kdegMdm2 * Mdm2_Ub4_Proteasome * kproteff;
		cummulative_p[26] = cummulative_p[25] + ksynp53 * p53_mRNA;
		cummulative_p[27] = cummulative_p[26] + kp53Ub * E2_Ub * Mdm2_p53;
		cummulative_p[28] = cummulative_p[27] + kp53PolyUb * Mdm2_p53_Ub * E2_Ub;
		cummulative_p[29] = cummulative_p[28] + kp53PolyUb * Mdm2_p53_Ub2 * E2_Ub;
		cummulative_p[30] = cummulative_p[29] + kp53PolyUb * Mdm2_p53_Ub3 * E2_Ub;
		cummulative_p[31] = cummulative_p[30] + kactDUBp53 * Mdm2_p53_Ub4 * p53DUB;
		cummulative_p[32] = cummulative_p[31] + kactDUBp53 * Mdm2_p53_Ub3 * p53DUB;
		cummulative_p[33] = cummulative_p[32] + kactDUBp53 * Mdm2_p53_Ub2 * p53DUB;
		cummulative_p[34] = cummulative_p[33] + kactDUBp53 * Mdm2_p53_Ub * p53DUB;
		cummulative_p[35] = cummulative_p[34] + kphosMdm2GSK3b * Mdm2_p53_Ub4 * GSK3b;
		cummulative_p[36] = cummulative_p[35] + kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53;
		cummulative_p[37] = cummulative_p[36] + kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53_P;
		cummulative_p[38] = cummulative_p[37] + kbinProt * Mdm2_P1_p53_Ub4 * Proteasome;
		cummulative_p[39] = cummulative_p[38] + kdegp53 * kproteff * p53_Ub4_Proteasome * ATP / (5000 + ATP);
		cummulative_p[40] = cummulative_p[39] + kbinMTTau * Tau;
		cummulative_p[41] = cummulative_p[40] + krelMTTau * MT_Tau;
		cummulative_p[42] = cummulative_p[41] + kphospTauGSK3bp53 * GSK3b_p53 * Tau;
		cummulative_p[43] = cummulative_p[42] + kphospTauGSK3bp53 * GSK3b_p53 * Tau_P1;
		cummulative_p[44] = cummulative_p[43] + kphospTauGSK3bp53 * GSK3b_p53_P * Tau;
		cummulative_p[45] = cummulative_p[44] + kphospTauGSK3bp53 * GSK3b_p53_P * Tau_P1;
		cummulative_p[46] = cummulative_p[45] + kphospTauGSK3b * GSK3b * Tau;
		cummulative_p[47] = cummulative_p[46] + kphospTauGSK3b * GSK3b * Tau_P1;
		cummulative_p[48] = cummulative_p[47] + kdephospTau * Tau_P2 * PP1;
		cummulative_p[49] = cummulative_p[48] + kdephospTau * Tau_P1 * PP1;
		cummulative_p[50] = cummulative_p[49] + kaggTauP1 * Tau_P1 * (Tau_P1 - 1) * 0.5;
		cummulative_p[51] = cummulative_p[50] + kaggTauP1 * Tau_P1 * AggTau;
		cummulative_p[52] = cummulative_p[51] + kaggTauP2 * Tau_P2 * (Tau_P2 - 1) * 0.5;
		cummulative_p[53] = cummulative_p[52] + kaggTauP2 * Tau_P2 * AggTau;
		cummulative_p[54] = cummulative_p[53] + kaggTau * Tau * (Tau - 1) * 0.5;
		cummulative_p[55] = cummulative_p[54] + kaggTau * Tau * AggTau;
		cummulative_p[56] = cummulative_p[55] + ktangfor * AggTau * (AggTau - 1) * 0.5;
		cummulative_p[57] = cummulative_p[56] + ktangfor * AggTau * NFT;
		cummulative_p[58] = cummulative_p[57] + kinhibprot * AggTau * Proteasome;
		cummulative_p[59] = cummulative_p[58] + kprodAbeta * Source;
		cummulative_p[60] = cummulative_p[59] + kprodAbeta2 * GSK3b_p53;
		cummulative_p[61] = cummulative_p[60] + kprodAbeta2 * GSK3b_p53_P;
		cummulative_p[62] = cummulative_p[61] + kinhibprot * AbetaDimer * Proteasome;
		cummulative_p[63] = cummulative_p[62] + kdegAbeta * Abeta;
		cummulative_p[64] = cummulative_p[63] + ksynp53mRNAAbeta * Abeta;
		cummulative_p[65] = cummulative_p[64] + kdam * IR;
		cummulative_p[66] = cummulative_p[65] + krepair * damDNA;
		cummulative_p[67] = cummulative_p[66] + kactATM * damDNA * ATMI;
		cummulative_p[68] = cummulative_p[67] + kphosp53 * p53 * ATMA;
		cummulative_p[69] = cummulative_p[68] + kdephosp53 * p53_P;
		cummulative_p[70] = cummulative_p[69] + kphosMdm2 * Mdm2 * ATMA;
		cummulative_p[71] = cummulative_p[70] + kdephosMdm2 * Mdm2_P;
		cummulative_p[72] = cummulative_p[71] + kMdm2PUb * Mdm2_P * E2_Ub;
		cummulative_p[73] = cummulative_p[72] + kMdm2PolyUb * Mdm2_P_Ub * E2_Ub;
		cummulative_p[74] = cummulative_p[73] + kMdm2PolyUb * Mdm2_P_Ub2 * E2_Ub;
		cummulative_p[75] = cummulative_p[74] + kMdm2PolyUb * Mdm2_P_Ub3 * E2_Ub;
		cummulative_p[76] = cummulative_p[75] + kactDUBMdm2 * Mdm2_P_Ub4 * Mdm2DUB;
		cummulative_p[77] = cummulative_p[76] + kactDUBMdm2 * Mdm2_P_Ub3 * Mdm2DUB;
		cummulative_p[78] = cummulative_p[77] + kactDUBMdm2 * Mdm2_P_Ub2 * Mdm2DUB;
		cummulative_p[79] = cummulative_p[78] + kactDUBMdm2 * Mdm2_P_Ub * Mdm2DUB;
		cummulative_p[80] = cummulative_p[79] + kbinProt * Mdm2_P_Ub4 * Proteasome;
		cummulative_p[81] = cummulative_p[80] + kdegMdm2 * Mdm2_P_Ub4_Proteasome * kproteff;
		cummulative_p[82] = cummulative_p[81] + kinactATM * ATMA;
		cummulative_p[83] = cummulative_p[82] + kgenROSAbeta * Abeta;
		cummulative_p[84] = cummulative_p[83] + kgenROSPlaque * AbetaPlaque;
		cummulative_p[85] = cummulative_p[84] + kgenROSAbeta * AggAbeta_Proteasome;
		cummulative_p[86] = cummulative_p[85] + kdamROS * ROS;
		cummulative_p[87] = cummulative_p[86] + ksynTau * Source;
		cummulative_p[88] = cummulative_p[87] + kbinTauProt * Tau * Proteasome;
		cummulative_p[89] = cummulative_p[88] + kdegTau20SProt * Proteasome_Tau;
		cummulative_p[90] = cummulative_p[89] + kaggAbeta * Abeta * (Abeta - 1) * 0.5;
		cummulative_p[91] = cummulative_p[90] + kpf * AbetaDimer * (AbetaDimer - 1) * 0.5;
		cummulative_p[92] = cummulative_p[91] + kpg * AbetaDimer * pow(AbetaPlaque, 2) / (pow(kpghalf, 2) + pow(AbetaPlaque, 2));
		cummulative_p[93] = cummulative_p[92] + kdisaggAbeta * AbetaDimer;
		cummulative_p[94] = cummulative_p[93] + kdisaggAbeta1 * AbetaPlaque;
		cummulative_p[95] = cummulative_p[94] + kdisaggAbeta2 * antiAb * AbetaPlaque;
		cummulative_p[96] = cummulative_p[95] + kbinAbantiAb * Abeta * antiAb;
		cummulative_p[97] = cummulative_p[96] + kbinAbantiAb * AbetaDimer * antiAb;
		cummulative_p[98] = cummulative_p[97] + 10 * kdegAbeta * Abeta_antiAb;
		cummulative_p[99] = cummulative_p[98] + 10 * kdegAbeta * AbetaDimer_antiAb;
		cummulative_p[100] = cummulative_p[99] + kactglia1 * GliaI * AbetaPlaque;
		cummulative_p[101] = cummulative_p[100] + kactglia1 * GliaM1 * AbetaPlaque;
		cummulative_p[102] = cummulative_p[101] + kactglia2 * GliaM2 * antiAb;
		cummulative_p[103] = cummulative_p[102] + kinactglia1 * GliaA;
		cummulative_p[104] = cummulative_p[103] + kinactglia2 * GliaM2;
		cummulative_p[105] = cummulative_p[104] + kinactglia2 * GliaM1;
		cummulative_p[106] = cummulative_p[105] + kbinAbetaGlia * AbetaPlaque * GliaA;
		cummulative_p[107] = cummulative_p[106] + krelAbetaGlia * AbetaPlaque_GliaA;
		cummulative_p[108] = cummulative_p[107] + kdegAbetaGlia * AbetaPlaque_GliaA;
		cummulative_p[109] = cummulative_p[108] + kgenROSGlia * AbetaPlaque_GliaA;
		cummulative_p[110] = cummulative_p[109] + kdegAntiAb * antiAb;
		cummulative_p[111] = cummulative_p[110] + kremROS * ROS;
		if (time >= segmentSize * numberOfExecutions + step * stepCount) {
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 0], species[0]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 1], species[1]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 2], species[2]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 3], species[3]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 4], species[4]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 5], species[5]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 6], species[6]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 7], species[7]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 8], species[8]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 9], species[9]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 10], species[10]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 11], species[11]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 12], species[12]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 13], species[13]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 14], species[14]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 15], species[15]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 16], species[16]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 17], species[17]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 18], species[18]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 19], species[19]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 20], species[20]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 21], species[21]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 22], species[22]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 23], species[23]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 24], species[24]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 25], species[25]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 26], species[26]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 27], species[27]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 28], species[28]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 29], species[29]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 30], species[30]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 31], species[31]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 32], species[32]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 33], species[33]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 34], species[34]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 35], species[35]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 36], species[36]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 37], species[37]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 38], species[38]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 39], species[39]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 40], species[40]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 41], species[41]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 42], species[42]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 43], species[43]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 44], species[44]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 45], species[45]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 46], species[46]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 47], species[47]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 48], species[48]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 49], species[49]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 50], species[50]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 51], species[51]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 52], species[52]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 53], species[53]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 54], species[54]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 55], species[55]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 56], species[56]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 57], species[57]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 58], species[58]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 59], species[59]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 60], species[60]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 61], species[61]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 62], species[62]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 63], species[63]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 64], species[64]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 65], species[65]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 66], species[66]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 67], species[67]);
			atomicAdd(&output[69 * 34 * numberOfExecutions + stepCount * 69 + 68], species[68]);
			stepCount++;
		}
		sum_p = cummulative_p[111];
		random = hiprand_uniform(&localState);
		if (sum_p > 0) timeStep = -log(random) / sum_p;
		else break;
		random = hiprand_uniform(&localState);
		random *= sum_p;
		indexMin = 0;
		indexMax = 111;
		while (indexMax > indexMin) {
			reaction = (indexMin + indexMax) / 2;
			if (cummulative_p[reaction - 1] <= random) {
				if (cummulative_p[reaction] > random) {
					break;
				}
				else {
					indexMin = reaction;
				}
			}
			else {
				indexMax = reaction;
			}
		}
		for (int i = 0; i < 5; i++) {
			if (reactionsSpecies[reaction][i] == -1) { break; }
			species[reactionsSpecies[reaction][i]] += reactionsValues[reaction][i];
		}
		if (triggerEvent0 == 0 && time >= 345600) {
			triggerEvent0 = 1;
			antiAb += 50;
		}
		time += timeStep;
	}
	Mdm2_global[threadIdx.x] = species[0];
	p53_global[threadIdx.x] = species[1];
	Mdm2_p53_global[threadIdx.x] = species[2];
	Mdm2_mRNA_global[threadIdx.x] = species[3];
	p53_mRNA_global[threadIdx.x] = species[4];
	ATMA_global[threadIdx.x] = species[5];
	ATMI_global[threadIdx.x] = species[6];
	p53_P_global[threadIdx.x] = species[7];
	Mdm2_P_global[threadIdx.x] = species[8];
	IR_global[threadIdx.x] = species[9];
	ROS_global[threadIdx.x] = species[10];
	damDNA_global[threadIdx.x] = species[11];
	E1_global[threadIdx.x] = species[12];
	E2_global[threadIdx.x] = species[13];
	E1_Ub_global[threadIdx.x] = species[14];
	E2_Ub_global[threadIdx.x] = species[15];
	Proteasome_global[threadIdx.x] = species[16];
	Ub_global[threadIdx.x] = species[17];
	p53DUB_global[threadIdx.x] = species[18];
	Mdm2DUB_global[threadIdx.x] = species[19];
	DUB_global[threadIdx.x] = species[20];
	Mdm2_p53_Ub_global[threadIdx.x] = species[21];
	Mdm2_p53_Ub2_global[threadIdx.x] = species[22];
	Mdm2_p53_Ub3_global[threadIdx.x] = species[23];
	Mdm2_p53_Ub4_global[threadIdx.x] = species[24];
	Mdm2_P1_p53_Ub4_global[threadIdx.x] = species[25];
	Mdm2_Ub_global[threadIdx.x] = species[26];
	Mdm2_Ub2_global[threadIdx.x] = species[27];
	Mdm2_Ub3_global[threadIdx.x] = species[28];
	Mdm2_Ub4_global[threadIdx.x] = species[29];
	Mdm2_P_Ub_global[threadIdx.x] = species[30];
	Mdm2_P_Ub2_global[threadIdx.x] = species[31];
	Mdm2_P_Ub3_global[threadIdx.x] = species[32];
	Mdm2_P_Ub4_global[threadIdx.x] = species[33];
	p53_Ub4_Proteasome_global[threadIdx.x] = species[34];
	Mdm2_Ub4_Proteasome_global[threadIdx.x] = species[35];
	Mdm2_P_Ub4_Proteasome_global[threadIdx.x] = species[36];
	GSK3b_global[threadIdx.x] = species[37];
	GSK3b_p53_global[threadIdx.x] = species[38];
	GSK3b_p53_P_global[threadIdx.x] = species[39];
	Abeta_global[threadIdx.x] = species[40];
	AggAbeta_Proteasome_global[threadIdx.x] = species[41];
	AbetaPlaque_global[threadIdx.x] = species[42];
	Tau_global[threadIdx.x] = species[43];
	Tau_P1_global[threadIdx.x] = species[44];
	Tau_P2_global[threadIdx.x] = species[45];
	MT_Tau_global[threadIdx.x] = species[46];
	AggTau_global[threadIdx.x] = species[47];
	AggTau_Proteasome_global[threadIdx.x] = species[48];
	Proteasome_Tau_global[threadIdx.x] = species[49];
	PP1_global[threadIdx.x] = species[50];
	NFT_global[threadIdx.x] = species[51];
	ATP_global[threadIdx.x] = species[52];
	ADP_global[threadIdx.x] = species[53];
	AMP_global[threadIdx.x] = species[54];
	AbetaDimer_global[threadIdx.x] = species[55];
	AbetaPlaque_GliaA_global[threadIdx.x] = species[56];
	GliaI_global[threadIdx.x] = species[57];
	GliaM1_global[threadIdx.x] = species[58];
	GliaM2_global[threadIdx.x] = species[59];
	GliaA_global[threadIdx.x] = species[60];
	antiAb_global[threadIdx.x] = species[61];
	Abeta_antiAb_global[threadIdx.x] = species[62];
	AbetaDimer_antiAb_global[threadIdx.x] = species[63];
	degAbetaGlia_global[threadIdx.x] = species[64];
	disaggPlaque1_global[threadIdx.x] = species[65];
	disaggPlaque2_global[threadIdx.x] = species[66];
	Source_global[threadIdx.x] = species[67];
	Sink_global[threadIdx.x] = species[68];
	state[threadIdx.x] = localState;
}

__global__
void initCurand(hiprandState* state, unsigned long long seed) {
	hiprand_init(seed, threadIdx.x, 0, &state[threadIdx.x]);
}

int main()
{
	hipError_t cudaStatus;
	float* output;
	float* dev_output;
	output = (float*)malloc(167 * 69 * sizeof(float));
	for (int i = 0; i < 167 * 69; i++) {
		output[i] = 0;
	}
	cudaStatus = hipMalloc(&dev_output, 167 * 69 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_output, output, 167 * 69 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float host_Mdm2 = 5.0000000000;
	float* dev_Mdm2;
	cudaStatus = hipMalloc(&dev_Mdm2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2, &host_Mdm2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_global;
	cudaStatus = hipMalloc(&Mdm2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_p53 = 5.0000000000;
	float* dev_p53;
	cudaStatus = hipMalloc(&dev_p53, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_p53, &host_p53, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* p53_global;
	cudaStatus = hipMalloc(&p53_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_p53 = 95.0000000000;
	float* dev_Mdm2_p53;
	cudaStatus = hipMalloc(&dev_Mdm2_p53, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_p53, &host_Mdm2_p53, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_p53_global;
	cudaStatus = hipMalloc(&Mdm2_p53_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_mRNA = 10.0000000000;
	float* dev_Mdm2_mRNA;
	cudaStatus = hipMalloc(&dev_Mdm2_mRNA, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_mRNA, &host_Mdm2_mRNA, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_mRNA_global;
	cudaStatus = hipMalloc(&Mdm2_mRNA_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_p53_mRNA = 10.0000000000;
	float* dev_p53_mRNA;
	cudaStatus = hipMalloc(&dev_p53_mRNA, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_p53_mRNA, &host_p53_mRNA, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* p53_mRNA_global;
	cudaStatus = hipMalloc(&p53_mRNA_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_ATMA = 0.0000000000;
	float* dev_ATMA;
	cudaStatus = hipMalloc(&dev_ATMA, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ATMA, &host_ATMA, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* ATMA_global;
	cudaStatus = hipMalloc(&ATMA_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_ATMI = 200.0000000000;
	float* dev_ATMI;
	cudaStatus = hipMalloc(&dev_ATMI, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ATMI, &host_ATMI, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* ATMI_global;
	cudaStatus = hipMalloc(&ATMI_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_p53_P = 0.0000000000;
	float* dev_p53_P;
	cudaStatus = hipMalloc(&dev_p53_P, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_p53_P, &host_p53_P, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* p53_P_global;
	cudaStatus = hipMalloc(&p53_P_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P = 0.0000000000;
	float* dev_Mdm2_P;
	cudaStatus = hipMalloc(&dev_Mdm2_P, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P, &host_Mdm2_P, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P_global;
	cudaStatus = hipMalloc(&Mdm2_P_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_IR = 0.0000000000;
	float* dev_IR;
	cudaStatus = hipMalloc(&dev_IR, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_IR, &host_IR, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* IR_global;
	cudaStatus = hipMalloc(&IR_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_ROS = 0.0000000000;
	float* dev_ROS;
	cudaStatus = hipMalloc(&dev_ROS, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ROS, &host_ROS, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* ROS_global;
	cudaStatus = hipMalloc(&ROS_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_damDNA = 0.0000000000;
	float* dev_damDNA;
	cudaStatus = hipMalloc(&dev_damDNA, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_damDNA, &host_damDNA, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* damDNA_global;
	cudaStatus = hipMalloc(&damDNA_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_E1 = 100.0000000000;
	float* dev_E1;
	cudaStatus = hipMalloc(&dev_E1, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_E1, &host_E1, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* E1_global;
	cudaStatus = hipMalloc(&E1_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_E2 = 100.0000000000;
	float* dev_E2;
	cudaStatus = hipMalloc(&dev_E2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_E2, &host_E2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* E2_global;
	cudaStatus = hipMalloc(&E2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_E1_Ub = 0.0000000000;
	float* dev_E1_Ub;
	cudaStatus = hipMalloc(&dev_E1_Ub, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_E1_Ub, &host_E1_Ub, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* E1_Ub_global;
	cudaStatus = hipMalloc(&E1_Ub_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_E2_Ub = 0.0000000000;
	float* dev_E2_Ub;
	cudaStatus = hipMalloc(&dev_E2_Ub, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_E2_Ub, &host_E2_Ub, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* E2_Ub_global;
	cudaStatus = hipMalloc(&E2_Ub_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Proteasome = 500.0000000000;
	float* dev_Proteasome;
	cudaStatus = hipMalloc(&dev_Proteasome, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Proteasome, &host_Proteasome, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Proteasome_global;
	cudaStatus = hipMalloc(&Proteasome_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Ub = 4000.0000000000;
	float* dev_Ub;
	cudaStatus = hipMalloc(&dev_Ub, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Ub, &host_Ub, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Ub_global;
	cudaStatus = hipMalloc(&Ub_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_p53DUB = 200.0000000000;
	float* dev_p53DUB;
	cudaStatus = hipMalloc(&dev_p53DUB, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_p53DUB, &host_p53DUB, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* p53DUB_global;
	cudaStatus = hipMalloc(&p53DUB_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2DUB = 200.0000000000;
	float* dev_Mdm2DUB;
	cudaStatus = hipMalloc(&dev_Mdm2DUB, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2DUB, &host_Mdm2DUB, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2DUB_global;
	cudaStatus = hipMalloc(&Mdm2DUB_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_DUB = 200.0000000000;
	float* dev_DUB;
	cudaStatus = hipMalloc(&dev_DUB, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_DUB, &host_DUB, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* DUB_global;
	cudaStatus = hipMalloc(&DUB_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_p53_Ub = 0.0000000000;
	float* dev_Mdm2_p53_Ub;
	cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub, &host_Mdm2_p53_Ub, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_p53_Ub_global;
	cudaStatus = hipMalloc(&Mdm2_p53_Ub_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_p53_Ub2 = 0.0000000000;
	float* dev_Mdm2_p53_Ub2;
	cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub2, &host_Mdm2_p53_Ub2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_p53_Ub2_global;
	cudaStatus = hipMalloc(&Mdm2_p53_Ub2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_p53_Ub3 = 0.0000000000;
	float* dev_Mdm2_p53_Ub3;
	cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub3, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub3, &host_Mdm2_p53_Ub3, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_p53_Ub3_global;
	cudaStatus = hipMalloc(&Mdm2_p53_Ub3_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_p53_Ub4 = 0.0000000000;
	float* dev_Mdm2_p53_Ub4;
	cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub4, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub4, &host_Mdm2_p53_Ub4, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_p53_Ub4_global;
	cudaStatus = hipMalloc(&Mdm2_p53_Ub4_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P1_p53_Ub4 = 0.0000000000;
	float* dev_Mdm2_P1_p53_Ub4;
	cudaStatus = hipMalloc(&dev_Mdm2_P1_p53_Ub4, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P1_p53_Ub4, &host_Mdm2_P1_p53_Ub4, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P1_p53_Ub4_global;
	cudaStatus = hipMalloc(&Mdm2_P1_p53_Ub4_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_Ub = 0.0000000000;
	float* dev_Mdm2_Ub;
	cudaStatus = hipMalloc(&dev_Mdm2_Ub, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_Ub, &host_Mdm2_Ub, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_Ub_global;
	cudaStatus = hipMalloc(&Mdm2_Ub_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_Ub2 = 0.0000000000;
	float* dev_Mdm2_Ub2;
	cudaStatus = hipMalloc(&dev_Mdm2_Ub2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_Ub2, &host_Mdm2_Ub2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_Ub2_global;
	cudaStatus = hipMalloc(&Mdm2_Ub2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_Ub3 = 0.0000000000;
	float* dev_Mdm2_Ub3;
	cudaStatus = hipMalloc(&dev_Mdm2_Ub3, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_Ub3, &host_Mdm2_Ub3, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_Ub3_global;
	cudaStatus = hipMalloc(&Mdm2_Ub3_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_Ub4 = 0.0000000000;
	float* dev_Mdm2_Ub4;
	cudaStatus = hipMalloc(&dev_Mdm2_Ub4, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_Ub4, &host_Mdm2_Ub4, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_Ub4_global;
	cudaStatus = hipMalloc(&Mdm2_Ub4_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P_Ub = 0.0000000000;
	float* dev_Mdm2_P_Ub;
	cudaStatus = hipMalloc(&dev_Mdm2_P_Ub, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P_Ub, &host_Mdm2_P_Ub, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P_Ub_global;
	cudaStatus = hipMalloc(&Mdm2_P_Ub_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P_Ub2 = 0.0000000000;
	float* dev_Mdm2_P_Ub2;
	cudaStatus = hipMalloc(&dev_Mdm2_P_Ub2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P_Ub2, &host_Mdm2_P_Ub2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P_Ub2_global;
	cudaStatus = hipMalloc(&Mdm2_P_Ub2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P_Ub3 = 0.0000000000;
	float* dev_Mdm2_P_Ub3;
	cudaStatus = hipMalloc(&dev_Mdm2_P_Ub3, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P_Ub3, &host_Mdm2_P_Ub3, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P_Ub3_global;
	cudaStatus = hipMalloc(&Mdm2_P_Ub3_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P_Ub4 = 0.0000000000;
	float* dev_Mdm2_P_Ub4;
	cudaStatus = hipMalloc(&dev_Mdm2_P_Ub4, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P_Ub4, &host_Mdm2_P_Ub4, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P_Ub4_global;
	cudaStatus = hipMalloc(&Mdm2_P_Ub4_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_p53_Ub4_Proteasome = 0.0000000000;
	float* dev_p53_Ub4_Proteasome;
	cudaStatus = hipMalloc(&dev_p53_Ub4_Proteasome, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_p53_Ub4_Proteasome, &host_p53_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* p53_Ub4_Proteasome_global;
	cudaStatus = hipMalloc(&p53_Ub4_Proteasome_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_Ub4_Proteasome = 0.0000000000;
	float* dev_Mdm2_Ub4_Proteasome;
	cudaStatus = hipMalloc(&dev_Mdm2_Ub4_Proteasome, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_Ub4_Proteasome, &host_Mdm2_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_Ub4_Proteasome_global;
	cudaStatus = hipMalloc(&Mdm2_Ub4_Proteasome_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Mdm2_P_Ub4_Proteasome = 0.0000000000;
	float* dev_Mdm2_P_Ub4_Proteasome;
	cudaStatus = hipMalloc(&dev_Mdm2_P_Ub4_Proteasome, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Mdm2_P_Ub4_Proteasome, &host_Mdm2_P_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Mdm2_P_Ub4_Proteasome_global;
	cudaStatus = hipMalloc(&Mdm2_P_Ub4_Proteasome_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GSK3b = 500.0000000000;
	float* dev_GSK3b;
	cudaStatus = hipMalloc(&dev_GSK3b, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GSK3b, &host_GSK3b, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GSK3b_global;
	cudaStatus = hipMalloc(&GSK3b_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GSK3b_p53 = 0.0000000000;
	float* dev_GSK3b_p53;
	cudaStatus = hipMalloc(&dev_GSK3b_p53, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GSK3b_p53, &host_GSK3b_p53, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GSK3b_p53_global;
	cudaStatus = hipMalloc(&GSK3b_p53_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GSK3b_p53_P = 0.0000000000;
	float* dev_GSK3b_p53_P;
	cudaStatus = hipMalloc(&dev_GSK3b_p53_P, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GSK3b_p53_P, &host_GSK3b_p53_P, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GSK3b_p53_P_global;
	cudaStatus = hipMalloc(&GSK3b_p53_P_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Abeta = 0.0000000000;
	float* dev_Abeta;
	cudaStatus = hipMalloc(&dev_Abeta, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Abeta, &host_Abeta, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Abeta_global;
	cudaStatus = hipMalloc(&Abeta_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AggAbeta_Proteasome = 0.0000000000;
	float* dev_AggAbeta_Proteasome;
	cudaStatus = hipMalloc(&dev_AggAbeta_Proteasome, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AggAbeta_Proteasome, &host_AggAbeta_Proteasome, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AggAbeta_Proteasome_global;
	cudaStatus = hipMalloc(&AggAbeta_Proteasome_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AbetaPlaque = 0.0000000000;
	float* dev_AbetaPlaque;
	cudaStatus = hipMalloc(&dev_AbetaPlaque, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AbetaPlaque, &host_AbetaPlaque, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AbetaPlaque_global;
	cudaStatus = hipMalloc(&AbetaPlaque_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Tau = 0.0000000000;
	float* dev_Tau;
	cudaStatus = hipMalloc(&dev_Tau, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Tau, &host_Tau, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Tau_global;
	cudaStatus = hipMalloc(&Tau_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Tau_P1 = 0.0000000000;
	float* dev_Tau_P1;
	cudaStatus = hipMalloc(&dev_Tau_P1, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Tau_P1, &host_Tau_P1, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Tau_P1_global;
	cudaStatus = hipMalloc(&Tau_P1_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Tau_P2 = 0.0000000000;
	float* dev_Tau_P2;
	cudaStatus = hipMalloc(&dev_Tau_P2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Tau_P2, &host_Tau_P2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Tau_P2_global;
	cudaStatus = hipMalloc(&Tau_P2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_MT_Tau = 100.0000000000;
	float* dev_MT_Tau;
	cudaStatus = hipMalloc(&dev_MT_Tau, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_MT_Tau, &host_MT_Tau, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* MT_Tau_global;
	cudaStatus = hipMalloc(&MT_Tau_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AggTau = 0.0000000000;
	float* dev_AggTau;
	cudaStatus = hipMalloc(&dev_AggTau, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AggTau, &host_AggTau, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AggTau_global;
	cudaStatus = hipMalloc(&AggTau_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AggTau_Proteasome = 0.0000000000;
	float* dev_AggTau_Proteasome;
	cudaStatus = hipMalloc(&dev_AggTau_Proteasome, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AggTau_Proteasome, &host_AggTau_Proteasome, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AggTau_Proteasome_global;
	cudaStatus = hipMalloc(&AggTau_Proteasome_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Proteasome_Tau = 0.0000000000;
	float* dev_Proteasome_Tau;
	cudaStatus = hipMalloc(&dev_Proteasome_Tau, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Proteasome_Tau, &host_Proteasome_Tau, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Proteasome_Tau_global;
	cudaStatus = hipMalloc(&Proteasome_Tau_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_PP1 = 50.0000000000;
	float* dev_PP1;
	cudaStatus = hipMalloc(&dev_PP1, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_PP1, &host_PP1, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* PP1_global;
	cudaStatus = hipMalloc(&PP1_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_NFT = 0.0000000000;
	float* dev_NFT;
	cudaStatus = hipMalloc(&dev_NFT, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_NFT, &host_NFT, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* NFT_global;
	cudaStatus = hipMalloc(&NFT_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_ATP = 10000.0000000000;
	float* dev_ATP;
	cudaStatus = hipMalloc(&dev_ATP, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ATP, &host_ATP, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* ATP_global;
	cudaStatus = hipMalloc(&ATP_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_ADP = 1000.0000000000;
	float* dev_ADP;
	cudaStatus = hipMalloc(&dev_ADP, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_ADP, &host_ADP, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* ADP_global;
	cudaStatus = hipMalloc(&ADP_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AMP = 1000.0000000000;
	float* dev_AMP;
	cudaStatus = hipMalloc(&dev_AMP, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AMP, &host_AMP, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AMP_global;
	cudaStatus = hipMalloc(&AMP_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AbetaDimer = 0.0000000000;
	float* dev_AbetaDimer;
	cudaStatus = hipMalloc(&dev_AbetaDimer, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AbetaDimer, &host_AbetaDimer, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AbetaDimer_global;
	cudaStatus = hipMalloc(&AbetaDimer_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AbetaPlaque_GliaA = 0.0000000000;
	float* dev_AbetaPlaque_GliaA;
	cudaStatus = hipMalloc(&dev_AbetaPlaque_GliaA, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AbetaPlaque_GliaA, &host_AbetaPlaque_GliaA, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AbetaPlaque_GliaA_global;
	cudaStatus = hipMalloc(&AbetaPlaque_GliaA_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GliaI = 100.0000000000;
	float* dev_GliaI;
	cudaStatus = hipMalloc(&dev_GliaI, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GliaI, &host_GliaI, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GliaI_global;
	cudaStatus = hipMalloc(&GliaI_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GliaM1 = 0.0000000000;
	float* dev_GliaM1;
	cudaStatus = hipMalloc(&dev_GliaM1, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GliaM1, &host_GliaM1, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GliaM1_global;
	cudaStatus = hipMalloc(&GliaM1_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GliaM2 = 0.0000000000;
	float* dev_GliaM2;
	cudaStatus = hipMalloc(&dev_GliaM2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GliaM2, &host_GliaM2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GliaM2_global;
	cudaStatus = hipMalloc(&GliaM2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_GliaA = 0.0000000000;
	float* dev_GliaA;
	cudaStatus = hipMalloc(&dev_GliaA, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_GliaA, &host_GliaA, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* GliaA_global;
	cudaStatus = hipMalloc(&GliaA_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_antiAb = 0.0000000000;
	float* dev_antiAb;
	cudaStatus = hipMalloc(&dev_antiAb, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_antiAb, &host_antiAb, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* antiAb_global;
	cudaStatus = hipMalloc(&antiAb_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Abeta_antiAb = 0.0000000000;
	float* dev_Abeta_antiAb;
	cudaStatus = hipMalloc(&dev_Abeta_antiAb, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Abeta_antiAb, &host_Abeta_antiAb, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Abeta_antiAb_global;
	cudaStatus = hipMalloc(&Abeta_antiAb_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_AbetaDimer_antiAb = 0.0000000000;
	float* dev_AbetaDimer_antiAb;
	cudaStatus = hipMalloc(&dev_AbetaDimer_antiAb, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_AbetaDimer_antiAb, &host_AbetaDimer_antiAb, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* AbetaDimer_antiAb_global;
	cudaStatus = hipMalloc(&AbetaDimer_antiAb_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_degAbetaGlia = 0.0000000000;
	float* dev_degAbetaGlia;
	cudaStatus = hipMalloc(&dev_degAbetaGlia, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_degAbetaGlia, &host_degAbetaGlia, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* degAbetaGlia_global;
	cudaStatus = hipMalloc(&degAbetaGlia_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_disaggPlaque1 = 0.0000000000;
	float* dev_disaggPlaque1;
	cudaStatus = hipMalloc(&dev_disaggPlaque1, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_disaggPlaque1, &host_disaggPlaque1, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* disaggPlaque1_global;
	cudaStatus = hipMalloc(&disaggPlaque1_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_disaggPlaque2 = 0.0000000000;
	float* dev_disaggPlaque2;
	cudaStatus = hipMalloc(&dev_disaggPlaque2, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_disaggPlaque2, &host_disaggPlaque2, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* disaggPlaque2_global;
	cudaStatus = hipMalloc(&disaggPlaque2_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Source = 1.0000000000;
	float* dev_Source;
	cudaStatus = hipMalloc(&dev_Source, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Source, &host_Source, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Source_global;
	cudaStatus = hipMalloc(&Source_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	float host_Sink = 1.0000000000;
	float* dev_Sink;
	cudaStatus = hipMalloc(&dev_Sink, sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_Sink, &host_Sink, sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	float* Sink_global;
	cudaStatus = hipMalloc(&Sink_global, 32 * sizeof(float));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	hiprandState *devStates;
	CUDA_CALL(hipMalloc((void **)&devStates, 32 * sizeof(hiprandState)));
	initCurand << <1, 32 >> > (devStates, SEED);
	for (int i = 0; i < 5; i++) {
		simulate << <1, 32 >> > (i, dev_output, devStates, 60.0000000000, 10000.0000000000, 2000, dev_Mdm2, Mdm2_global, dev_p53, p53_global, dev_Mdm2_p53, Mdm2_p53_global, dev_Mdm2_mRNA, Mdm2_mRNA_global, dev_p53_mRNA, p53_mRNA_global, dev_ATMA, ATMA_global, dev_ATMI, ATMI_global, dev_p53_P, p53_P_global, dev_Mdm2_P, Mdm2_P_global, dev_IR, IR_global, dev_ROS, ROS_global, dev_damDNA, damDNA_global, dev_E1, E1_global, dev_E2, E2_global, dev_E1_Ub, E1_Ub_global, dev_E2_Ub, E2_Ub_global, dev_Proteasome, Proteasome_global, dev_Ub, Ub_global, dev_p53DUB, p53DUB_global, dev_Mdm2DUB, Mdm2DUB_global, dev_DUB, DUB_global, dev_Mdm2_p53_Ub, Mdm2_p53_Ub_global, dev_Mdm2_p53_Ub2, Mdm2_p53_Ub2_global, dev_Mdm2_p53_Ub3, Mdm2_p53_Ub3_global, dev_Mdm2_p53_Ub4, Mdm2_p53_Ub4_global, dev_Mdm2_P1_p53_Ub4, Mdm2_P1_p53_Ub4_global, dev_Mdm2_Ub, Mdm2_Ub_global, dev_Mdm2_Ub2, Mdm2_Ub2_global, dev_Mdm2_Ub3, Mdm2_Ub3_global, dev_Mdm2_Ub4, Mdm2_Ub4_global, dev_Mdm2_P_Ub, Mdm2_P_Ub_global, dev_Mdm2_P_Ub2, Mdm2_P_Ub2_global, dev_Mdm2_P_Ub3, Mdm2_P_Ub3_global, dev_Mdm2_P_Ub4, Mdm2_P_Ub4_global, dev_p53_Ub4_Proteasome, p53_Ub4_Proteasome_global, dev_Mdm2_Ub4_Proteasome, Mdm2_Ub4_Proteasome_global, dev_Mdm2_P_Ub4_Proteasome, Mdm2_P_Ub4_Proteasome_global, dev_GSK3b, GSK3b_global, dev_GSK3b_p53, GSK3b_p53_global, dev_GSK3b_p53_P, GSK3b_p53_P_global, dev_Abeta, Abeta_global, dev_AggAbeta_Proteasome, AggAbeta_Proteasome_global, dev_AbetaPlaque, AbetaPlaque_global, dev_Tau, Tau_global, dev_Tau_P1, Tau_P1_global, dev_Tau_P2, Tau_P2_global, dev_MT_Tau, MT_Tau_global, dev_AggTau, AggTau_global, dev_AggTau_Proteasome, AggTau_Proteasome_global, dev_Proteasome_Tau, Proteasome_Tau_global, dev_PP1, PP1_global, dev_NFT, NFT_global, dev_ATP, ATP_global, dev_ADP, ADP_global, dev_AMP, AMP_global, dev_AbetaDimer, AbetaDimer_global, dev_AbetaPlaque_GliaA, AbetaPlaque_GliaA_global, dev_GliaI, GliaI_global, dev_GliaM1, GliaM1_global, dev_GliaM2, GliaM2_global, dev_GliaA, GliaA_global, dev_antiAb, antiAb_global, dev_Abeta_antiAb, Abeta_antiAb_global, dev_AbetaDimer_antiAb, AbetaDimer_antiAb_global, dev_degAbetaGlia, degAbetaGlia_global, dev_disaggPlaque1, disaggPlaque1_global, dev_disaggPlaque2, disaggPlaque2_global, dev_Source, Source_global, dev_Sink, Sink_global);

		cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess) { fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

		cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
	}


	cudaStatus = hipMemcpy(output, dev_output, 167 * 69 * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2, dev_Mdm2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_p53, dev_p53, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_p53, dev_Mdm2_p53, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_mRNA, dev_Mdm2_mRNA, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_p53_mRNA, dev_p53_mRNA, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_ATMA, dev_ATMA, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_ATMI, dev_ATMI, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_p53_P, dev_p53_P, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P, dev_Mdm2_P, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_IR, dev_IR, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_ROS, dev_ROS, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_damDNA, dev_damDNA, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_E1, dev_E1, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_E2, dev_E2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_E1_Ub, dev_E1_Ub, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_E2_Ub, dev_E2_Ub, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Proteasome, dev_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Ub, dev_Ub, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_p53DUB, dev_p53DUB, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2DUB, dev_Mdm2DUB, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_DUB, dev_DUB, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_p53_Ub, dev_Mdm2_p53_Ub, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_p53_Ub2, dev_Mdm2_p53_Ub2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_p53_Ub3, dev_Mdm2_p53_Ub3, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_p53_Ub4, dev_Mdm2_p53_Ub4, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P1_p53_Ub4, dev_Mdm2_P1_p53_Ub4, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_Ub, dev_Mdm2_Ub, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_Ub2, dev_Mdm2_Ub2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_Ub3, dev_Mdm2_Ub3, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_Ub4, dev_Mdm2_Ub4, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P_Ub, dev_Mdm2_P_Ub, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P_Ub2, dev_Mdm2_P_Ub2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P_Ub3, dev_Mdm2_P_Ub3, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P_Ub4, dev_Mdm2_P_Ub4, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_p53_Ub4_Proteasome, dev_p53_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_Ub4_Proteasome, dev_Mdm2_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Mdm2_P_Ub4_Proteasome, dev_Mdm2_P_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GSK3b, dev_GSK3b, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GSK3b_p53, dev_GSK3b_p53, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GSK3b_p53_P, dev_GSK3b_p53_P, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Abeta, dev_Abeta, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AggAbeta_Proteasome, dev_AggAbeta_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AbetaPlaque, dev_AbetaPlaque, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Tau, dev_Tau, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Tau_P1, dev_Tau_P1, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Tau_P2, dev_Tau_P2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_MT_Tau, dev_MT_Tau, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AggTau, dev_AggTau, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AggTau_Proteasome, dev_AggTau_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Proteasome_Tau, dev_Proteasome_Tau, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_PP1, dev_PP1, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_NFT, dev_NFT, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_ATP, dev_ATP, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_ADP, dev_ADP, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AMP, dev_AMP, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AbetaDimer, dev_AbetaDimer, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AbetaPlaque_GliaA, dev_AbetaPlaque_GliaA, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GliaI, dev_GliaI, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GliaM1, dev_GliaM1, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GliaM2, dev_GliaM2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_GliaA, dev_GliaA, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_antiAb, dev_antiAb, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Abeta_antiAb, dev_Abeta_antiAb, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_AbetaDimer_antiAb, dev_AbetaDimer_antiAb, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_degAbetaGlia, dev_degAbetaGlia, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_disaggPlaque1, dev_disaggPlaque1, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_disaggPlaque2, dev_disaggPlaque2, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Source, dev_Source, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(&host_Sink, dev_Sink, sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	FILE* results = fopen("results.csv", "w");
	if (results == NULL) {
		printf("Error acesssing results!");
		exit(1);
	}
	fprintf(results, "time");
	fprintf(results, ", Mdm2");
	fprintf(results, ", p53");
	fprintf(results, ", Mdm2_p53");
	fprintf(results, ", Mdm2_mRNA");
	fprintf(results, ", p53_mRNA");
	fprintf(results, ", ATMA");
	fprintf(results, ", ATMI");
	fprintf(results, ", p53_P");
	fprintf(results, ", Mdm2_P");
	fprintf(results, ", IR");
	fprintf(results, ", ROS");
	fprintf(results, ", damDNA");
	fprintf(results, ", E1");
	fprintf(results, ", E2");
	fprintf(results, ", E1_Ub");
	fprintf(results, ", E2_Ub");
	fprintf(results, ", Proteasome");
	fprintf(results, ", Ub");
	fprintf(results, ", p53DUB");
	fprintf(results, ", Mdm2DUB");
	fprintf(results, ", DUB");
	fprintf(results, ", Mdm2_p53_Ub");
	fprintf(results, ", Mdm2_p53_Ub2");
	fprintf(results, ", Mdm2_p53_Ub3");
	fprintf(results, ", Mdm2_p53_Ub4");
	fprintf(results, ", Mdm2_P1_p53_Ub4");
	fprintf(results, ", Mdm2_Ub");
	fprintf(results, ", Mdm2_Ub2");
	fprintf(results, ", Mdm2_Ub3");
	fprintf(results, ", Mdm2_Ub4");
	fprintf(results, ", Mdm2_P_Ub");
	fprintf(results, ", Mdm2_P_Ub2");
	fprintf(results, ", Mdm2_P_Ub3");
	fprintf(results, ", Mdm2_P_Ub4");
	fprintf(results, ", p53_Ub4_Proteasome");
	fprintf(results, ", Mdm2_Ub4_Proteasome");
	fprintf(results, ", Mdm2_P_Ub4_Proteasome");
	fprintf(results, ", GSK3b");
	fprintf(results, ", GSK3b_p53");
	fprintf(results, ", GSK3b_p53_P");
	fprintf(results, ", Abeta");
	fprintf(results, ", AggAbeta_Proteasome");
	fprintf(results, ", AbetaPlaque");
	fprintf(results, ", Tau");
	fprintf(results, ", Tau_P1");
	fprintf(results, ", Tau_P2");
	fprintf(results, ", MT_Tau");
	fprintf(results, ", AggTau");
	fprintf(results, ", AggTau_Proteasome");
	fprintf(results, ", Proteasome_Tau");
	fprintf(results, ", PP1");
	fprintf(results, ", NFT");
	fprintf(results, ", ATP");
	fprintf(results, ", ADP");
	fprintf(results, ", AMP");
	fprintf(results, ", AbetaDimer");
	fprintf(results, ", AbetaPlaque_GliaA");
	fprintf(results, ", GliaI");
	fprintf(results, ", GliaM1");
	fprintf(results, ", GliaM2");
	fprintf(results, ", GliaA");
	fprintf(results, ", antiAb");
	fprintf(results, ", Abeta_antiAb");
	fprintf(results, ", AbetaDimer_antiAb");
	fprintf(results, ", degAbetaGlia");
	fprintf(results, ", disaggPlaque1");
	fprintf(results, ", disaggPlaque2");
	fprintf(results, ", Source");
	fprintf(results, ", Sink");
	fprintf(results, "\n");
	for (int i = 0; i < 167; i++) {
		fprintf(results, "%.10lf", 60.0000000000*i);
		for (int j = 0; j < 69; j++) {
			fprintf(results, ", %.10lf", output[69 * i + j] / 32);
		}
		fprintf(results, "\n");
	}
	fprintf(results, "\n");
Error:
	hipFree(dev_output);
	hipFree(dev_Mdm2);
	hipFree(Mdm2_global);
	hipFree(dev_p53);
	hipFree(p53_global);
	hipFree(dev_Mdm2_p53);
	hipFree(Mdm2_p53_global);
	hipFree(dev_Mdm2_mRNA);
	hipFree(Mdm2_mRNA_global);
	hipFree(dev_p53_mRNA);
	hipFree(p53_mRNA_global);
	hipFree(dev_ATMA);
	hipFree(ATMA_global);
	hipFree(dev_ATMI);
	hipFree(ATMI_global);
	hipFree(dev_p53_P);
	hipFree(p53_P_global);
	hipFree(dev_Mdm2_P);
	hipFree(Mdm2_P_global);
	hipFree(dev_IR);
	hipFree(IR_global);
	hipFree(dev_ROS);
	hipFree(ROS_global);
	hipFree(dev_damDNA);
	hipFree(damDNA_global);
	hipFree(dev_E1);
	hipFree(E1_global);
	hipFree(dev_E2);
	hipFree(E2_global);
	hipFree(dev_E1_Ub);
	hipFree(E1_Ub_global);
	hipFree(dev_E2_Ub);
	hipFree(E2_Ub_global);
	hipFree(dev_Proteasome);
	hipFree(Proteasome_global);
	hipFree(dev_Ub);
	hipFree(Ub_global);
	hipFree(dev_p53DUB);
	hipFree(p53DUB_global);
	hipFree(dev_Mdm2DUB);
	hipFree(Mdm2DUB_global);
	hipFree(dev_DUB);
	hipFree(DUB_global);
	hipFree(dev_Mdm2_p53_Ub);
	hipFree(Mdm2_p53_Ub_global);
	hipFree(dev_Mdm2_p53_Ub2);
	hipFree(Mdm2_p53_Ub2_global);
	hipFree(dev_Mdm2_p53_Ub3);
	hipFree(Mdm2_p53_Ub3_global);
	hipFree(dev_Mdm2_p53_Ub4);
	hipFree(Mdm2_p53_Ub4_global);
	hipFree(dev_Mdm2_P1_p53_Ub4);
	hipFree(Mdm2_P1_p53_Ub4_global);
	hipFree(dev_Mdm2_Ub);
	hipFree(Mdm2_Ub_global);
	hipFree(dev_Mdm2_Ub2);
	hipFree(Mdm2_Ub2_global);
	hipFree(dev_Mdm2_Ub3);
	hipFree(Mdm2_Ub3_global);
	hipFree(dev_Mdm2_Ub4);
	hipFree(Mdm2_Ub4_global);
	hipFree(dev_Mdm2_P_Ub);
	hipFree(Mdm2_P_Ub_global);
	hipFree(dev_Mdm2_P_Ub2);
	hipFree(Mdm2_P_Ub2_global);
	hipFree(dev_Mdm2_P_Ub3);
	hipFree(Mdm2_P_Ub3_global);
	hipFree(dev_Mdm2_P_Ub4);
	hipFree(Mdm2_P_Ub4_global);
	hipFree(dev_p53_Ub4_Proteasome);
	hipFree(p53_Ub4_Proteasome_global);
	hipFree(dev_Mdm2_Ub4_Proteasome);
	hipFree(Mdm2_Ub4_Proteasome_global);
	hipFree(dev_Mdm2_P_Ub4_Proteasome);
	hipFree(Mdm2_P_Ub4_Proteasome_global);
	hipFree(dev_GSK3b);
	hipFree(GSK3b_global);
	hipFree(dev_GSK3b_p53);
	hipFree(GSK3b_p53_global);
	hipFree(dev_GSK3b_p53_P);
	hipFree(GSK3b_p53_P_global);
	hipFree(dev_Abeta);
	hipFree(Abeta_global);
	hipFree(dev_AggAbeta_Proteasome);
	hipFree(AggAbeta_Proteasome_global);
	hipFree(dev_AbetaPlaque);
	hipFree(AbetaPlaque_global);
	hipFree(dev_Tau);
	hipFree(Tau_global);
	hipFree(dev_Tau_P1);
	hipFree(Tau_P1_global);
	hipFree(dev_Tau_P2);
	hipFree(Tau_P2_global);
	hipFree(dev_MT_Tau);
	hipFree(MT_Tau_global);
	hipFree(dev_AggTau);
	hipFree(AggTau_global);
	hipFree(dev_AggTau_Proteasome);
	hipFree(AggTau_Proteasome_global);
	hipFree(dev_Proteasome_Tau);
	hipFree(Proteasome_Tau_global);
	hipFree(dev_PP1);
	hipFree(PP1_global);
	hipFree(dev_NFT);
	hipFree(NFT_global);
	hipFree(dev_ATP);
	hipFree(ATP_global);
	hipFree(dev_ADP);
	hipFree(ADP_global);
	hipFree(dev_AMP);
	hipFree(AMP_global);
	hipFree(dev_AbetaDimer);
	hipFree(AbetaDimer_global);
	hipFree(dev_AbetaPlaque_GliaA);
	hipFree(AbetaPlaque_GliaA_global);
	hipFree(dev_GliaI);
	hipFree(GliaI_global);
	hipFree(dev_GliaM1);
	hipFree(GliaM1_global);
	hipFree(dev_GliaM2);
	hipFree(GliaM2_global);
	hipFree(dev_GliaA);
	hipFree(GliaA_global);
	hipFree(dev_antiAb);
	hipFree(antiAb_global);
	hipFree(dev_Abeta_antiAb);
	hipFree(Abeta_antiAb_global);
	hipFree(dev_AbetaDimer_antiAb);
	hipFree(AbetaDimer_antiAb_global);
	hipFree(dev_degAbetaGlia);
	hipFree(degAbetaGlia_global);
	hipFree(dev_disaggPlaque1);
	hipFree(disaggPlaque1_global);
	hipFree(dev_disaggPlaque2);
	hipFree(disaggPlaque2_global);
	hipFree(dev_Source);
	hipFree(Source_global);
	hipFree(dev_Sink);
	hipFree(Sink_global);

	return 0;
}
