#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <stdlib.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

#define pow powf

#define SEED 23
#define cell 1.000000
#define ksynp53mRNA 0.001000
#define kdegp53mRNA 0.000100
#define ksynMdm2mRNA 0.000500
#define kdegMdm2mRNA 0.000500
#define ksynMdm2mRNAGSK3bp53 0.000700
#define ksynp53 0.007000
#define kdegp53 0.005000
#define kbinMdm2p53 0.001155
#define krelMdm2p53 0.000012
#define kbinGSK3bp53 0.000002
#define krelGSK3bp53 0.002000
#define ksynMdm2 0.000495
#define kdegMdm2 0.010000
#define kbinE1Ub 0.000200
#define kbinE2Ub 0.001000
#define kp53Ub 0.000050
#define kp53PolyUb 0.010000
#define kbinProt 0.000002
#define kactDUBp53 0.000000
#define kactDUBProtp53 0.000100
#define kactDUBMdm2 0.000000
#define kMdm2Ub 0.000005
#define kMdm2PUb 0.000007
#define kMdm2PolyUb 0.004560
#define kdam 0.080000
#define krepair 0.000020
#define kactATM 0.000100
#define kinactATM 0.000500
#define kphosp53 0.000200
#define kdephosp53 0.500000
#define kphosMdm2 2.000000
#define kdephosMdm2 0.500000
#define kphosMdm2GSK3b 0.005000
#define kphosMdm2GSK3bp53 0.500000
#define kphospTauGSK3bp53 0.100000
#define kphospTauGSK3b 0.000200
#define kdephospTau 0.010000
#define kbinMTTau 0.100000
#define krelMTTau 0.000100
#define ksynTau 0.000080
#define kbinTauProt 0.000000
#define kdegTau20SProt 0.010000
#define kaggTau 0.000000
#define kaggTauP1 0.000000
#define kaggTauP2 0.000000
#define ktangfor 0.001000
#define kinhibprot 0.000000
#define ksynp53mRNAAbeta 0.000010
#define kdamROS 0.000010
#define kgenROSAbeta 0.000020
#define kgenROSPlaque 0.000010
#define kgenROSGlia 0.000010
#define kproteff 1.000000
#define kremROS 0.000070
#define kprodAbeta 0.000019
#define kprodAbeta2 0.000019
#define kdegAbeta 0.000015
#define kaggAbeta 0.000003
#define kdisaggAbeta 0.000001
#define kdisaggAbeta1 0.000200
#define kdisaggAbeta2 0.000001
#define kdegAbetaGlia 0.005000
#define kpf 0.200000
#define kpg 0.150000
#define kpghalf 10.000000
#define kactglia1 0.000001
#define kactglia2 0.000001
#define kinactglia1 0.000005
#define kinactglia2 0.000005
#define kbinAbetaGlia 0.000010
#define krelAbetaGlia 0.000050
#define kdegAntiAb 0.000003
#define kbinAbantiAb 0.000001
#define speciesUpdate(i) \
switch (i) { \
case 0: \
p53_mRNA += 1.000000; \
break; \
case 1: \
p53_mRNA -= 1.000000; \
break; \
case 2: \
Mdm2_mRNA -= 1.000000; \
Mdm2_mRNA += 1.000000; \
Mdm2 += 1.000000; \
break; \
case 3: \
p53 -= 1.000000; \
p53 += 1.000000; \
Mdm2_mRNA += 1.000000; \
break; \
case 4: \
p53_P -= 1.000000; \
p53_P += 1.000000; \
Mdm2_mRNA += 1.000000; \
break; \
case 5: \
GSK3b_p53 -= 1.000000; \
GSK3b_p53 += 1.000000; \
Mdm2_mRNA += 1.000000; \
break; \
case 6: \
GSK3b_p53_P -= 1.000000; \
GSK3b_p53_P += 1.000000; \
Mdm2_mRNA += 1.000000; \
break; \
case 7: \
Mdm2_mRNA -= 1.000000; \
break; \
case 8: \
p53 -= 1.000000; \
Mdm2 -= 1.000000; \
Mdm2_p53 += 1.000000; \
break; \
case 9: \
Mdm2_p53 -= 1.000000; \
p53 += 1.000000; \
Mdm2 += 1.000000; \
break; \
case 10: \
GSK3b -= 1.000000; \
p53 -= 1.000000; \
GSK3b_p53 += 1.000000; \
break; \
case 11: \
GSK3b_p53 -= 1.000000; \
GSK3b += 1.000000; \
p53 += 1.000000; \
break; \
case 12: \
GSK3b -= 1.000000; \
p53_P -= 1.000000; \
GSK3b_p53_P += 1.000000; \
break; \
case 13: \
GSK3b_p53_P -= 1.000000; \
GSK3b += 1.000000; \
p53_P += 1.000000; \
break; \
case 14: \
E1 -= 1.000000; \
Ub -= 1.000000; \
E1_Ub += 1.000000; \
break; \
case 15: \
E2 -= 1.000000; \
E1_Ub -= 1.000000; \
E2_Ub += 1.000000; \
E1 += 1.000000; \
break; \
case 16: \
Mdm2 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_Ub += 1.000000; \
E2 += 1.000000; \
break; \
case 17: \
Mdm2_Ub -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_Ub2 += 1.000000; \
E2 += 1.000000; \
break; \
case 18: \
Mdm2_Ub2 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_Ub3 += 1.000000; \
E2 += 1.000000; \
break; \
case 19: \
Mdm2_Ub3 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_Ub4 += 1.000000; \
E2 += 1.000000; \
break; \
case 20: \
Mdm2_Ub4 -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_Ub3 += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 21: \
Mdm2_Ub3 -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_Ub2 += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 22: \
Mdm2_Ub2 -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_Ub += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 23: \
Mdm2_Ub -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2 += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 24: \
Mdm2_Ub4 -= 1.000000; \
Proteasome -= 1.000000; \
Mdm2_Ub4_Proteasome += 1.000000; \
break; \
case 25: \
Mdm2_Ub4_Proteasome -= 1.000000; \
Proteasome += 1.000000; \
Ub += 4.000000; \
break; \
case 26: \
p53_mRNA -= 1.000000; \
p53 += 1.000000; \
p53_mRNA += 1.000000; \
break; \
case 27: \
E2_Ub -= 1.000000; \
Mdm2_p53 -= 1.000000; \
Mdm2_p53_Ub += 1.000000; \
E2 += 1.000000; \
break; \
case 28: \
Mdm2_p53_Ub -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_p53_Ub2 += 1.000000; \
E2 += 1.000000; \
break; \
case 29: \
Mdm2_p53_Ub2 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_p53_Ub3 += 1.000000; \
E2 += 1.000000; \
break; \
case 30: \
Mdm2_p53_Ub3 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_p53_Ub4 += 1.000000; \
E2 += 1.000000; \
break; \
case 31: \
Mdm2_p53_Ub4 -= 1.000000; \
p53DUB -= 1.000000; \
Mdm2_p53_Ub3 += 1.000000; \
p53DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 32: \
Mdm2_p53_Ub3 -= 1.000000; \
p53DUB -= 1.000000; \
Mdm2_p53_Ub2 += 1.000000; \
p53DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 33: \
Mdm2_p53_Ub2 -= 1.000000; \
p53DUB -= 1.000000; \
Mdm2_p53_Ub += 1.000000; \
p53DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 34: \
Mdm2_p53_Ub -= 1.000000; \
p53DUB -= 1.000000; \
Mdm2_p53 += 1.000000; \
p53DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 35: \
Mdm2_p53_Ub4 -= 1.000000; \
GSK3b -= 1.000000; \
Mdm2_P1_p53_Ub4 += 1.000000; \
GSK3b += 1.000000; \
break; \
case 36: \
Mdm2_p53_Ub4 -= 1.000000; \
GSK3b_p53 -= 1.000000; \
Mdm2_P1_p53_Ub4 += 1.000000; \
GSK3b_p53 += 1.000000; \
break; \
case 37: \
Mdm2_p53_Ub4 -= 1.000000; \
GSK3b_p53_P -= 1.000000; \
Mdm2_P1_p53_Ub4 += 1.000000; \
GSK3b_p53_P += 1.000000; \
break; \
case 38: \
Mdm2_P1_p53_Ub4 -= 1.000000; \
Proteasome -= 1.000000; \
p53_Ub4_Proteasome += 1.000000; \
Mdm2 += 1.000000; \
break; \
case 39: \
p53_Ub4_Proteasome -= 1.000000; \
Ub += 4.000000; \
Proteasome += 1.000000; \
break; \
case 40: \
Tau -= 1.000000; \
MT_Tau += 1.000000; \
break; \
case 41: \
MT_Tau -= 1.000000; \
Tau += 1.000000; \
break; \
case 42: \
GSK3b_p53 -= 1.000000; \
Tau -= 1.000000; \
GSK3b_p53 += 1.000000; \
Tau_P1 += 1.000000; \
break; \
case 43: \
GSK3b_p53 -= 1.000000; \
Tau_P1 -= 1.000000; \
GSK3b_p53 += 1.000000; \
Tau_P2 += 1.000000; \
break; \
case 44: \
GSK3b_p53_P -= 1.000000; \
Tau -= 1.000000; \
GSK3b_p53_P += 1.000000; \
Tau_P1 += 1.000000; \
break; \
case 45: \
GSK3b_p53_P -= 1.000000; \
Tau_P1 -= 1.000000; \
GSK3b_p53_P += 1.000000; \
Tau_P2 += 1.000000; \
break; \
case 46: \
GSK3b -= 1.000000; \
Tau -= 1.000000; \
GSK3b += 1.000000; \
Tau_P1 += 1.000000; \
break; \
case 47: \
GSK3b -= 1.000000; \
Tau_P1 -= 1.000000; \
GSK3b += 1.000000; \
Tau_P2 += 1.000000; \
break; \
case 48: \
Tau_P2 -= 1.000000; \
PP1 -= 1.000000; \
Tau_P1 += 1.000000; \
PP1 += 1.000000; \
break; \
case 49: \
Tau_P1 -= 1.000000; \
PP1 -= 1.000000; \
Tau += 1.000000; \
PP1 += 1.000000; \
break; \
case 50: \
Tau_P1 -= 2.000000; \
AggTau += 2.000000; \
break; \
case 51: \
Tau_P1 -= 1.000000; \
AggTau -= 1.000000; \
AggTau += 2.000000; \
break; \
case 52: \
Tau_P2 -= 2.000000; \
AggTau += 2.000000; \
break; \
case 53: \
Tau_P2 -= 1.000000; \
AggTau -= 1.000000; \
AggTau += 2.000000; \
break; \
case 54: \
Tau -= 2.000000; \
AggTau += 2.000000; \
break; \
case 55: \
Tau -= 1.000000; \
AggTau -= 1.000000; \
AggTau += 2.000000; \
break; \
case 56: \
AggTau -= 2.000000; \
NFT += 2.000000; \
break; \
case 57: \
AggTau -= 1.000000; \
NFT -= 1.000000; \
NFT += 2.000000; \
break; \
case 58: \
AggTau -= 1.000000; \
Proteasome -= 1.000000; \
AggTau_Proteasome += 1.000000; \
break; \
case 59: \
Abeta += 1.000000; \
break; \
case 60: \
GSK3b_p53 -= 1.000000; \
Abeta += 1.000000; \
GSK3b_p53 += 1.000000; \
break; \
case 61: \
GSK3b_p53_P -= 1.000000; \
Abeta += 1.000000; \
GSK3b_p53_P += 1.000000; \
break; \
case 62: \
AbetaDimer -= 1.000000; \
Proteasome -= 1.000000; \
AggAbeta_Proteasome += 1.000000; \
break; \
case 63: \
Abeta -= 1.000000; \
break; \
case 64: \
Abeta -= 1.000000; \
p53_mRNA += 1.000000; \
Abeta += 1.000000; \
break; \
case 65: \
IR -= 1.000000; \
IR += 1.000000; \
damDNA += 1.000000; \
break; \
case 66: \
damDNA -= 1.000000; \
break; \
case 67: \
damDNA -= 1.000000; \
ATMI -= 1.000000; \
damDNA += 1.000000; \
ATMA += 1.000000; \
break; \
case 68: \
p53 -= 1.000000; \
ATMA -= 1.000000; \
p53_P += 1.000000; \
ATMA += 1.000000; \
break; \
case 69: \
p53_P -= 1.000000; \
p53 += 1.000000; \
break; \
case 70: \
Mdm2 -= 1.000000; \
ATMA -= 1.000000; \
Mdm2_P += 1.000000; \
ATMA += 1.000000; \
break; \
case 71: \
Mdm2_P -= 1.000000; \
Mdm2 += 1.000000; \
break; \
case 72: \
Mdm2_P -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_P_Ub += 1.000000; \
E2 += 1.000000; \
break; \
case 73: \
Mdm2_P_Ub -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_P_Ub2 += 1.000000; \
E2 += 1.000000; \
break; \
case 74: \
Mdm2_P_Ub2 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_P_Ub3 += 1.000000; \
E2 += 1.000000; \
break; \
case 75: \
Mdm2_P_Ub3 -= 1.000000; \
E2_Ub -= 1.000000; \
Mdm2_P_Ub4 += 1.000000; \
E2 += 1.000000; \
break; \
case 76: \
Mdm2_P_Ub4 -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_P_Ub3 += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 77: \
Mdm2_P_Ub3 -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_P_Ub2 += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 78: \
Mdm2_P_Ub2 -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_P_Ub += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 79: \
Mdm2_P_Ub -= 1.000000; \
Mdm2DUB -= 1.000000; \
Mdm2_P += 1.000000; \
Mdm2DUB += 1.000000; \
Ub += 1.000000; \
break; \
case 80: \
Mdm2_P_Ub4 -= 1.000000; \
Proteasome -= 1.000000; \
Mdm2_P_Ub4_Proteasome += 1.000000; \
break; \
case 81: \
Mdm2_P_Ub4_Proteasome -= 1.000000; \
Proteasome += 1.000000; \
Ub += 4.000000; \
break; \
case 82: \
ATMA -= 1.000000; \
ATMI += 1.000000; \
break; \
case 83: \
Abeta -= 1.000000; \
Abeta += 1.000000; \
ROS += 1.000000; \
break; \
case 84: \
AbetaPlaque -= 1.000000; \
AbetaPlaque += 1.000000; \
ROS += 1.000000; \
break; \
case 85: \
AggAbeta_Proteasome -= 1.000000; \
AggAbeta_Proteasome += 1.000000; \
ROS += 1.000000; \
break; \
case 86: \
ROS -= 1.000000; \
ROS += 1.000000; \
damDNA += 1.000000; \
break; \
case 87: \
Tau += 1.000000; \
break; \
case 88: \
Tau -= 1.000000; \
Proteasome -= 1.000000; \
Proteasome_Tau += 1.000000; \
break; \
case 89: \
Proteasome_Tau -= 1.000000; \
Proteasome += 1.000000; \
break; \
case 90: \
Abeta -= 2.000000; \
AbetaDimer += 1.000000; \
break; \
case 91: \
AbetaDimer -= 2.000000; \
AbetaPlaque += 1.000000; \
break; \
case 92: \
AbetaDimer -= 1.000000; \
AbetaPlaque -= 1.000000; \
AbetaPlaque += 2.000000; \
break; \
case 93: \
AbetaDimer -= 1.000000; \
Abeta += 2.000000; \
break; \
case 94: \
AbetaPlaque -= 1.000000; \
AbetaDimer += 1.000000; \
disaggPlaque1 += 1.000000; \
break; \
case 95: \
AbetaPlaque -= 1.000000; \
antiAb -= 1.000000; \
AbetaDimer += 1.000000; \
antiAb += 1.000000; \
disaggPlaque2 += 1.000000; \
break; \
case 96: \
Abeta -= 1.000000; \
antiAb -= 1.000000; \
Abeta_antiAb += 1.000000; \
break; \
case 97: \
AbetaDimer -= 1.000000; \
antiAb -= 1.000000; \
AbetaDimer_antiAb += 1.000000; \
break; \
case 98: \
Abeta_antiAb -= 1.000000; \
antiAb += 1.000000; \
break; \
case 99: \
AbetaDimer_antiAb -= 1.000000; \
antiAb += 1.000000; \
break; \
case 100: \
GliaI -= 1.000000; \
AbetaPlaque -= 1.000000; \
GliaM1 += 1.000000; \
AbetaPlaque += 1.000000; \
break; \
case 101: \
GliaM1 -= 1.000000; \
AbetaPlaque -= 1.000000; \
GliaM2 += 1.000000; \
AbetaPlaque += 1.000000; \
break; \
case 102: \
GliaM2 -= 1.000000; \
antiAb -= 1.000000; \
GliaA += 1.000000; \
antiAb += 1.000000; \
break; \
case 103: \
GliaA -= 1.000000; \
GliaM2 += 1.000000; \
break; \
case 104: \
GliaM2 -= 1.000000; \
GliaM1 += 1.000000; \
break; \
case 105: \
GliaM1 -= 1.000000; \
GliaI += 1.000000; \
break; \
case 106: \
AbetaPlaque -= 1.000000; \
GliaA -= 1.000000; \
AbetaPlaque_GliaA += 1.000000; \
break; \
case 107: \
AbetaPlaque_GliaA -= 1.000000; \
AbetaPlaque += 1.000000; \
GliaA += 1.000000; \
break; \
case 108: \
AbetaPlaque_GliaA -= 1.000000; \
GliaA += 1.000000; \
degAbetaGlia += 1.000000; \
break; \
case 109: \
AbetaPlaque_GliaA -= 1.000000; \
AbetaPlaque_GliaA += 1.000000; \
ROS += 1.000000; \
break; \
case 110: \
antiAb -= 1.000000; \
break; \
case 111: \
ROS -= 1.000000; \
break; \
} \

__global__ 
void simulate (float* output, hiprandState *state, float step, float endTime, float* Mdm2_aux, float* p53_aux, float* Mdm2_p53_aux, float* Mdm2_mRNA_aux, float* p53_mRNA_aux, float* ATMA_aux, float* ATMI_aux, float* p53_P_aux, float* Mdm2_P_aux, float* IR_aux, float* ROS_aux, float* damDNA_aux, float* E1_aux, float* E2_aux, float* E1_Ub_aux, float* E2_Ub_aux, float* Proteasome_aux, float* Ub_aux, float* p53DUB_aux, float* Mdm2DUB_aux, float* DUB_aux, float* Mdm2_p53_Ub_aux, float* Mdm2_p53_Ub2_aux, float* Mdm2_p53_Ub3_aux, float* Mdm2_p53_Ub4_aux, float* Mdm2_P1_p53_Ub4_aux, float* Mdm2_Ub_aux, float* Mdm2_Ub2_aux, float* Mdm2_Ub3_aux, float* Mdm2_Ub4_aux, float* Mdm2_P_Ub_aux, float* Mdm2_P_Ub2_aux, float* Mdm2_P_Ub3_aux, float* Mdm2_P_Ub4_aux, float* p53_Ub4_Proteasome_aux, float* Mdm2_Ub4_Proteasome_aux, float* Mdm2_P_Ub4_Proteasome_aux, float* GSK3b_aux, float* GSK3b_p53_aux, float* GSK3b_p53_P_aux, float* Abeta_aux, float* AggAbeta_Proteasome_aux, float* AbetaPlaque_aux, float* Tau_aux, float* Tau_P1_aux, float* Tau_P2_aux, float* MT_Tau_aux, float* AggTau_aux, float* AggTau_Proteasome_aux, float* Proteasome_Tau_aux, float* PP1_aux, float* NFT_aux, float* ATP_aux, float* ADP_aux, float* AMP_aux, float* AbetaDimer_aux, float* AbetaPlaque_GliaA_aux, float* GliaI_aux, float* GliaM1_aux, float* GliaM2_aux, float* GliaA_aux, float* antiAb_aux, float* Abeta_antiAb_aux, float* AbetaDimer_antiAb_aux, float* degAbetaGlia_aux, float* disaggPlaque1_aux, float* disaggPlaque2_aux, float* Source_aux, float* Sink_aux) {
int reaction, stepCount = 0;
float time = 0;
float sum_p, sum_p_aux, timeStep, random;
float p[112];
int triggerEvent0 = 0;
float Mdm2 = *Mdm2_aux;
float p53 = *p53_aux;
float Mdm2_p53 = *Mdm2_p53_aux;
float Mdm2_mRNA = *Mdm2_mRNA_aux;
float p53_mRNA = *p53_mRNA_aux;
float ATMA = *ATMA_aux;
float ATMI = *ATMI_aux;
float p53_P = *p53_P_aux;
float Mdm2_P = *Mdm2_P_aux;
float IR = *IR_aux;
float ROS = *ROS_aux;
float damDNA = *damDNA_aux;
float E1 = *E1_aux;
float E2 = *E2_aux;
float E1_Ub = *E1_Ub_aux;
float E2_Ub = *E2_Ub_aux;
float Proteasome = *Proteasome_aux;
float Ub = *Ub_aux;
float p53DUB = *p53DUB_aux;
float Mdm2DUB = *Mdm2DUB_aux;
float DUB = *DUB_aux;
float Mdm2_p53_Ub = *Mdm2_p53_Ub_aux;
float Mdm2_p53_Ub2 = *Mdm2_p53_Ub2_aux;
float Mdm2_p53_Ub3 = *Mdm2_p53_Ub3_aux;
float Mdm2_p53_Ub4 = *Mdm2_p53_Ub4_aux;
float Mdm2_P1_p53_Ub4 = *Mdm2_P1_p53_Ub4_aux;
float Mdm2_Ub = *Mdm2_Ub_aux;
float Mdm2_Ub2 = *Mdm2_Ub2_aux;
float Mdm2_Ub3 = *Mdm2_Ub3_aux;
float Mdm2_Ub4 = *Mdm2_Ub4_aux;
float Mdm2_P_Ub = *Mdm2_P_Ub_aux;
float Mdm2_P_Ub2 = *Mdm2_P_Ub2_aux;
float Mdm2_P_Ub3 = *Mdm2_P_Ub3_aux;
float Mdm2_P_Ub4 = *Mdm2_P_Ub4_aux;
float p53_Ub4_Proteasome = *p53_Ub4_Proteasome_aux;
float Mdm2_Ub4_Proteasome = *Mdm2_Ub4_Proteasome_aux;
float Mdm2_P_Ub4_Proteasome = *Mdm2_P_Ub4_Proteasome_aux;
float GSK3b = *GSK3b_aux;
float GSK3b_p53 = *GSK3b_p53_aux;
float GSK3b_p53_P = *GSK3b_p53_P_aux;
float Abeta = *Abeta_aux;
float AggAbeta_Proteasome = *AggAbeta_Proteasome_aux;
float AbetaPlaque = *AbetaPlaque_aux;
float Tau = *Tau_aux;
float Tau_P1 = *Tau_P1_aux;
float Tau_P2 = *Tau_P2_aux;
float MT_Tau = *MT_Tau_aux;
float AggTau = *AggTau_aux;
float AggTau_Proteasome = *AggTau_Proteasome_aux;
float Proteasome_Tau = *Proteasome_Tau_aux;
float PP1 = *PP1_aux;
float NFT = *NFT_aux;
float ATP = *ATP_aux;
float ADP = *ADP_aux;
float AMP = *AMP_aux;
float AbetaDimer = *AbetaDimer_aux;
float AbetaPlaque_GliaA = *AbetaPlaque_GliaA_aux;
float GliaI = *GliaI_aux;
float GliaM1 = *GliaM1_aux;
float GliaM2 = *GliaM2_aux;
float GliaA = *GliaA_aux;
float antiAb = *antiAb_aux;
float Abeta_antiAb = *Abeta_antiAb_aux;
float AbetaDimer_antiAb = *AbetaDimer_antiAb_aux;
float degAbetaGlia = *degAbetaGlia_aux;
float disaggPlaque1 = *disaggPlaque1_aux;
float disaggPlaque2 = *disaggPlaque2_aux;
float Source = *Source_aux;
float Sink = *Sink_aux;
while(time < endTime){
p[0] = ksynp53mRNA * Source; 
p[1] = kdegp53mRNA * p53_mRNA; 
p[2] = ksynMdm2 * Mdm2_mRNA; 
p[3] = ksynMdm2mRNA * p53; 
p[4] = ksynMdm2mRNA * p53_P; 
p[5] = ksynMdm2mRNAGSK3bp53 * GSK3b_p53; 
p[6] = ksynMdm2mRNAGSK3bp53 * GSK3b_p53_P; 
p[7] = kdegMdm2mRNA * Mdm2_mRNA; 
p[8] = kbinMdm2p53 * p53 * Mdm2; 
p[9] = krelMdm2p53 * Mdm2_p53; 
p[10] = kbinGSK3bp53 * GSK3b * p53; 
p[11] = krelGSK3bp53 * GSK3b_p53; 
p[12] = kbinGSK3bp53 * GSK3b * p53_P; 
p[13] = krelGSK3bp53 * GSK3b_p53_P; 
p[14] = kbinE1Ub * E1 * Ub * ATP / (5000 + ATP); 
p[15] = kbinE2Ub * E2 * E1_Ub; 
p[16] = kMdm2Ub * Mdm2 * E2_Ub; 
p[17] = kMdm2PolyUb * Mdm2_Ub * E2_Ub; 
p[18] = kMdm2PolyUb * Mdm2_Ub2 * E2_Ub; 
p[19] = kMdm2PolyUb * Mdm2_Ub3 * E2_Ub; 
p[20] = kactDUBMdm2 * Mdm2_Ub4 * Mdm2DUB; 
p[21] = kactDUBMdm2 * Mdm2_Ub3 * Mdm2DUB; 
p[22] = kactDUBMdm2 * Mdm2_Ub2 * Mdm2DUB; 
p[23] = kactDUBMdm2 * Mdm2_Ub * Mdm2DUB; 
p[24] = kbinProt * Mdm2_Ub4 * Proteasome; 
p[25] = kdegMdm2 * Mdm2_Ub4_Proteasome * kproteff; 
p[26] = ksynp53 * p53_mRNA; 
p[27] = kp53Ub * E2_Ub * Mdm2_p53; 
p[28] = kp53PolyUb * Mdm2_p53_Ub * E2_Ub; 
p[29] = kp53PolyUb * Mdm2_p53_Ub2 * E2_Ub; 
p[30] = kp53PolyUb * Mdm2_p53_Ub3 * E2_Ub; 
p[31] = kactDUBp53 * Mdm2_p53_Ub4 * p53DUB; 
p[32] = kactDUBp53 * Mdm2_p53_Ub3 * p53DUB; 
p[33] = kactDUBp53 * Mdm2_p53_Ub2 * p53DUB; 
p[34] = kactDUBp53 * Mdm2_p53_Ub * p53DUB; 
p[35] = kphosMdm2GSK3b * Mdm2_p53_Ub4 * GSK3b; 
p[36] = kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53; 
p[37] = kphosMdm2GSK3bp53 * Mdm2_p53_Ub4 * GSK3b_p53_P; 
p[38] = kbinProt * Mdm2_P1_p53_Ub4 * Proteasome; 
p[39] = kdegp53 * kproteff * p53_Ub4_Proteasome * ATP / (5000 + ATP); 
p[40] = kbinMTTau * Tau; 
p[41] = krelMTTau * MT_Tau; 
p[42] = kphospTauGSK3bp53 * GSK3b_p53 * Tau; 
p[43] = kphospTauGSK3bp53 * GSK3b_p53 * Tau_P1; 
p[44] = kphospTauGSK3bp53 * GSK3b_p53_P * Tau; 
p[45] = kphospTauGSK3bp53 * GSK3b_p53_P * Tau_P1; 
p[46] = kphospTauGSK3b * GSK3b * Tau; 
p[47] = kphospTauGSK3b * GSK3b * Tau_P1; 
p[48] = kdephospTau * Tau_P2 * PP1; 
p[49] = kdephospTau * Tau_P1 * PP1; 
p[50] = kaggTauP1 * Tau_P1 * (Tau_P1 - 1) * 0.5; 
p[51] = kaggTauP1 * Tau_P1 * AggTau; 
p[52] = kaggTauP2 * Tau_P2 * (Tau_P2 - 1) * 0.5; 
p[53] = kaggTauP2 * Tau_P2 * AggTau; 
p[54] = kaggTau * Tau * (Tau - 1) * 0.5; 
p[55] = kaggTau * Tau * AggTau; 
p[56] = ktangfor * AggTau * (AggTau - 1) * 0.5; 
p[57] = ktangfor * AggTau * NFT; 
p[58] = kinhibprot * AggTau * Proteasome; 
p[59] = kprodAbeta * Source; 
p[60] = kprodAbeta2 * GSK3b_p53; 
p[61] = kprodAbeta2 * GSK3b_p53_P; 
p[62] = kinhibprot * AbetaDimer * Proteasome; 
p[63] = kdegAbeta * Abeta; 
p[64] = ksynp53mRNAAbeta * Abeta; 
p[65] = kdam * IR; 
p[66] = krepair * damDNA; 
p[67] = kactATM * damDNA * ATMI; 
p[68] = kphosp53 * p53 * ATMA; 
p[69] = kdephosp53 * p53_P; 
p[70] = kphosMdm2 * Mdm2 * ATMA; 
p[71] = kdephosMdm2 * Mdm2_P; 
p[72] = kMdm2PUb * Mdm2_P * E2_Ub; 
p[73] = kMdm2PolyUb * Mdm2_P_Ub * E2_Ub; 
p[74] = kMdm2PolyUb * Mdm2_P_Ub2 * E2_Ub; 
p[75] = kMdm2PolyUb * Mdm2_P_Ub3 * E2_Ub; 
p[76] = kactDUBMdm2 * Mdm2_P_Ub4 * Mdm2DUB; 
p[77] = kactDUBMdm2 * Mdm2_P_Ub3 * Mdm2DUB; 
p[78] = kactDUBMdm2 * Mdm2_P_Ub2 * Mdm2DUB; 
p[79] = kactDUBMdm2 * Mdm2_P_Ub * Mdm2DUB; 
p[80] = kbinProt * Mdm2_P_Ub4 * Proteasome; 
p[81] = kdegMdm2 * Mdm2_P_Ub4_Proteasome * kproteff; 
p[82] = kinactATM * ATMA; 
p[83] = kgenROSAbeta * Abeta; 
p[84] = kgenROSPlaque * AbetaPlaque; 
p[85] = kgenROSAbeta * AggAbeta_Proteasome; 
p[86] = kdamROS * ROS; 
p[87] = ksynTau * Source; 
p[88] = kbinTauProt * Tau * Proteasome; 
p[89] = kdegTau20SProt * Proteasome_Tau; 
p[90] = kaggAbeta * Abeta * (Abeta - 1) * 0.5; 
p[91] = kpf * AbetaDimer * (AbetaDimer - 1) * 0.5; 
p[92] = kpg * AbetaDimer * pow(AbetaPlaque, 2) / (pow(kpghalf, 2) + pow(AbetaPlaque, 2)); 
p[93] = kdisaggAbeta * AbetaDimer; 
p[94] = kdisaggAbeta1 * AbetaPlaque; 
p[95] = kdisaggAbeta2 * antiAb * AbetaPlaque; 
p[96] = kbinAbantiAb * Abeta * antiAb; 
p[97] = kbinAbantiAb * AbetaDimer * antiAb; 
p[98] = 10 * kdegAbeta * Abeta_antiAb; 
p[99] = 10 * kdegAbeta * AbetaDimer_antiAb; 
p[100] = kactglia1 * GliaI * AbetaPlaque; 
p[101] = kactglia1 * GliaM1 * AbetaPlaque; 
p[102] = kactglia2 * GliaM2 * antiAb; 
p[103] = kinactglia1 * GliaA; 
p[104] = kinactglia2 * GliaM2; 
p[105] = kinactglia2 * GliaM1; 
p[106] = kbinAbetaGlia * AbetaPlaque * GliaA; 
p[107] = krelAbetaGlia * AbetaPlaque_GliaA; 
p[108] = kdegAbetaGlia * AbetaPlaque_GliaA; 
p[109] = kgenROSGlia * AbetaPlaque_GliaA; 
p[110] = kdegAntiAb * antiAb; 
p[111] = kremROS * ROS; 
if(time >= step * stepCount){
atomicAdd(&output[stepCount*69 + 0], Mdm2);
atomicAdd(&output[stepCount*69 + 1], p53);
atomicAdd(&output[stepCount*69 + 2], Mdm2_p53);
atomicAdd(&output[stepCount*69 + 3], Mdm2_mRNA);
atomicAdd(&output[stepCount*69 + 4], p53_mRNA);
atomicAdd(&output[stepCount*69 + 5], ATMA);
atomicAdd(&output[stepCount*69 + 6], ATMI);
atomicAdd(&output[stepCount*69 + 7], p53_P);
atomicAdd(&output[stepCount*69 + 8], Mdm2_P);
atomicAdd(&output[stepCount*69 + 9], IR);
atomicAdd(&output[stepCount*69 + 10], ROS);
atomicAdd(&output[stepCount*69 + 11], damDNA);
atomicAdd(&output[stepCount*69 + 12], E1);
atomicAdd(&output[stepCount*69 + 13], E2);
atomicAdd(&output[stepCount*69 + 14], E1_Ub);
atomicAdd(&output[stepCount*69 + 15], E2_Ub);
atomicAdd(&output[stepCount*69 + 16], Proteasome);
atomicAdd(&output[stepCount*69 + 17], Ub);
atomicAdd(&output[stepCount*69 + 18], p53DUB);
atomicAdd(&output[stepCount*69 + 19], Mdm2DUB);
atomicAdd(&output[stepCount*69 + 20], DUB);
atomicAdd(&output[stepCount*69 + 21], Mdm2_p53_Ub);
atomicAdd(&output[stepCount*69 + 22], Mdm2_p53_Ub2);
atomicAdd(&output[stepCount*69 + 23], Mdm2_p53_Ub3);
atomicAdd(&output[stepCount*69 + 24], Mdm2_p53_Ub4);
atomicAdd(&output[stepCount*69 + 25], Mdm2_P1_p53_Ub4);
atomicAdd(&output[stepCount*69 + 26], Mdm2_Ub);
atomicAdd(&output[stepCount*69 + 27], Mdm2_Ub2);
atomicAdd(&output[stepCount*69 + 28], Mdm2_Ub3);
atomicAdd(&output[stepCount*69 + 29], Mdm2_Ub4);
atomicAdd(&output[stepCount*69 + 30], Mdm2_P_Ub);
atomicAdd(&output[stepCount*69 + 31], Mdm2_P_Ub2);
atomicAdd(&output[stepCount*69 + 32], Mdm2_P_Ub3);
atomicAdd(&output[stepCount*69 + 33], Mdm2_P_Ub4);
atomicAdd(&output[stepCount*69 + 34], p53_Ub4_Proteasome);
atomicAdd(&output[stepCount*69 + 35], Mdm2_Ub4_Proteasome);
atomicAdd(&output[stepCount*69 + 36], Mdm2_P_Ub4_Proteasome);
atomicAdd(&output[stepCount*69 + 37], GSK3b);
atomicAdd(&output[stepCount*69 + 38], GSK3b_p53);
atomicAdd(&output[stepCount*69 + 39], GSK3b_p53_P);
atomicAdd(&output[stepCount*69 + 40], Abeta);
atomicAdd(&output[stepCount*69 + 41], AggAbeta_Proteasome);
atomicAdd(&output[stepCount*69 + 42], AbetaPlaque);
atomicAdd(&output[stepCount*69 + 43], Tau);
atomicAdd(&output[stepCount*69 + 44], Tau_P1);
atomicAdd(&output[stepCount*69 + 45], Tau_P2);
atomicAdd(&output[stepCount*69 + 46], MT_Tau);
atomicAdd(&output[stepCount*69 + 47], AggTau);
atomicAdd(&output[stepCount*69 + 48], AggTau_Proteasome);
atomicAdd(&output[stepCount*69 + 49], Proteasome_Tau);
atomicAdd(&output[stepCount*69 + 50], PP1);
atomicAdd(&output[stepCount*69 + 51], NFT);
atomicAdd(&output[stepCount*69 + 52], ATP);
atomicAdd(&output[stepCount*69 + 53], ADP);
atomicAdd(&output[stepCount*69 + 54], AMP);
atomicAdd(&output[stepCount*69 + 55], AbetaDimer);
atomicAdd(&output[stepCount*69 + 56], AbetaPlaque_GliaA);
atomicAdd(&output[stepCount*69 + 57], GliaI);
atomicAdd(&output[stepCount*69 + 58], GliaM1);
atomicAdd(&output[stepCount*69 + 59], GliaM2);
atomicAdd(&output[stepCount*69 + 60], GliaA);
atomicAdd(&output[stepCount*69 + 61], antiAb);
atomicAdd(&output[stepCount*69 + 62], Abeta_antiAb);
atomicAdd(&output[stepCount*69 + 63], AbetaDimer_antiAb);
atomicAdd(&output[stepCount*69 + 64], degAbetaGlia);
atomicAdd(&output[stepCount*69 + 65], disaggPlaque1);
atomicAdd(&output[stepCount*69 + 66], disaggPlaque2);
atomicAdd(&output[stepCount*69 + 67], Source);
atomicAdd(&output[stepCount*69 + 68], Sink);
stepCount++;
}
sum_p = 0;
for(int i = 0; i < 112; i++){
sum_p += p[i];
}
hiprandState localState = state[threadIdx.x];
random = hiprand_uniform(&localState);
if(sum_p > 0) timeStep = -log(random)/sum_p;
else break;
random = hiprand_uniform(&localState);
reaction = -1;
sum_p_aux = 0;
random *= sum_p;
for(int i = 0; i < 112; i++){
sum_p_aux += p[i];
if(random < sum_p_aux){
reaction = i;
break;
}
}
speciesUpdate(reaction);
if(triggerEvent0 == 0 && time >= 345600){
triggerEvent0 = 1;
antiAb += 50;
}
time += timeStep;
}
}

__global__ 
void initCurand(hiprandState* state, unsigned long long seed){
hiprand_init(seed, threadIdx.x, 0, &state[threadIdx.x]);
}

int main()
{
hipError_t cudaStatus;
float* output;
float* dev_output;
output = (float*)malloc(34*69*sizeof(float));
for(int i = 0; i < 34*69; i++){
output[i] = 0;
}
cudaStatus = hipMalloc(&dev_output, 34*69*sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_output, output, 34*69*sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2 = 5.000000;
float* dev_Mdm2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2, &Mdm2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53 = 5.000000;
float* dev_p53 = 0;
cudaStatus = hipMalloc(&dev_p53, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53, &p53, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53 = 95.000000;
float* dev_Mdm2_p53 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53, &Mdm2_p53, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_mRNA = 10.000000;
float* dev_Mdm2_mRNA = 0;
cudaStatus = hipMalloc(&dev_Mdm2_mRNA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_mRNA, &Mdm2_mRNA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53_mRNA = 10.000000;
float* dev_p53_mRNA = 0;
cudaStatus = hipMalloc(&dev_p53_mRNA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53_mRNA, &p53_mRNA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ATMA = 0.000000;
float* dev_ATMA = 0;
cudaStatus = hipMalloc(&dev_ATMA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ATMA, &ATMA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ATMI = 200.000000;
float* dev_ATMI = 0;
cudaStatus = hipMalloc(&dev_ATMI, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ATMI, &ATMI, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53_P = 0.000000;
float* dev_p53_P = 0;
cudaStatus = hipMalloc(&dev_p53_P, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53_P, &p53_P, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P = 0.000000;
float* dev_Mdm2_P = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P, &Mdm2_P, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float IR = 0.000000;
float* dev_IR = 0;
cudaStatus = hipMalloc(&dev_IR, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_IR, &IR, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ROS = 0.000000;
float* dev_ROS = 0;
cudaStatus = hipMalloc(&dev_ROS, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ROS, &ROS, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float damDNA = 0.000000;
float* dev_damDNA = 0;
cudaStatus = hipMalloc(&dev_damDNA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_damDNA, &damDNA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E1 = 100.000000;
float* dev_E1 = 0;
cudaStatus = hipMalloc(&dev_E1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E1, &E1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E2 = 100.000000;
float* dev_E2 = 0;
cudaStatus = hipMalloc(&dev_E2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E2, &E2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E1_Ub = 0.000000;
float* dev_E1_Ub = 0;
cudaStatus = hipMalloc(&dev_E1_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E1_Ub, &E1_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float E2_Ub = 0.000000;
float* dev_E2_Ub = 0;
cudaStatus = hipMalloc(&dev_E2_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_E2_Ub, &E2_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Proteasome = 500.000000;
float* dev_Proteasome = 0;
cudaStatus = hipMalloc(&dev_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Proteasome, &Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Ub = 4000.000000;
float* dev_Ub = 0;
cudaStatus = hipMalloc(&dev_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Ub, &Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53DUB = 200.000000;
float* dev_p53DUB = 0;
cudaStatus = hipMalloc(&dev_p53DUB, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53DUB, &p53DUB, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2DUB = 200.000000;
float* dev_Mdm2DUB = 0;
cudaStatus = hipMalloc(&dev_Mdm2DUB, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2DUB, &Mdm2DUB, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float DUB = 200.000000;
float* dev_DUB = 0;
cudaStatus = hipMalloc(&dev_DUB, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_DUB, &DUB, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub = 0.000000;
float* dev_Mdm2_p53_Ub = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub, &Mdm2_p53_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub2 = 0.000000;
float* dev_Mdm2_p53_Ub2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub2, &Mdm2_p53_Ub2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub3 = 0.000000;
float* dev_Mdm2_p53_Ub3 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub3, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub3, &Mdm2_p53_Ub3, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_p53_Ub4 = 0.000000;
float* dev_Mdm2_p53_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_p53_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_p53_Ub4, &Mdm2_p53_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P1_p53_Ub4 = 0.000000;
float* dev_Mdm2_P1_p53_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P1_p53_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P1_p53_Ub4, &Mdm2_P1_p53_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub = 0.000000;
float* dev_Mdm2_Ub = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub, &Mdm2_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub2 = 0.000000;
float* dev_Mdm2_Ub2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub2, &Mdm2_Ub2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub3 = 0.000000;
float* dev_Mdm2_Ub3 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub3, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub3, &Mdm2_Ub3, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub4 = 0.000000;
float* dev_Mdm2_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub4, &Mdm2_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub = 0.000000;
float* dev_Mdm2_P_Ub = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub, &Mdm2_P_Ub, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub2 = 0.000000;
float* dev_Mdm2_P_Ub2 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub2, &Mdm2_P_Ub2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub3 = 0.000000;
float* dev_Mdm2_P_Ub3 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub3, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub3, &Mdm2_P_Ub3, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub4 = 0.000000;
float* dev_Mdm2_P_Ub4 = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub4, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub4, &Mdm2_P_Ub4, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float p53_Ub4_Proteasome = 0.000000;
float* dev_p53_Ub4_Proteasome = 0;
cudaStatus = hipMalloc(&dev_p53_Ub4_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_p53_Ub4_Proteasome, &p53_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_Ub4_Proteasome = 0.000000;
float* dev_Mdm2_Ub4_Proteasome = 0;
cudaStatus = hipMalloc(&dev_Mdm2_Ub4_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_Ub4_Proteasome, &Mdm2_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Mdm2_P_Ub4_Proteasome = 0.000000;
float* dev_Mdm2_P_Ub4_Proteasome = 0;
cudaStatus = hipMalloc(&dev_Mdm2_P_Ub4_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Mdm2_P_Ub4_Proteasome, &Mdm2_P_Ub4_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GSK3b = 500.000000;
float* dev_GSK3b = 0;
cudaStatus = hipMalloc(&dev_GSK3b, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GSK3b, &GSK3b, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GSK3b_p53 = 0.000000;
float* dev_GSK3b_p53 = 0;
cudaStatus = hipMalloc(&dev_GSK3b_p53, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GSK3b_p53, &GSK3b_p53, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GSK3b_p53_P = 0.000000;
float* dev_GSK3b_p53_P = 0;
cudaStatus = hipMalloc(&dev_GSK3b_p53_P, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GSK3b_p53_P, &GSK3b_p53_P, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Abeta = 0.000000;
float* dev_Abeta = 0;
cudaStatus = hipMalloc(&dev_Abeta, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Abeta, &Abeta, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AggAbeta_Proteasome = 0.000000;
float* dev_AggAbeta_Proteasome = 0;
cudaStatus = hipMalloc(&dev_AggAbeta_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AggAbeta_Proteasome, &AggAbeta_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaPlaque = 0.000000;
float* dev_AbetaPlaque = 0;
cudaStatus = hipMalloc(&dev_AbetaPlaque, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaPlaque, &AbetaPlaque, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Tau = 0.000000;
float* dev_Tau = 0;
cudaStatus = hipMalloc(&dev_Tau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Tau, &Tau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Tau_P1 = 0.000000;
float* dev_Tau_P1 = 0;
cudaStatus = hipMalloc(&dev_Tau_P1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Tau_P1, &Tau_P1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Tau_P2 = 0.000000;
float* dev_Tau_P2 = 0;
cudaStatus = hipMalloc(&dev_Tau_P2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Tau_P2, &Tau_P2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float MT_Tau = 100.000000;
float* dev_MT_Tau = 0;
cudaStatus = hipMalloc(&dev_MT_Tau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_MT_Tau, &MT_Tau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AggTau = 0.000000;
float* dev_AggTau = 0;
cudaStatus = hipMalloc(&dev_AggTau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AggTau, &AggTau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AggTau_Proteasome = 0.000000;
float* dev_AggTau_Proteasome = 0;
cudaStatus = hipMalloc(&dev_AggTau_Proteasome, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AggTau_Proteasome, &AggTau_Proteasome, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Proteasome_Tau = 0.000000;
float* dev_Proteasome_Tau = 0;
cudaStatus = hipMalloc(&dev_Proteasome_Tau, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Proteasome_Tau, &Proteasome_Tau, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float PP1 = 50.000000;
float* dev_PP1 = 0;
cudaStatus = hipMalloc(&dev_PP1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_PP1, &PP1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float NFT = 0.000000;
float* dev_NFT = 0;
cudaStatus = hipMalloc(&dev_NFT, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_NFT, &NFT, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ATP = 10000.000000;
float* dev_ATP = 0;
cudaStatus = hipMalloc(&dev_ATP, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ATP, &ATP, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float ADP = 1000.000000;
float* dev_ADP = 0;
cudaStatus = hipMalloc(&dev_ADP, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_ADP, &ADP, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AMP = 1000.000000;
float* dev_AMP = 0;
cudaStatus = hipMalloc(&dev_AMP, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AMP, &AMP, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaDimer = 0.000000;
float* dev_AbetaDimer = 0;
cudaStatus = hipMalloc(&dev_AbetaDimer, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaDimer, &AbetaDimer, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaPlaque_GliaA = 0.000000;
float* dev_AbetaPlaque_GliaA = 0;
cudaStatus = hipMalloc(&dev_AbetaPlaque_GliaA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaPlaque_GliaA, &AbetaPlaque_GliaA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaI = 100.000000;
float* dev_GliaI = 0;
cudaStatus = hipMalloc(&dev_GliaI, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaI, &GliaI, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaM1 = 0.000000;
float* dev_GliaM1 = 0;
cudaStatus = hipMalloc(&dev_GliaM1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaM1, &GliaM1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaM2 = 0.000000;
float* dev_GliaM2 = 0;
cudaStatus = hipMalloc(&dev_GliaM2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaM2, &GliaM2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float GliaA = 0.000000;
float* dev_GliaA = 0;
cudaStatus = hipMalloc(&dev_GliaA, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_GliaA, &GliaA, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float antiAb = 0.000000;
float* dev_antiAb = 0;
cudaStatus = hipMalloc(&dev_antiAb, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_antiAb, &antiAb, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Abeta_antiAb = 0.000000;
float* dev_Abeta_antiAb = 0;
cudaStatus = hipMalloc(&dev_Abeta_antiAb, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Abeta_antiAb, &Abeta_antiAb, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float AbetaDimer_antiAb = 0.000000;
float* dev_AbetaDimer_antiAb = 0;
cudaStatus = hipMalloc(&dev_AbetaDimer_antiAb, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_AbetaDimer_antiAb, &AbetaDimer_antiAb, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float degAbetaGlia = 0.000000;
float* dev_degAbetaGlia = 0;
cudaStatus = hipMalloc(&dev_degAbetaGlia, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_degAbetaGlia, &degAbetaGlia, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float disaggPlaque1 = 0.000000;
float* dev_disaggPlaque1 = 0;
cudaStatus = hipMalloc(&dev_disaggPlaque1, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_disaggPlaque1, &disaggPlaque1, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float disaggPlaque2 = 0.000000;
float* dev_disaggPlaque2 = 0;
cudaStatus = hipMalloc(&dev_disaggPlaque2, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_disaggPlaque2, &disaggPlaque2, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Source = 1.000000;
float* dev_Source = 0;
cudaStatus = hipMalloc(&dev_Source, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Source, &Source, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
float Sink = 1.000000;
float* dev_Sink = 0;
cudaStatus = hipMalloc(&dev_Sink, sizeof(float));
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMalloc failed!");goto Error;}
cudaStatus = hipMemcpy(dev_Sink, &Sink, sizeof(float), hipMemcpyHostToDevice);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
hiprandState *devStates;
CUDA_CALL(hipMalloc((void **)&devStates, 32 * sizeof(hiprandState)));
initCurand<<<1, 32>>>(devStates, SEED);
simulate<<<1, 32>>>(dev_output, devStates, 60.000000, 2000.000000, dev_Mdm2, dev_p53, dev_Mdm2_p53, dev_Mdm2_mRNA, dev_p53_mRNA, dev_ATMA, dev_ATMI, dev_p53_P, dev_Mdm2_P, dev_IR, dev_ROS, dev_damDNA, dev_E1, dev_E2, dev_E1_Ub, dev_E2_Ub, dev_Proteasome, dev_Ub, dev_p53DUB, dev_Mdm2DUB, dev_DUB, dev_Mdm2_p53_Ub, dev_Mdm2_p53_Ub2, dev_Mdm2_p53_Ub3, dev_Mdm2_p53_Ub4, dev_Mdm2_P1_p53_Ub4, dev_Mdm2_Ub, dev_Mdm2_Ub2, dev_Mdm2_Ub3, dev_Mdm2_Ub4, dev_Mdm2_P_Ub, dev_Mdm2_P_Ub2, dev_Mdm2_P_Ub3, dev_Mdm2_P_Ub4, dev_p53_Ub4_Proteasome, dev_Mdm2_Ub4_Proteasome, dev_Mdm2_P_Ub4_Proteasome, dev_GSK3b, dev_GSK3b_p53, dev_GSK3b_p53_P, dev_Abeta, dev_AggAbeta_Proteasome, dev_AbetaPlaque, dev_Tau, dev_Tau_P1, dev_Tau_P2, dev_MT_Tau, dev_AggTau, dev_AggTau_Proteasome, dev_Proteasome_Tau, dev_PP1, dev_NFT, dev_ATP, dev_ADP, dev_AMP, dev_AbetaDimer, dev_AbetaPlaque_GliaA, dev_GliaI, dev_GliaM1, dev_GliaM2, dev_GliaA, dev_antiAb, dev_Abeta_antiAb, dev_AbetaDimer_antiAb, dev_degAbetaGlia, dev_disaggPlaque1, dev_disaggPlaque2, dev_Source, dev_Sink);

cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess) {fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));goto Error;}

cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess) {fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);goto Error;}

cudaStatus = hipMemcpy(output, dev_output, 34*69*sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2, dev_Mdm2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53, dev_p53, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53, dev_Mdm2_p53, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_mRNA, dev_Mdm2_mRNA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53_mRNA, dev_p53_mRNA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ATMA, dev_ATMA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ATMI, dev_ATMI, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53_P, dev_p53_P, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P, dev_Mdm2_P, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&IR, dev_IR, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ROS, dev_ROS, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&damDNA, dev_damDNA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E1, dev_E1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E2, dev_E2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E1_Ub, dev_E1_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&E2_Ub, dev_E2_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Proteasome, dev_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Ub, dev_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53DUB, dev_p53DUB, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2DUB, dev_Mdm2DUB, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&DUB, dev_DUB, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub, dev_Mdm2_p53_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub2, dev_Mdm2_p53_Ub2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub3, dev_Mdm2_p53_Ub3, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_p53_Ub4, dev_Mdm2_p53_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P1_p53_Ub4, dev_Mdm2_P1_p53_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub, dev_Mdm2_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub2, dev_Mdm2_Ub2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub3, dev_Mdm2_Ub3, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub4, dev_Mdm2_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub, dev_Mdm2_P_Ub, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub2, dev_Mdm2_P_Ub2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub3, dev_Mdm2_P_Ub3, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub4, dev_Mdm2_P_Ub4, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&p53_Ub4_Proteasome, dev_p53_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_Ub4_Proteasome, dev_Mdm2_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Mdm2_P_Ub4_Proteasome, dev_Mdm2_P_Ub4_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GSK3b, dev_GSK3b, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GSK3b_p53, dev_GSK3b_p53, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GSK3b_p53_P, dev_GSK3b_p53_P, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Abeta, dev_Abeta, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AggAbeta_Proteasome, dev_AggAbeta_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaPlaque, dev_AbetaPlaque, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Tau, dev_Tau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Tau_P1, dev_Tau_P1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Tau_P2, dev_Tau_P2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&MT_Tau, dev_MT_Tau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AggTau, dev_AggTau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AggTau_Proteasome, dev_AggTau_Proteasome, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Proteasome_Tau, dev_Proteasome_Tau, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&PP1, dev_PP1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&NFT, dev_NFT, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ATP, dev_ATP, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&ADP, dev_ADP, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AMP, dev_AMP, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaDimer, dev_AbetaDimer, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaPlaque_GliaA, dev_AbetaPlaque_GliaA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaI, dev_GliaI, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaM1, dev_GliaM1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaM2, dev_GliaM2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&GliaA, dev_GliaA, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&antiAb, dev_antiAb, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Abeta_antiAb, dev_Abeta_antiAb, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&AbetaDimer_antiAb, dev_AbetaDimer_antiAb, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&degAbetaGlia, dev_degAbetaGlia, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&disaggPlaque1, dev_disaggPlaque1, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&disaggPlaque2, dev_disaggPlaque2, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Source, dev_Source, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipMemcpy(&Sink, dev_Sink, sizeof(float), hipMemcpyDeviceToHost);
if (cudaStatus != hipSuccess) {fprintf(stderr, "hipMemcpy failed!");goto Error;}
cudaStatus = hipGetLastError(); if (cudaStatus != hipSuccess) {fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));goto Error;}

cudaStatus = hipDeviceSynchronize(); if (cudaStatus != hipSuccess) {fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);goto Error;}FILE* results = fopen("results.csv", "w");
if(results == NULL){
printf("Error acesssing results!");
exit(1);
}
fprintf(results, "time");
fprintf(results, ", Mdm2");
fprintf(results, ", p53");
fprintf(results, ", Mdm2_p53");
fprintf(results, ", Mdm2_mRNA");
fprintf(results, ", p53_mRNA");
fprintf(results, ", ATMA");
fprintf(results, ", ATMI");
fprintf(results, ", p53_P");
fprintf(results, ", Mdm2_P");
fprintf(results, ", IR");
fprintf(results, ", ROS");
fprintf(results, ", damDNA");
fprintf(results, ", E1");
fprintf(results, ", E2");
fprintf(results, ", E1_Ub");
fprintf(results, ", E2_Ub");
fprintf(results, ", Proteasome");
fprintf(results, ", Ub");
fprintf(results, ", p53DUB");
fprintf(results, ", Mdm2DUB");
fprintf(results, ", DUB");
fprintf(results, ", Mdm2_p53_Ub");
fprintf(results, ", Mdm2_p53_Ub2");
fprintf(results, ", Mdm2_p53_Ub3");
fprintf(results, ", Mdm2_p53_Ub4");
fprintf(results, ", Mdm2_P1_p53_Ub4");
fprintf(results, ", Mdm2_Ub");
fprintf(results, ", Mdm2_Ub2");
fprintf(results, ", Mdm2_Ub3");
fprintf(results, ", Mdm2_Ub4");
fprintf(results, ", Mdm2_P_Ub");
fprintf(results, ", Mdm2_P_Ub2");
fprintf(results, ", Mdm2_P_Ub3");
fprintf(results, ", Mdm2_P_Ub4");
fprintf(results, ", p53_Ub4_Proteasome");
fprintf(results, ", Mdm2_Ub4_Proteasome");
fprintf(results, ", Mdm2_P_Ub4_Proteasome");
fprintf(results, ", GSK3b");
fprintf(results, ", GSK3b_p53");
fprintf(results, ", GSK3b_p53_P");
fprintf(results, ", Abeta");
fprintf(results, ", AggAbeta_Proteasome");
fprintf(results, ", AbetaPlaque");
fprintf(results, ", Tau");
fprintf(results, ", Tau_P1");
fprintf(results, ", Tau_P2");
fprintf(results, ", MT_Tau");
fprintf(results, ", AggTau");
fprintf(results, ", AggTau_Proteasome");
fprintf(results, ", Proteasome_Tau");
fprintf(results, ", PP1");
fprintf(results, ", NFT");
fprintf(results, ", ATP");
fprintf(results, ", ADP");
fprintf(results, ", AMP");
fprintf(results, ", AbetaDimer");
fprintf(results, ", AbetaPlaque_GliaA");
fprintf(results, ", GliaI");
fprintf(results, ", GliaM1");
fprintf(results, ", GliaM2");
fprintf(results, ", GliaA");
fprintf(results, ", antiAb");
fprintf(results, ", Abeta_antiAb");
fprintf(results, ", AbetaDimer_antiAb");
fprintf(results, ", degAbetaGlia");
fprintf(results, ", disaggPlaque1");
fprintf(results, ", disaggPlaque2");
fprintf(results, ", Source");
fprintf(results, ", Sink");
fprintf(results, "\n");
for(int i = 0; i < 34; i++){
fprintf(results, "%lf", 60.000000*i);
for(int j = 0; j < 69; j++){
fprintf(results, ", %lf", output[69*i+j]/32);
}
fprintf(results, "\n");
}
fprintf(results, "\n");
Error:
hipFree(dev_output);
hipFree(dev_Mdm2);
hipFree(dev_p53);
hipFree(dev_Mdm2_p53);
hipFree(dev_Mdm2_mRNA);
hipFree(dev_p53_mRNA);
hipFree(dev_ATMA);
hipFree(dev_ATMI);
hipFree(dev_p53_P);
hipFree(dev_Mdm2_P);
hipFree(dev_IR);
hipFree(dev_ROS);
hipFree(dev_damDNA);
hipFree(dev_E1);
hipFree(dev_E2);
hipFree(dev_E1_Ub);
hipFree(dev_E2_Ub);
hipFree(dev_Proteasome);
hipFree(dev_Ub);
hipFree(dev_p53DUB);
hipFree(dev_Mdm2DUB);
hipFree(dev_DUB);
hipFree(dev_Mdm2_p53_Ub);
hipFree(dev_Mdm2_p53_Ub2);
hipFree(dev_Mdm2_p53_Ub3);
hipFree(dev_Mdm2_p53_Ub4);
hipFree(dev_Mdm2_P1_p53_Ub4);
hipFree(dev_Mdm2_Ub);
hipFree(dev_Mdm2_Ub2);
hipFree(dev_Mdm2_Ub3);
hipFree(dev_Mdm2_Ub4);
hipFree(dev_Mdm2_P_Ub);
hipFree(dev_Mdm2_P_Ub2);
hipFree(dev_Mdm2_P_Ub3);
hipFree(dev_Mdm2_P_Ub4);
hipFree(dev_p53_Ub4_Proteasome);
hipFree(dev_Mdm2_Ub4_Proteasome);
hipFree(dev_Mdm2_P_Ub4_Proteasome);
hipFree(dev_GSK3b);
hipFree(dev_GSK3b_p53);
hipFree(dev_GSK3b_p53_P);
hipFree(dev_Abeta);
hipFree(dev_AggAbeta_Proteasome);
hipFree(dev_AbetaPlaque);
hipFree(dev_Tau);
hipFree(dev_Tau_P1);
hipFree(dev_Tau_P2);
hipFree(dev_MT_Tau);
hipFree(dev_AggTau);
hipFree(dev_AggTau_Proteasome);
hipFree(dev_Proteasome_Tau);
hipFree(dev_PP1);
hipFree(dev_NFT);
hipFree(dev_ATP);
hipFree(dev_ADP);
hipFree(dev_AMP);
hipFree(dev_AbetaDimer);
hipFree(dev_AbetaPlaque_GliaA);
hipFree(dev_GliaI);
hipFree(dev_GliaM1);
hipFree(dev_GliaM2);
hipFree(dev_GliaA);
hipFree(dev_antiAb);
hipFree(dev_Abeta_antiAb);
hipFree(dev_AbetaDimer_antiAb);
hipFree(dev_degAbetaGlia);
hipFree(dev_disaggPlaque1);
hipFree(dev_disaggPlaque2);
hipFree(dev_Source);
hipFree(dev_Sink);

    return 0;
}